#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This application demonstrates how to use the CUDA API to use multiple GPUs,
 * with an emphasis on simple illustration of the techniques (not on performance).
 *
 * Note that in order to detect multiple GPUs in your system you have to disable
 * SLI in the nvidia control panel. Otherwise only one GPU is visible to the
 * application. On the other side, you can still extend your desktop to screens
 * attached to both GPUs.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_timer.h>
#include <hip/hip_runtime_api.h>

#include "DPMMSubClusters.h"
#include "ds.h"
#include "module_tests.h"

#include <random>
#include "hiprand.h"
#include "cudaKernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int MAX_GPU_COUNT = 32;
const int DATA_N        = 1048576 * 32;

////////////////////////////////////////////////////////////////////////////////
// Simple reduction kernel.
// Refer to the 'reduction' CUDA Sample describing
// reduction optimization strategies
////////////////////////////////////////////////////////////////////////////////
__global__ static void reduceKernel(float *d_Result, float *d_Input, int N)
{
    const int     tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int threadN = gridDim.x * blockDim.x;
    float sum = 0;

    for (int pos = tid; pos < N; pos += threadN)
        sum += d_Input[pos];

    d_Result[tid] = sum;
}

//std::mt19937_64 eng(rand()); //Use the 64-bit Mersenne Twister 19937 generator
							   //and seed it with entropy.

//__global__ void sample_log_cat_array_gpu(double *log_likelihood_array, int cols, int rows, double *labels)
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//	//Define the distribution, by default it goes from 0 to MAX(unsigned long long)
//
//		//Setup the weights (in this case linearly weighted)
//
//	std::vector<double> weights(cols);
//	for (int j = 0; j < cols; ++j)
//	{
//		weights[j] = log_likelihood_array[rows*j+idx];
//	}
//
//	// Create the distribution with those weights
//	std::discrete_distribution<int> distr(weights.begin(), weights.end());
//	//Generate random numbers
//	labels[idx] = distr(eng) + 1;
//}

/*
init_model()

Initialize the model, loading the data from external 'npy' files, specified in the params file.
All prior data as been included previously, and is globaly accessed by the function.

Returns an 'dp_parallel_sampling' (e.g.the main data structure) with the configured parameters and data.
*/

//dp_parallel_sampling init_model()
//{
//	dp_parallel_sampling result;
	//if random_seed != nothing
	//	@eval @everywhere seed!($random_seed)
	//	end
	//	if (use_verbose)
	//	{
//	println("Loading and distributing data:")
	//		@time data = distribute(Float32.(load_data(data_path, prefix = data_prefix)))
	//	else
	//	{	data = distribute(Float32.(load_data(data_path, prefix = data_prefix)))
	//		}
	//		total_dim = size(data, 2)
	//		model_hyperparams = model_hyper_params(hyper_params, ?, total_dim)

	//		labels = distribute(rand(1:initial_clusters, (size(data, 2))) . + ((outlier_mod > 0) ? 1 : 0))
	//		labels_subcluster = distribute(rand(1:2, (size(data, 2))))
	//		group = local_group(model_hyperparams, data, labels, labels_subcluster, local_cluster[], Float32[])
	//		return dp_parallel_sampling(model_hyperparams, group)
//	return result;
//}

/*
dp_parallel(model_params::String; verbose = true, save_model = true, burnout = 5, gt = nothing)

Run the model in advanced mode.
# Args and Kwargs
- 'model_params::String' A path to a parameters file(see below)
- 'verbose' will perform prints on every iteration.
- 'save_model' will save a checkpoint every 'X' iterations, where 'X' is specified in the parameter file.
- 'burnout' how long to wait after creating a cluster, and allowing it to split / merge
- 'gt' Ground truth, when supplied, will perform NMI and VI analysis on every iteration.

# Return values
dp_model, iter_count, nmi_score_history, liklihood_history, cluster_count_history
- 'dp_model' The DPMM model inferred
- 'iter_count' Timing for each iteration
- 'nmi_score_history' NMI score per iteration(if gt suppled)
- 'likelihood_history' Log likelihood per iteration.
- 'cluster_count_history' Cluster counts per iteration.
*/

/*
void dp_parallel(char* model_params, bool verbose = true, char* gt = NULL)
{
	include(model_params)
		global use_verbose = verbose
		dp_model = init_model()
		global leader_dict = get_node_leaders_dict()
		global should_save_model = enable_saving
		global ground_truth = gt
		global burnout_period = burnout_period
		global max_num_of_clusters = max_clusters
		init_first_clusters!(dp_model, initial_clusters)
		if (use_verbose)
		{
			println("Node Leaders:")
				println(leader_dict)
		}
	@eval @everywhere global hard_clustering = $hard_clustering
		return run_model(dp_model, 1, model_params)
}
*/


int main3(int argc, char **argv)
{
    //Solver config
    TGPUplan      plan[MAX_GPU_COUNT];

    //GPU reduction results
    float     h_SumGPU[MAX_GPU_COUNT];

    float sumGPU;

    int i, j, gpuBase, GPU_N;

    const int  BLOCK_N = 32;
    const int THREAD_N = 256;
    const int  ACCUM_N = BLOCK_N * THREAD_N;

    printf("Starting DPMMSubClusters\n");
    checkCudaErrors(hipGetDeviceCount(&GPU_N));

    if (GPU_N > MAX_GPU_COUNT)
    {
        GPU_N = MAX_GPU_COUNT;
    }

    printf("CUDA-capable device count: %i\n", GPU_N);

    printf("Generating input data...\n\n");

    //Subdividing input data across GPUs
    //Get data sizes for each GPU
    for (i = 0; i < GPU_N; i++)
    {
        plan[i].dataN = DATA_N / GPU_N;
    }

    //Take into account "odd" data sizes
    for (i = 0; i < DATA_N % GPU_N; i++)
    {
        plan[i].dataN++;
    }

    //Assign data ranges to GPUs
    gpuBase = 0;

    for (i = 0; i < GPU_N; i++)
    {
        plan[i].h_Sum = h_SumGPU + i;
        gpuBase += plan[i].dataN;
    }

    //Create streams for issuing GPU command asynchronously and allocate memory (GPU and System page-locked)
    for (i = 0; i < GPU_N; i++)
    {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipStreamCreate(&plan[i].stream));
        //Allocate memory
        checkCudaErrors(hipMalloc((void **)&plan[i].d_Data, plan[i].dataN * sizeof(float)));
        checkCudaErrors(hipMalloc((void **)&plan[i].d_Sum, ACCUM_N * sizeof(float)));
        checkCudaErrors(hipHostMalloc((void **)&plan[i].h_Sum_from_device, ACCUM_N * sizeof(float)));
        checkCudaErrors(hipHostMalloc((void **)&plan[i].h_Data, plan[i].dataN * sizeof(float)));

        for (j = 0; j < plan[i].dataN; j++)
        {
            plan[i].h_Data[j] = (float)rand() / (float)RAND_MAX;
        }
    }

    //Start timing and compute on GPU(s)
    printf("Computing with %d GPUs...\n", GPU_N);
    // create and start timer
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);

    // start the timer
    sdkStartTimer(&timer);

    //Copy data to GPU, launch the kernel and copy data back. All asynchronously
    for (i = 0; i < GPU_N; i++)
    {
        //Set device
        checkCudaErrors(hipSetDevice(i));

        //Copy input data from CPU
        checkCudaErrors(hipMemcpyAsync(plan[i].d_Data, plan[i].h_Data, plan[i].dataN * sizeof(float), hipMemcpyHostToDevice, plan[i].stream));

        //Perform GPU computations
        reduceKernel<<<BLOCK_N, THREAD_N, 0, plan[i].stream>>>(plan[i].d_Sum, plan[i].d_Data, plan[i].dataN);
        getLastCudaError("reduceKernel() execution failed.\n");

        //Read back GPU results
        checkCudaErrors(hipMemcpyAsync(plan[i].h_Sum_from_device, plan[i].d_Sum, ACCUM_N *sizeof(float), hipMemcpyDeviceToHost, plan[i].stream));
    }

    //Process GPU results
    for (i = 0; i < GPU_N; i++)
    {
        float sum;

        //Set device
        checkCudaErrors(hipSetDevice(i));

        //Wait for all operations to finish
        hipStreamSynchronize(plan[i].stream);

        //Finalize GPU reduction for current subvector
        sum = 0;

        for (j = 0; j < ACCUM_N; j++)
        {
            sum += plan[i].h_Sum_from_device[j];
        }

        *(plan[i].h_Sum) = (float)sum;

        //Shut down this GPU
        checkCudaErrors(hipHostFree(plan[i].h_Sum_from_device));
        checkCudaErrors(hipFree(plan[i].d_Sum));
        checkCudaErrors(hipFree(plan[i].d_Data));
        checkCudaErrors(hipStreamDestroy(plan[i].stream));
    }

    sumGPU = 0;

    for (i = 0; i < GPU_N; i++)
    {
        sumGPU += h_SumGPU[i];
    }

    sdkStopTimer(&timer);
    printf("  GPU Processing time: %f (ms)\n\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
  
    // GPU results
    printf("  GPU sum: %f\n", sumGPU);

    // Cleanup and shutdown
    for (i = 0; i < GPU_N; i++)
    {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipHostFree(plan[i].h_Data));
    }

    exit(EXIT_SUCCESS);
}

int main(int argc, char** argv)
{
 //   setNbThreads(12);
 //   Eigen::initParallel();
    printf("Eigen uses %ld threads\n", Eigen::nbThreads());

    module_tests mt;
    //	mt.RunModuleFromFile("mnm_data.npy", "E:\\VIL\\DPMMSubClusters\\x64\\Debug\\");
    //	MatrixXd mat;
    //	mt.ReadPnyFileIntoData("mnm_data.npy", "E:\\VIL\\DPMMSubClusters\\x64\\Debug\\", mat);
        //mt.RandomMess();
    mt.RandomMessHighDim();

    return main3(argc, argv);
}