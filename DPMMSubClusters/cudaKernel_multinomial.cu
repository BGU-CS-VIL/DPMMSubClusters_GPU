#ifndef CudaKernel_multinomial_CU
#define CudaKernel_multinomial_CU

#include "cudaKernel_multinomial.cuh"

void cudaKernel_multinomial::log_likelihood(Eigen::VectorXd& r, const Eigen::MatrixXd& x, const distribution_sample* distribution_sample)
{
	const multinomial_dist* pDistribution_sample = (multinomial_dist*)distribution_sample;
	Eigen::VectorXd alpha_vec = Eigen::VectorXd::Map(pDistribution_sample->alpha.data(), pDistribution_sample->alpha.size());
	r = (alpha_vec.adjoint() * x).row(0);
}

void cudaKernel_multinomial::log_likelihood_v2(double* d_r, int* d_indices, int indicesSize, int dim, const distribution_sample* distribution_sample, hipStream_t& stream)
{
	//TODO
}


#endif //CudaKernel_multinomial_CU