#include "hip/hip_runtime.h"
#ifndef CudaKernel_multinomial_CU
#define CudaKernel_multinomial_CU

#include "cudaKernel_multinomial.cuh"


__global__ void get_first_row_multiple_alpha_with_x(double* d_points, int dim, int* d_indices, int indicesSize, double* d_alpha, double* d_r)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < indicesSize)
	{
		double sum = 0;
		for (int i = 0; i < dim; i++)
		{
			sum += d_alpha[i] * d_points[IDX2C(i, d_indices[idx], dim)];
		}
		d_r[idx] = sum;
	}
}


__global__ void get_first_row_multiple_alpha_with_x_all(double* d_points, int dim, int indicesSize, double* d_alpha, double* d_r)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < indicesSize)
	{
		double sum = 0;
		for (int i = 0; i < dim; i++)
		{
			sum += d_alpha[i] * d_points[IDX2C(i, idx, dim)];
		}
		d_r[idx] = sum;
	}
}

void cudaKernel_multinomial::log_likelihood_sub_labels(
	double* d_r,
	int r_offset,
	int* d_indices,
	int indicesSize,
	int dim,
	const std::shared_ptr<distribution_sample>& distribution_sample,
	hipStream_t& stream,
	int deviceId)
{
	double* d_alpha;
	multinomial_dist* pDistribution_sample = dynamic_cast<multinomial_dist*>(distribution_sample.get());

	runCuda(hipMallocAsync((void**)&d_alpha, sizeof(double) * pDistribution_sample->alpha.size(), stream));
	runCuda(hipMemcpyAsync(d_alpha, pDistribution_sample->alpha.data(), sizeof(double) * pDistribution_sample->alpha.size(), hipMemcpyHostToDevice, stream));
	
	dim3 blocks_size = dim3(indicesSize / threads.x + 1);

	get_first_row_multiple_alpha_with_x << <blocks_size, threads, 0, stream >> > (gpuCapabilities[deviceId].d_points, dim, d_indices, indicesSize, d_alpha, d_r+r_offset);

	runCuda(hipFreeAsync(d_alpha, stream));
}

void cudaKernel_multinomial::log_likelihood_labels(
	double* d_r,
	int dim,
	double weight,
	const std::shared_ptr<distribution_sample>& distribution_sample,
	hipStream_t& stream,
	int deviceId)
{
	double* d_alpha;
	multinomial_dist* pDistribution_sample = dynamic_cast<multinomial_dist*>(distribution_sample.get());

	runCuda(hipMallocAsync((void**)&d_alpha, sizeof(double) * pDistribution_sample->alpha.size(), stream));
	runCuda(hipMemcpyAsync(d_alpha, pDistribution_sample->alpha.data(), sizeof(double) * pDistribution_sample->alpha.size(), hipMemcpyHostToDevice, stream));

	get_first_row_multiple_alpha_with_x_all << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].d_points, dim, numLabels, d_alpha, d_r);

	runCuda(hipFreeAsync(d_alpha, stream));
}

#endif //CudaKernel_multinomial_CU