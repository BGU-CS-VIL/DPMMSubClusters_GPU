#include "hip/hip_runtime.h"
#ifndef CudaKernel_multinomial_CU
#define CudaKernel_multinomial_CU

#include "cudaKernel_multinomial.cuh"
#include "priors/multinomial_sufficient_statistics.h"


__global__ void get_first_row_multiple_alpha_with_x(double* d_points, int dim, int* d_indices, int indicesSize, double* d_alpha, double* d_r)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < indicesSize)
	{
		double sum = 0;
		for (int i = 0; i < dim; i++)
		{
			sum += d_alpha[i] * d_points[IDX2C(i, d_indices[idx], dim)];
		}
		d_r[idx] = sum;
	}
}

__global__ void get_first_row_multiple_alpha_with_x_all(double* d_points, int dim, int indicesSize, double* d_alpha, double* d_r)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < indicesSize)
	{
		double sum = 0;
		for (int i = 0; i < dim; i++)
		{
			sum += d_alpha[i] * d_points[IDX2C(i, idx, dim)];
		}
		d_r[idx] = sum;
	}
}

void cudaKernel_multinomial::log_likelihood_sub_labels(
	double* d_r,
	int r_offset,
	int* d_indices,
	int indicesSize,
	int dim,
	const std::shared_ptr<distribution_sample>& distribution_sample,
	hipStream_t& stream,
	int deviceId)
{
	double* d_alpha;
	multinomial_dist* pDistribution_sample = dynamic_cast<multinomial_dist*>(distribution_sample.get());

	runCuda(hipMallocAsync((void**)&d_alpha, sizeof(double) * pDistribution_sample->alpha.size(), stream));
	runCuda(hipMemcpyAsync(d_alpha, pDistribution_sample->alpha.data(), sizeof(double) * pDistribution_sample->alpha.size(), hipMemcpyHostToDevice, stream));
	
	dim3 blocks_size = dim3(indicesSize / threads.x + 1);

	get_first_row_multiple_alpha_with_x << <blocks_size, threads, 0, stream >> > (gpuCapabilities[deviceId].d_points, dim, d_indices, indicesSize, d_alpha, d_r+r_offset);
	runCuda(hipPeekAtLastError());
	runCuda(hipFreeAsync(d_alpha, stream));
}

void cudaKernel_multinomial::log_likelihood_labels(
	double* d_r,
	double weight,
	const std::shared_ptr<distribution_sample>& distribution_sample,
	hipStream_t& stream,
	int deviceId)
{
	double* d_alpha;
	multinomial_dist* pDistribution_sample = dynamic_cast<multinomial_dist*>(distribution_sample.get());
	
	runCuda(hipMallocAsync((void**)&d_alpha, sizeof(double) * pDistribution_sample->alpha.size(), stream));
	runCuda(hipMemcpyAsync(d_alpha, pDistribution_sample->alpha.data(), sizeof(double) * pDistribution_sample->alpha.size(), hipMemcpyHostToDevice, stream));

	get_first_row_multiple_alpha_with_x_all << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].d_points, gpuCapabilities[deviceId].pointsRows, numLabels, d_alpha, d_r);
	runCuda(hipPeekAtLastError());
	runCuda(hipFreeAsync(d_alpha, stream));
}

void cudaKernel_multinomial::do_create_sufficient_statistics(
	double* d_pts,
	int rows,
	int* d_cols,
	const std::shared_ptr<hyperparams>& hyperParams,
	const std::shared_ptr<hyperparams>& posterior,
	hipStream_t& stream,
	std::shared_ptr<sufficient_statistics>& ss,
	int deviceId)
{
	int cols;
	runCuda(hipMemcpyAsync(&cols, d_cols, sizeof(int), hipMemcpyDeviceToHost, stream));

	ss = std::make_shared<multinomial_sufficient_statistics>();
	double* d_sum_rowwise;
	runCuda(hipMallocAsync(&d_sum_rowwise, rows * sizeof(double), stream));

	sum_rowwise(d_pts, d_sum_rowwise, rows, cols, stream);
	ss->points_sum.resize(rows);
	ss->N = cols;
	runCuda(hipMemcpyAsync(ss->points_sum.data(), d_sum_rowwise, sizeof(double) * rows, hipMemcpyDeviceToHost, stream));

	runCuda(hipFreeAsync(d_sum_rowwise, stream));
}


#endif //CudaKernel_multinomial_CU