#ifndef CudaKernel_multinomial_CU
#define CudaKernel_multinomial_CU

#include "cudaKernel_multinomial.cuh"

void cudaKernel_multinomial::log_likelihood(Eigen::VectorXd& r, const Eigen::MatrixXd& x, const distribution_sample* distribution_sample)
{
	const multinomial_dist* pDistribution_sample = (multinomial_dist*)distribution_sample;
	Eigen::VectorXd alpha_vec = Eigen::VectorXd::Map(pDistribution_sample->alpha.data(), pDistribution_sample->alpha.size());
	r = (alpha_vec.adjoint() * x).row(0);
}

void cudaKernel_multinomial::log_likelihood_v2(
	double* d_r,
	int r_offset,
	int* d_indices,
	int indicesSize,
	int dim,
	const distribution_sample* distribution_sample,
	hipStream_t& stream,
	int deviceId)
{
	const multinomial_dist* pDistribution_sample = (multinomial_dist*)distribution_sample;
	Eigen::VectorXd alpha_vec = Eigen::VectorXd::Map(pDistribution_sample->alpha.data(), pDistribution_sample->alpha.size());

//TODO
//	Copy alpha_vec to d_alpha_vec
//	Do this is cuda:
//	r = (alpha_vec.adjoint() * x).row(0);
}

void cudaKernel_multinomial::log_likelihood_v3(
	double* d_r,
	int dim,
	double weight,
	const distribution_sample* distribution_sample,
	hipStream_t& stream,
	int deviceId)
{
	//TODO

}

#endif //CudaKernel_multinomial_CU