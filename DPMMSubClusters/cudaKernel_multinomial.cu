#include "hip/hip_runtime.h"
#ifndef CudaKernel_multinomial_CU
#define CudaKernel_multinomial_CU

#include "cudaKernel_multinomial.cuh"


__global__ void get_first_row_multiple_alpha_with_x(double* d_points, int dim, int* d_indices, int indicesSize, double* d_alpha, double* d_r)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < indicesSize)
	{
		double sum = 0;
		for (int i = 0; i < dim; i++)
		{
			sum += d_alpha[i] * d_points[IDX2C(i, d_indices[idx], dim)];
		}
		d_r[idx] = sum;
	}
}


__global__ void get_first_row_multiple_alpha_with_x_all(double* d_points, int dim, int indicesSize, double* d_alpha, double* d_r)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < indicesSize)
	{
		double sum = 0;
		for (int i = 0; i < dim; i++)
		{
			sum += d_alpha[i] * d_points[IDX2C(i, idx, dim)];
		}
		d_r[idx] = sum;
	}
}

void cudaKernel_multinomial::log_likelihood(Eigen::VectorXd& r, const Eigen::MatrixXd& x, const distribution_sample* distribution_sample)
{
	const multinomial_dist* pDistribution_sample = (multinomial_dist*)distribution_sample;
	Eigen::VectorXd alpha_vec = Eigen::VectorXd::Map(pDistribution_sample->alpha.data(), pDistribution_sample->alpha.size());
	r = (alpha_vec.adjoint() * x).row(0);
}

void cudaKernel_multinomial::log_likelihood_v2(
	double* d_r,
	int r_offset,
	int* d_indices,
	int indicesSize,
	int dim,
	const distribution_sample* distribution_sample,
	hipStream_t& stream,
	int deviceId)
{
	double* d_alpha;
	const multinomial_dist* pDistribution_sample = (multinomial_dist*)distribution_sample;
	runCuda(hipMallocAsync((void**)&d_alpha, sizeof(double) * pDistribution_sample->alpha.size(), stream));
	runCuda(hipMemcpyAsync(d_alpha, pDistribution_sample->alpha.data(), sizeof(double) * pDistribution_sample->alpha.size(), hipMemcpyHostToDevice, stream));
	
	dim3 blocks_size = dim3(indicesSize / threads.x + 1);

	get_first_row_multiple_alpha_with_x << <blocks_size, threads, 0, stream >> > (gpuCapabilities[deviceId].d_points, dim, d_indices, indicesSize, d_alpha, d_r+r_offset);

	runCuda(hipFreeAsync(d_alpha, stream));

//TODO
//	Copy alpha_vec to d_alpha_vec
//	Do this is cuda:
//	r = (alpha_vec.adjoint() * x).row(0);
}

void cudaKernel_multinomial::log_likelihood_v3(
	double* d_r,
	int dim,
	double weight,
	const distribution_sample* distribution_sample,
	hipStream_t& stream,
	int deviceId)
{
	double* d_alpha;
	const multinomial_dist* pDistribution_sample = (multinomial_dist*)distribution_sample;
	runCuda(hipMallocAsync((void**)&d_alpha, sizeof(double) * pDistribution_sample->alpha.size(), stream));
	runCuda(hipMemcpyAsync(d_alpha, pDistribution_sample->alpha.data(), sizeof(double) * pDistribution_sample->alpha.size(), hipMemcpyHostToDevice, stream));

	get_first_row_multiple_alpha_with_x_all << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].d_points, dim, numLabels, d_alpha, d_r);

	runCuda(hipFreeAsync(d_alpha, stream));

}

#endif //CudaKernel_multinomial_CU