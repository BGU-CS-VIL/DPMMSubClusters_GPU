#include "hip/hip_runtime.h"
#ifndef CudaKernel_CU
#define CudaKernel_CU

#include <omp.h>

#include <hip/hip_runtime.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<time.h>
#include "cudaKernel.cuh"
#include "check_time.h"


// function to define seed
__global__ void initCurand(hiprandState *state, unsigned long long seed, int maxIdx) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		hiprand_init(seed, idx, 0, &state[idx]);
	}
}

__device__ void sample_by_probability(hiprandState* state, double* weight, int numClusters, int rows, int idx, int* index,
	double* y, int* a, int* b)
{
	int i;
	int j;
	int k;
	int n = numClusters;

	a[idx] = 0;
	for (i = 1; i <= n; i++)
	{
		a[i * rows + idx] = i;
	}
	a[(n + 1) * rows + idx] = n + 1;

	b[idx] = 0;
	for (i = 1; i <= n; i++)
	{
		b[i * rows + idx] = i;
	}
	b[(n + 1) * rows + idx] = n + 1;

	//  Copy Y from X.
	//  Scale the probability vector and set sentinel values at the ends.

	y[idx] = 0.0;
	for (i = 1; i <= n; i++)
	{
		y[i * rows + idx] = weight[(i - 1) * rows + idx] * (double)(n);
	}
	y[(n + 1) * rows + idx] = 2.0;

	i = 0;
	j = n + 1;
	for (; ; )
	{

		//  Find i so Y[B[i]] needs more.

		do
		{
			i++;
		} while (y[b[i * rows + idx] * rows + idx] < 1.0);

		//	  Find j so Y[B[j]] wants less.

		do
		{
			j--;
		} while (1.0 <= y[b[j * rows + idx] * rows + idx]);

		if (j <= i)
		{
			break;
		}

		// Swap B[i] and B[j].

		k = b[i * rows + idx];
		b[i * rows + idx] = b[j * rows + idx];
		b[j * rows + idx] = k;
	}

	i = j;
	j++;

	while (0 < i)
	{

		//  Find J such that Y[B[j]] needs more.

		while (y[b[j * rows + idx] * rows + idx] <= 1.0)
		{
			j++;
		}

		//  Meanwhile, Y[B[i]] wants less.

		if (n < j)
		{
			break;
		}

		//  B[i] will donate to B[j] to fix up.

		y[b[j * rows + idx] * rows + idx] = y[b[j * rows + idx] * rows + idx] - (1.0 - y[b[i * rows + idx] * rows + idx]);
		a[b[i * rows + idx] * rows + idx] = b[j * rows + idx];

		// Y[B[j]] now wants less so readjust ordering.

		if (y[b[j * rows + idx] * rows + idx] < 1.0)
		{
			k = b[i * rows + idx];
			b[i * rows + idx] = b[j * rows + idx];
			b[j * rows + idx] = k;
			j++;
		}
		else
		{
			i--;
		}
	}

	double r;

	//  Let i = random uniform integer from {1,2,...N};

	i = 1 + (int)(n * hiprand_uniform(state));
	r = hiprand_uniform(state);

	if (y[i * rows + idx] < r)
	{
		i = a[i * rows + idx];

	}
	*index = i;
}

__global__ void sample_log_cat_array_all(hiprandState *state, int *dev_sample, int maxIdx, int numClusters, double *d_log_likelihood_array, double *y, int *a, int *b)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_by_probability(&state[idx], d_log_likelihood_array, numClusters, maxIdx, idx, &dev_sample[idx], y, a, b);
	}
}

__global__ void sample_log_cat_array_sub_cluster_all(hiprandState *state, int *dev_sample, int maxIdx, int num, double *d_log_likelihood_array, int *indices, double *y, int *a, int *b)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_by_probability(&state[indices[idx]], d_log_likelihood_array, num, maxIdx, idx, &dev_sample[indices[idx]], y, a, b);
	}
}

__device__ void sample_sub_label(hiprandState *state, int *d_label)
{
	*d_label = ((int)(hiprand_uniform(state) * 2)) % 2 + 1;
}

__global__ void sample_sub_labels_all(hiprandState *state, int *d_labels, int maxIdx)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_sub_label(&state[idx], &d_labels[idx]);
	}
}

__device__ void sample_label(hiprandState *state, int *d_label, int initial_clusters, double outlier_mod)
{
	*d_label = ((int)(hiprand_uniform(state)*initial_clusters)) % initial_clusters + 1 + ((outlier_mod > 0) ? 1 : 0);
}

__global__ void sample_labels_all(hiprandState *state, int *d_labels, int maxIdx, int initial_clusters, double outlier_mod)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_label(&state[idx], &d_labels[idx], initial_clusters, outlier_mod);
	}
}

__device__ void remove_empty_clusters_worker(int *d_label, int limit)
{
	if (*d_label > limit)
	{
		*d_label -= 1;
	}
}

__global__ void remove_empty_clusters_worker_all(int *d_labels, int maxIdx, int limit)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		remove_empty_clusters_worker(&d_labels[idx], limit);
	}
}

__global__ void find_indices(int *d_labels, int maxIdx, int label, int *d_indices, int *d_indicesSize)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		if (d_labels[idx] == label)
		{
			int curIndex = atomicAdd(d_indicesSize, 1);
			d_indices[curIndex] = idx;
		}
	}
}

__device__ void split_cluster_local_worker(hiprandState *state, int *d_labels, int *d_sub_label, int index, int newIndex)
{
	if (*d_labels == index + 1)
	{
		if (*d_sub_label == 2)
		{
			*d_labels = newIndex + 1;
		}
		*d_sub_label = hiprand(state) % 2 + 1;
	}
}

__global__ void split_cluster_local_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index, int newIndex)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		split_cluster_local_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index, newIndex);
	}
}

__device__ void merge_clusters_worker(hiprandState *state, int *d_labels, int *d_sub_labels, int index, int newIndex)
{
	if (*d_labels == index + 1)
	{
		*d_sub_labels = 1;
	}
	if (*d_labels == newIndex + 1)
	{
		*d_sub_labels = 2;
	}
	if (*d_labels == newIndex + 1)
	{
		*d_labels = index + 1;
	}
}

__global__ void merge_clusters_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index, int newIndex)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		merge_clusters_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index, newIndex);
	}
}

__device__ void reset_bad_clusters_worker(hiprandState *state, int *d_labels, int *d_sub_labels, int index)
{
	if (*d_labels == index + 1)
	{
		*d_sub_labels = hiprand(state) % 2 + 1;
	}
}

__global__ void reset_bad_clusters_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		reset_bad_clusters_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index);
	}
}

__global__ void get_sub_labels_count_all(int *d_sub_labels, int maxIdx, int *l, int *r)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		if (d_sub_labels[idx] == 1)
		{
			atomicAdd(l, 1);
		}
		else if (d_sub_labels[idx] == 2)
		{
			atomicAdd(r, 1);
		}
	}
}

__global__ void create_suff_stats_dict_worker_all(
	int* d_sub_labels,
	int maxIdx,
	int *d_indices,
	int *d_indicesSize,
	double *group_pts,
	int group_pts_rows,
	double *pts,
	double *pts1,
	double *pts2,
	int *d_j1,
	int *d_j2)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < *d_indicesSize)
	{
		if (d_sub_labels[d_indices[idx]] == 1)
		{
			int curj1 = atomicAdd(d_j1, 1);
			for (int i = 0; i < group_pts_rows; i++)
			{
				double pt = group_pts[IDX2C(i, d_indices[idx], group_pts_rows)];
				pts[IDX2C(i, idx, group_pts_rows)] = pt;
				pts1[IDX2C(i, curj1, group_pts_rows)] = pt;
			}
		}
		else if (d_sub_labels[d_indices[idx]] == 2)
		{
			int curj2 = atomicAdd(d_j2, 1);
			for (int i = 0; i < group_pts_rows; i++)
			{
				double pt = group_pts[IDX2C(i, d_indices[idx], group_pts_rows)];
				pts[IDX2C(i, idx, group_pts_rows)] = pt;
				pts2[IDX2C(i, curj2, group_pts_rows)] = pt;
			}
		}
	}
}

__global__ void dcolwise_dot_all_kernel(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, int r_offset)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double sum = 0;
		for (int i = 0; i < rows; i++)
		{
			sum += d_a[idx * rows + i] * d_b[idx * rows + i];
		}

		d_r[idx + r_offset] = scalar - sum / 2;
	}
}

__global__ void dcolwise_dot_with_log_kernel(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, double weight)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double sum = 0;
		for (int i = 0; i < rows; i++)
		{
			sum += d_a[idx * rows + i] * d_b[idx * rows + i];
		}

		d_r[idx] = scalar - sum / 2 + __logf(weight);
	}
}

__global__ void build_log_likelihood_array_sub_cluster_kernel(int maxIdx, double* d_r, int r_offset, double* d_lr_weights)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double maxRow;
		double sum;

		d_r[idx] += __logf(d_lr_weights[0]);
		d_r[idx + r_offset] += __logf(d_lr_weights[1]);
		maxRow = fmax(d_r[idx], d_r[idx + r_offset]);

		d_r[idx] = __expf(d_r[idx] - maxRow);
		d_r[idx + r_offset] = __expf(d_r[idx + r_offset] - maxRow);
		sum = d_r[idx] + d_r[idx + r_offset];

		d_r[idx] = d_r[idx] / sum;
		d_r[idx + r_offset] = d_r[idx + r_offset] / sum;
	}
}

__global__ void build_log_likelihood_array_kernel(int maxIdx, double* d_log_likelihood_array, int rows, int cols)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double maxRow = 0;
		bool first = true;
		for (int j = 0; j < cols; j++)
		{
			if (first || d_log_likelihood_array[IDX2C(idx, j, rows)] > maxRow)
			{
				first = false;
				maxRow = d_log_likelihood_array[IDX2C(idx, j, rows)];
			}
		}

		double sum = 0;
		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = exp(d_log_likelihood_array[IDX2C(idx, j, rows)] - maxRow);
			sum += d_log_likelihood_array[IDX2C(idx, j, rows)];
		}

		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = d_log_likelihood_array[IDX2C(idx, j, rows)] / sum;
		}
	}
}

__global__ void update_labels_by_max_index_kernel(double* parr, int* d_labels, int maxIdx, int dim)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		double max = 0;
		bool first = true;
		int max_index = 0;
		for (int i = 0; i < dim; ++i)
		{
			if (first || max < parr[IDX2C(idx, i, maxIdx)])
			{
				first = false;
				max = parr[IDX2C(idx, i, maxIdx)];
				max_index = i;
			}
		}
		d_labels[idx] = max_index + 1;
	}
}

__global__ void gpu_matrix_mult(double* a, double* b, double* c, int m, int n, int k)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	double sum = 0;

	if (col < k && row < m)
	{
		for (int i = 0; i < n; i++)
		{
			sum += a[IDX2C(row, i, m)] * b[IDX2C(i, col, n)];
		}
		c[IDX2C(row, col, m)] = sum;
	}
}

__global__ void sum_rowwise_kernel(double* d_A, double* d_B, int rows, int cols)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < rows)
	{
		double sum = 0;
		for (int j = 0; j < cols; j++)
		{
			sum += d_A[IDX2C(idx, j, rows)];
		}
		d_B[idx] = sum;
	}
}

void cudaKernel::init(int numLabelsIn, MatrixXd &points, unsigned long long seed)
{
	printf("Init cuda\n");

	int numGPU;
	int driverVersion = 0, runtimeVersion = 0;

	lastDevice = 0;

	runCuda(hipGetDeviceCount(&numGPU));
	numGPU = 1;

	printf("Number of GPUs: %i\n", numGPU);
	printf("number of host CPUs:\t%d\n", omp_get_num_procs());

	for (int i = 0; i < numGPU; i++)
	{
		hipSetDevice(i);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, i);

		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);

		printf("\nDevice %d: \"%s\"\n", i, deviceProp.name);
		printf("CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
		printf("CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

		int* dummy;
		hipStream_t stream;

		hipStreamCreate(&stream);

		hipError_t err = hipMallocAsync((void**)&dummy, sizeof(int), stream);
		if (hipSuccess != err)
		{
			hipGetLastError();
			printf("Not capable device. Can't perform asynchronous memory allocation.\n");
		}
		else
		{
			gpuCapabilities[i] = gpuCapability();
		}
		hipStreamSynchronize(stream);
		hipStreamDestroy(stream);
	}
		
	printf("\nNumber of GPUs that will be used: %i\n\n", (int)gpuCapabilities.size());

	numLabels = numLabelsIn;
	threads = dim3(512);
	blocks = dim3(numLabels / threads.x + 1);

	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		hipSetDevice(iter->first);
		runCuda(hipDeviceReset());
		runCuda(hipMalloc((void**)&(iter->second.devState), numLabels * sizeof(hiprandState)));
		initCurand << <blocks, threads >> > (iter->second.devState, seed, numLabels);
		runCuda(hipPeekAtLastError());

		runCuda(hipMalloc((void**)&(iter->second.d_labels), numLabels * sizeof(int)));
		runCuda(hipMalloc((void**)&(iter->second.d_sub_labels), numLabels * sizeof(int)));
		runCuda(hipMalloc((void**)&(iter->second.d_points), points.size() * sizeof(double)));
		runCuda(hipMemcpy(iter->second.d_points, points.data(), points.size() * sizeof(double), hipMemcpyHostToDevice));
		iter->second.pointsRows = (int)points.rows();
		iter->second.pointsCols = (int)points.cols();
	}

	if (gpuCapabilities.size() > 0)
	{
		hipSetDevice(gpuCapabilities.begin()->first);
	}
}

void cudaKernel::release()
{
	printf("Release cuda\n");

	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		if (iter->second.devState != NULL)
		{
			runCuda(hipFree(iter->second.devState));
		}

		if (iter->second.d_labels != NULL)
		{
			runCuda(hipFree(iter->second.d_labels));
		}

		if (iter->second.d_sub_labels != NULL)
		{
			runCuda(hipFree(iter->second.d_sub_labels));
		}

		if (iter->second.d_points != NULL)
		{
			runCuda(hipFree(iter->second.d_points));
		}
	}
}

int cudaKernel::peak_first_device()
{
	int result;
	//++lastDevice;
	//if (lastDevice >= gpuCapabilities.size())
	//	lastDevice = 0;

	//int i = 0;
	//for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); i <= lastDevice && iter != gpuCapabilities.end(); iter++, ++i)
	//{
	//	result = iter->first;
	//}
	//hipSetDevice(result);
	//return result;
	result = gpuCapabilities.begin()->first;
	hipSetDevice(result);
	return result;
}

int cudaKernel::peak_any_device()
{
	int result;
	++lastDevice;
	if (lastDevice >= gpuCapabilities.size())
		lastDevice = 0;

	int i = 0;
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); i <= lastDevice && iter != gpuCapabilities.end(); iter++, ++i)
	{
		result = iter->first;
	}
	hipSetDevice(result);
	return result;
}

void cudaKernel::sample_log_cat_array_sub_cluster(
	double* d_r,
	int r_offset,
	int* d_indices,
	int indicesSize,
	double* d_lr_weights,
	hipStream_t& stream,
	int deviceId)
{
	double* d_y;
	int* d_a;
	int* d_b;

	runCuda(hipMallocAsync((void**)&d_y, sizeof(double) * indicesSize * (2 + 2), stream));
	runCuda(hipMallocAsync((void**)&d_a, sizeof(int) * indicesSize * (2 + 2), stream));
	runCuda(hipMallocAsync((void**)&d_b, sizeof(int) * indicesSize * (2 + 2), stream));

	build_log_likelihood_array_sub_cluster_kernel << <blocks, threads, 0, stream >> > (indicesSize, d_r, r_offset, d_lr_weights);
	runCuda(hipPeekAtLastError());

	//TODO - Can we remove d_y, d_a, d_b?
	sample_log_cat_array_sub_cluster_all << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_sub_labels, indicesSize, 2, d_r, d_indices, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());

	runCuda(hipFreeAsync(d_y, stream));
	runCuda(hipFreeAsync(d_a, stream));
	runCuda(hipFreeAsync(d_b, stream));

	update_sub_labels_to_all_other_devices(deviceId, stream);
}

void cudaKernel::sample_log_cat_array(
	double* d_r,
	int dim,
	hipStream_t& stream,
	int deviceId)
{
	double* d_y;
	int* d_a;
	int* d_b;

	runCuda(hipMallocAsync((void**)&d_y, sizeof(double) * numLabels * (dim + 2), stream));
	runCuda(hipMallocAsync((void**)&d_a, sizeof(int) * numLabels * (dim + 2), stream));
	runCuda(hipMallocAsync((void**)&d_b, sizeof(int) * numLabels * (dim + 2), stream));
	build_log_likelihood_array_kernel << <blocks, threads, 0, stream >> > (numLabels, d_r, numLabels, dim);
	runCuda(hipPeekAtLastError());

	sample_log_cat_array_all << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, dim, d_r, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());

	runCuda(hipFreeAsync(d_y, stream));
	runCuda(hipFreeAsync(d_a, stream));
	runCuda(hipFreeAsync(d_b, stream));

	update_labels_to_all_other_devices(deviceId, stream);
}

void cudaKernel::sample_sub_clusters_worker(LabelType label, int* d_indices, int &indicesSize, hipStream_t& stream, int deviceId)
{
	int* d_indicesSize;
	runCuda(hipMallocAsync(&d_indicesSize, sizeof(int), stream));
	runCuda(hipMemsetAsync(d_indicesSize, 0, sizeof(int), stream));

	find_indices << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].d_labels, numLabels, label, d_indices, d_indicesSize);
	runCuda(hipMemcpyAsync(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost, stream));

	runCuda(hipPeekAtLastError());
}

void cudaKernel::create_sufficient_statistics(
	LabelType label,
	LabelType& indicesSize,
	const std::shared_ptr<hyperparams>& hyperParams,
	const std::shared_ptr<hyperparams>& posterior,
	std::shared_ptr<thin_suff_stats>& tss)
{
	CHECK_TIME("cudaKernel::create_sufficient_statistics");

	int deviceId = peak_first_device();
	hipStream_t stream;
	runCuda(hipStreamCreate(&stream));
	int pointsRows = gpuCapabilities[deviceId].pointsRows;
	int* d_indices;
	runCuda(hipMallocAsync((void**)&d_indices, sizeof(int) * numLabels, stream));

	int* d_indicesSize;
	runCuda(hipMallocAsync(&d_indicesSize, sizeof(int), stream));
	runCuda(hipMemsetAsync(d_indicesSize, 0, sizeof(int), stream));

	find_indices << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].d_labels, numLabels, label, d_indices, d_indicesSize);
	runCuda(hipPeekAtLastError());
	runCuda(hipMemcpyAsync(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost));

	double* d_pts;
	runCuda(hipMallocAsync((void**)&d_pts, sizeof(double) * pointsRows * indicesSize, stream));

	double* d_pts1;
	runCuda(hipMallocAsync((void**)&d_pts1, sizeof(double) * pointsRows * indicesSize, stream));

	double* d_pts2;
	runCuda(hipMallocAsync((void**)&d_pts2, sizeof(double) * pointsRows * indicesSize, stream));

	int* d_j1;
	int* d_j2;
	runCuda(hipMallocAsync(&d_j1, sizeof(int), stream));
	runCuda(hipMemsetAsync(d_j1, 0, sizeof(int), stream));
	runCuda(hipMallocAsync(&d_j2, sizeof(int), stream));
	runCuda(hipMemsetAsync(d_j2, 0, sizeof(int), stream));

	dim3 blocks_size = dim3(indicesSize / threads.x + 1);
	create_suff_stats_dict_worker_all << <blocks_size, threads, 0, stream >> > (
		gpuCapabilities[deviceId].d_sub_labels,
		numLabels,
		d_indices,
		d_indicesSize,
		gpuCapabilities[deviceId].d_points,
		pointsRows,
		d_pts,
		d_pts1,
		d_pts2,
		d_j1,
		d_j2);
	runCuda(hipPeekAtLastError());

	int j1;
	int j2;
	runCuda(hipMemcpyAsync(&j1, d_j1, sizeof(int), hipMemcpyDeviceToHost, stream));
	runCuda(hipMemcpyAsync(&j2, d_j2, sizeof(int), hipMemcpyDeviceToHost, stream));

	runCuda(hipStreamSynchronize(stream));
	do_create_sufficient_statistics(d_pts1, pointsRows, j1, hyperParams, posterior, stream, tss->l_suff);
	do_create_sufficient_statistics(d_pts2, pointsRows, j2, hyperParams, posterior, stream, tss->r_suff);
	do_create_sufficient_statistics(d_pts, pointsRows, indicesSize, hyperParams, posterior, stream, tss->cluster_suff);

	runCuda(hipFreeAsync(d_j1, stream));
	runCuda(hipFreeAsync(d_j2, stream));
	runCuda(hipFreeAsync(d_indicesSize, stream));
	runCuda(hipFreeAsync(d_pts, stream));
	runCuda(hipFreeAsync(d_pts1, stream));
	runCuda(hipFreeAsync(d_pts2, stream));
	runCuda(hipStreamSynchronize(stream));
	runCuda(hipStreamDestroy(stream));
}

// A -> (N x M) 
void cudaKernel::multiplie_matrix_by_transpose(double* d_A, double* d_B, int N, int M)
{
	hipblasHandle_t handle;
	runCuda(hipblasCreate(&handle));
	double alpha = 1.0;
	double beta = 0.0;
	runCuda(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, M, &alpha, d_A, N, d_A, N, &beta, d_B, N));

	runCuda(hipblasDestroy(handle));
}

void cudaKernel::create_suff_stats_dict_worker(
	LabelType label,
	LabelType& indicesSize,
	Eigen::MatrixXd& pts,
	Eigen::MatrixXd& pts1,
	Eigen::MatrixXd& pts2)
{
	CHECK_TIME("cudaKernel::create_suff_stats_dict_worker");
	int deviceId = peak_first_device();
	int pointsRows = gpuCapabilities[deviceId].pointsRows;
	int* d_indices;
	runCuda(hipMalloc((void**)&d_indices, sizeof(int) * numLabels));

	int* d_indicesSize;
	runCuda(hipMalloc(&d_indicesSize, sizeof(int)));
	runCuda(hipMemset(d_indicesSize, 0, sizeof(int)));

	find_indices << <blocks, threads >> > (gpuCapabilities[deviceId].d_labels, numLabels, label, d_indices, d_indicesSize);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
	runCuda(hipMemcpy(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost));

	double* d_pts;
	runCuda(hipMalloc((void**)&d_pts, sizeof(double) * pointsRows * indicesSize));

	double* d_pts1;
	runCuda(hipMalloc((void**)&d_pts1, sizeof(double) * pointsRows * indicesSize));

	double* d_pts2;
	runCuda(hipMalloc((void**)&d_pts2, sizeof(double) * pointsRows * indicesSize));

	int* d_j1;
	int* d_j2;
	runCuda(hipMalloc(&d_j1, sizeof(int)));
	runCuda(hipMemset(d_j1, 0, sizeof(int)));
	runCuda(hipMalloc(&d_j2, sizeof(int)));
	runCuda(hipMemset(d_j2, 0, sizeof(int)));

	dim3 blocks_size = dim3(indicesSize / threads.x + 1);
	create_suff_stats_dict_worker_all << <blocks_size, threads >> > (
		gpuCapabilities[deviceId].d_sub_labels,
		numLabels,
		d_indices,
		d_indicesSize,
		gpuCapabilities[deviceId].d_points,
		pointsRows,
		d_pts,
		d_pts1,
		d_pts2,
		d_j1,
		d_j2);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	int j1;
	int j2;
	runCuda(hipMemcpy(&j1, d_j1, sizeof(int), hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(&j2, d_j2, sizeof(int), hipMemcpyDeviceToHost));

	pts.resize(pointsRows, indicesSize);
	pts1.resize(pointsRows, j1);
	pts2.resize(pointsRows, j2);

	runCuda(hipMemcpy(pts.data(), d_pts, sizeof(double) * pointsRows * indicesSize, hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(pts1.data(), d_pts1, sizeof(double) * pointsRows * j1, hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(pts2.data(), d_pts2, sizeof(double) * pointsRows * j2, hipMemcpyDeviceToHost));

	runCuda(hipFree(d_j1));
	runCuda(hipFree(d_j2));
	runCuda(hipFree(d_indicesSize));
	runCuda(hipFree(d_pts));
	runCuda(hipFree(d_pts1));
	runCuda(hipFree(d_pts2));
}

void cudaKernel::sample_sub_labels()
{
	int deviceId = peak_first_device();
	sample_sub_labels_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_sub_labels, numLabels);
	runCuda(hipPeekAtLastError());

	update_sub_labels_to_all_other_devices(deviceId);
}

void cudaKernel::sample_labels(int initial_clusters, double outlier_mod)
{
	int deviceId = peak_first_device();
	sample_labels_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, initial_clusters, outlier_mod);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	update_labels_to_all_other_devices(deviceId);
}

void cudaKernel::get_sub_labels(LabelsType &subLabels)
{
	int deviceId = peak_first_device();
	int *h_subLabels;
	h_subLabels = (int*)malloc(numLabels * sizeof(int));
	runCuda(hipMemcpy(h_subLabels, gpuCapabilities[deviceId].d_sub_labels, numLabels * sizeof(int), hipMemcpyDeviceToHost));

	subLabels.resize(numLabels);
	for (size_t i = 0; i < numLabels; i++)
	{
		subLabels[i] = h_subLabels[i];
	}

	free(h_subLabels);
}

void cudaKernel::get_labels(LabelsType &labels)
{
	int deviceId = peak_first_device();
	int *h_labels;
	h_labels = (int*)malloc(numLabels * sizeof(int));
	runCuda(hipMemcpy(h_labels, gpuCapabilities[deviceId].d_labels, numLabels * sizeof(int), hipMemcpyDeviceToHost));

	labels.resize(numLabels);
	for (size_t i = 0; i < numLabels; i++)
	{
		labels[i] = h_labels[i];
	}

	free(h_labels);
}

void cudaKernel::update_labels(int *updateLabels, int numLabels, int deviceId)
{
	runCuda(hipMemcpy(gpuCapabilities[deviceId].d_labels, updateLabels, sizeof(int)*numLabels, hipMemcpyHostToDevice));
}

void cudaKernel::update_labels_to_all_other_devices(int srcDeviceId)
{
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		bool needToFree;
		device_to_device_copy(srcDeviceId, iter->first, numLabels, gpuCapabilities[srcDeviceId].d_labels, iter->second.d_labels, true, needToFree);
	}
}

void cudaKernel::update_labels_to_all_other_devices(int srcDeviceId, hipStream_t& stream)
{
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		bool needToFree;
		device_to_device_copy(srcDeviceId, iter->first, numLabels, gpuCapabilities[srcDeviceId].d_labels, iter->second.d_labels, true, needToFree, stream);
	}
}

void cudaKernel::update_sub_labels_to_all_other_devices(int srcDeviceId)
{
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		bool needToFree;
		device_to_device_copy(srcDeviceId, iter->first, numLabels, gpuCapabilities[srcDeviceId].d_sub_labels, iter->second.d_sub_labels, true, needToFree);
	}
}

void cudaKernel::update_sub_labels_to_all_other_devices(int srcDeviceId, hipStream_t& stream)
{
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		bool needToFree;
		device_to_device_copy(srcDeviceId, iter->first, numLabels, gpuCapabilities[srcDeviceId].d_sub_labels, iter->second.d_sub_labels, true, needToFree, stream);
	}
}

void cudaKernel::update_labels_by_max_index(double* parr, int dim, hipStream_t& stream, int deviceId)
{
	update_labels_by_max_index_kernel << <blocks, threads, 0, stream >> > (parr, gpuCapabilities[deviceId].d_labels, numLabels, dim);

	update_labels_to_all_other_devices(deviceId, stream);
}


void cudaKernel::remove_empty_clusters_worker(int limit)
{
	int deviceId = peak_first_device();
	remove_empty_clusters_worker_all << <blocks, threads >> > (gpuCapabilities[deviceId].d_labels, numLabels, limit);
	runCuda(hipPeekAtLastError());

	update_labels_to_all_other_devices(deviceId);
}

void cudaKernel::split_cluster_local_worker(LabelType index, LabelType newIndex)
{
	int deviceId = peak_first_device();
	split_cluster_local_worker_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, gpuCapabilities[deviceId].d_sub_labels, index, newIndex);
	runCuda(hipPeekAtLastError());

	update_labels_to_all_other_devices(deviceId);
	update_sub_labels_to_all_other_devices(deviceId);
}

void cudaKernel::merge_clusters_worker(LabelType index, LabelType newIndex)
{
	int deviceId = peak_first_device();
	merge_clusters_worker_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, gpuCapabilities[deviceId].d_sub_labels, index, newIndex);
	runCuda(hipPeekAtLastError());

	update_labels_to_all_other_devices(deviceId);
	update_sub_labels_to_all_other_devices(deviceId);
}

void cudaKernel::reset_bad_clusters_worker(LabelType index)
{
	int deviceId = peak_first_device();

	reset_bad_clusters_worker_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, gpuCapabilities[deviceId].d_sub_labels, index);
	runCuda(hipPeekAtLastError());

	update_sub_labels_to_all_other_devices(deviceId);
}

void cudaKernel::get_sub_labels_count(int &l, int &r)
{
	int deviceId = peak_first_device();
	int *d_l;
	runCuda(hipMalloc((void **)&d_l, sizeof(int)));
	runCuda(hipMemset(d_l, 0, sizeof(int)));
	
	int *d_r;
	runCuda(hipMalloc((void **)&d_r, sizeof(int)));
	runCuda(hipMemset(d_r, 0, sizeof(int)));

	get_sub_labels_count_all << <blocks, threads >> > (gpuCapabilities[deviceId].d_sub_labels, numLabels, d_l, d_r);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	runCuda(hipMemcpy(&l, d_l, sizeof(int), hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(&r, d_r, sizeof(int), hipMemcpyDeviceToHost));

	runCuda(hipFree(d_l));
	runCuda(hipFree(d_r));
}

// C(m,k) = A(m,n) * B(n,k)
void cudaKernel::naive_matrix_multiply(double* d_A, double* d_B, double* d_C, int m, int n, int k, hipStream_t& stream)
{
	const int BlockSize = 16;

	unsigned int grid_rows = (m + BlockSize - 1) / BlockSize;
	unsigned int grid_cols = (k + BlockSize - 1) / BlockSize;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BlockSize, BlockSize);

	if (k > 0)
	{
		gpu_matrix_mult << <dimGrid, dimBlock, 0, stream >> > (d_A, d_B, d_C, m, n, k);
		runCuda(hipPeekAtLastError());
	}
}

void cudaKernel::naive_matrix_multiply(double* d_A, double* d_B, double* d_C, int m, int n, int k)
{
	const int BlockSize = 16;

	unsigned int grid_rows = (m + BlockSize - 1) / BlockSize;
	unsigned int grid_cols = (k + BlockSize - 1) / BlockSize;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BlockSize, BlockSize);

	if (k > 0)
	{
		gpu_matrix_mult << <dimGrid, dimBlock >> > (d_A, d_B, d_C, m, n, k);
		runCuda(hipPeekAtLastError());
	}
}

void cudaKernel::dcolwise_dot_all_sub_labels(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, int r_offset, hipStream_t& stream)
{
	dcolwise_dot_all_kernel << <blocks, threads, 0, stream >> > (maxIdx, rows, d_a, d_b, scalar, d_r, r_offset);
}

void cudaKernel::dcolwise_dot_all_labels(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, double weight, hipStream_t& stream)
{
	dcolwise_dot_with_log_kernel << <blocks, threads, 0, stream >> > (maxIdx, rows, d_a, d_b, scalar, d_r, weight);
}

typedef struct
{
	//Stream for asynchronous command execution
	hipStream_t stream;

	int* d_indices;
	int indicesSize;
	double* d_r;
	double* d_lr_weights;
	int deviceId;
} subclusters_labels_plan;

void cudaKernel::create_subclusters_labels(int numClusters, std::vector<std::shared_ptr<thin_cluster_params>>& cluster_params, int dim)
{
//	omp_set_num_threads(20);
//	#pragma omp parallel
	{
//		unsigned int i = omp_get_thread_num();

//		printf("**** i=%d\n", i);
	}

	subclusters_labels_plan* plan = new subclusters_labels_plan[numClusters];

	//Allocate memory for all streams
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel
	for (int i = 0; i < numClusters; i++)
	{
//		unsigned int i = omp_get_thread_num();

		plan[i].deviceId = peak_any_device();
		runCuda(hipStreamCreate(&(plan[i].stream)));
		runCuda(hipMallocAsync((void**)&(plan[i].d_indices), sizeof(int) * numLabels, plan[i].stream));

		//Both
		runCuda(hipMallocAsync((void**)&(plan[i].d_lr_weights), sizeof(double) * cluster_params[i]->lr_weights.size(), plan[i].stream));
		runCuda(hipMemcpyAsync(plan[i].d_lr_weights, cluster_params[i]->lr_weights.data(), sizeof(double) * cluster_params[i]->lr_weights.size(), hipMemcpyHostToDevice, plan[i].stream));
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel
	for (int i = 0; i < numClusters; i++)
	{
//		unsigned int i = omp_get_thread_num();

		hipSetDevice(plan[i].deviceId);
		//Find indices
		//Can be used on any GPU
		sample_sub_clusters_worker(i + 1, plan[i].d_indices, plan[i].indicesSize, plan[i].stream, plan[i].deviceId);

		//Return the likelihood in r vector.
		//Can be used on any GPU
		runCuda(hipMallocAsync((void**)&(plan[i].d_r), sizeof(double) * plan[i].indicesSize * 2, plan[i].stream));

		log_likelihood_sub_labels(plan[i].d_r, 0, plan[i].d_indices, plan[i].indicesSize, dim, cluster_params[i]->l_dist, plan[i].stream, plan[i].deviceId);
		log_likelihood_sub_labels(plan[i].d_r, plan[i].indicesSize, plan[i].d_indices, plan[i].indicesSize, dim, cluster_params[i]->r_dist, plan[i].stream, plan[i].deviceId);
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipStreamSynchronize(plan[i].stream));
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		//run on one GPU (plan[0].deviceId) - maybe could be optimized
		hipSetDevice(plan[0].deviceId);
		
		int* d_indices;
		bool needToFree_d_indices;
		device_to_device_copy(plan[i].deviceId, plan[0].deviceId, plan[i].indicesSize, plan[i].d_indices, d_indices, false, needToFree_d_indices);

		double* d_lr_weights;
		bool needToFree_d_lr_weights;
		device_to_device_copy(plan[i].deviceId, plan[0].deviceId, (int)cluster_params[i]->lr_weights.size(), plan[i].d_lr_weights, d_lr_weights, false, needToFree_d_lr_weights);

		double* d_r;
		bool needToFree_d_r;
		device_to_device_copy(plan[i].deviceId, plan[0].deviceId, plan[i].indicesSize * 2, plan[i].d_r, d_r, false, needToFree_d_r);

		sample_log_cat_array_sub_cluster(d_r, plan[i].indicesSize, d_indices, plan[i].indicesSize, d_lr_weights, plan[0].stream, plan[0].deviceId);

		if (needToFree_d_indices)
		{
			runCuda(hipFreeAsync(d_indices, plan[0].stream));
		}
		if (needToFree_d_lr_weights)
		{
			runCuda(hipFreeAsync(d_lr_weights, plan[0].stream));
		}
		if (needToFree_d_r)
		{
			runCuda(hipFreeAsync(d_r, plan[0].stream));
		}
	}

	//Wait for all operations to finish
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipStreamSynchronize(plan[i].stream));
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipFreeAsync(plan[i].d_indices, plan[i].stream));
		runCuda(hipFreeAsync(plan[i].d_r, plan[i].stream));

		runCuda(hipFreeAsync(plan[i].d_lr_weights, plan[i].stream));

		runCuda(hipStreamDestroy(plan[i].stream));
	}

	delete[]plan;
}

typedef struct
{
	//Stream for asynchronous command execution
	hipStream_t stream;
	int deviceId;
	double* d_r;
} clusters_labels_plan;

void cudaKernel::create_clusters_labels(int numClusters, std::vector<std::shared_ptr<thin_cluster_params>>& cluster_params, std::vector<double>& weights, bool bFinal)
{
	int masterDevice = -1;
	clusters_labels_plan* plan = new clusters_labels_plan[numClusters];

	//Allocate memory for all streams
	for (int i = 0; i < numClusters; i++)
	{
		plan[i].deviceId = peak_first_device();

		runCuda(hipStreamCreate(&(plan[i].stream)));

		if (i == 0)
		{
			masterDevice = plan[i].deviceId;
			runCuda(hipMalloc((void**)&(plan[i].d_r), sizeof(double) * numLabels * numClusters));
		}
		else if (masterDevice != plan[i].deviceId)
		{
			runCuda(hipMalloc((void**)&(plan[i].d_r), sizeof(double) * numLabels));
		}
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		if (masterDevice == plan[i].deviceId)
		{
			log_likelihood_labels(plan[0].d_r + i * numLabels, weights[i], cluster_params[i]->cluster_dist, plan[i].stream, plan[i].deviceId);
		}
		else
		{
			log_likelihood_labels(plan[i].d_r, weights[i], cluster_params[i]->cluster_dist, plan[i].stream, plan[i].deviceId);
		}
	}

	//Wait for all operations to finish
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipStreamSynchronize(plan[i].stream));
	}

	//Copy d_r from all streams and GPU to one
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		if (masterDevice != plan[i].deviceId)
		{
			//For improvement maybe worth to check if hipDeviceCanAccessPeer
			double* r = new double[numLabels];

			hipSetDevice(plan[i].deviceId);
			runCuda(hipMemcpy(r, plan[i].d_r, numLabels * sizeof(double), hipMemcpyDeviceToHost));

			hipSetDevice(plan[0].deviceId);
			runCuda(hipMemcpy(plan[0].d_r + i * numLabels, r, numLabels * sizeof(double), hipMemcpyHostToDevice));

			delete []r;
		}
	}

	hipSetDevice(plan[0].deviceId);
	if (bFinal)
	{
		update_labels_by_max_index(plan[0].d_r, numClusters, plan[0].stream, plan[0].deviceId);
	}
	else
	{
		sample_log_cat_array(plan[0].d_r, numClusters, plan[0].stream, plan[0].deviceId);
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipStreamSynchronize(plan[i].stream));
		runCuda(hipStreamDestroy(plan[i].stream));
		if (i == 0 || masterDevice != plan[i].deviceId)
		{
			runCuda(hipFree(plan[i].d_r));
		}
	}

	delete[]plan;
}

void cudaKernel::checkCUDAError(hipError_t err, const char* file, int line)
{
	if (hipSuccess != err)
	{
		printf("Cuda error: %s(%d):%s.\n", file, line, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void cudaKernel::checkCUDAError(hipblasStatus_t err, const char* file, int line)
{
	if (HIPBLAS_STATUS_SUCCESS != err)
	{
		printf("Cuda error: %s(%d):%d.\n", file, line, err);
		exit(EXIT_FAILURE);
	}
}

template<typename T>
void cudaKernel::device_to_device_copy(int srcDeviceId, int trgDeviceId, int dataSize, T* srcData, T*& trgData, bool alreadyAllocated, bool& needToFree, hipStream_t& stream)
{
	if (srcDeviceId != trgDeviceId)
	{
		runCuda(hipStreamSynchronize(stream));
	}
	device_to_device_copy(srcDeviceId, trgDeviceId, dataSize, srcData, trgData, alreadyAllocated, needToFree);
}

template<typename T>
void cudaKernel::device_to_device_copy(int srcDeviceId, int trgDeviceId, int dataSize, T* srcData, T* &trgData, bool alreadyAllocated, bool &needToFree)
{
	needToFree = false;

	if (srcDeviceId == trgDeviceId)
	{
		trgData = srcData;
	}
	else
	{
		T* data = new T[dataSize];

		hipSetDevice(srcDeviceId);
		runCuda(hipMemcpy(data, srcData, dataSize * sizeof(T), hipMemcpyDeviceToHost));

		hipSetDevice(trgDeviceId);

		if (!alreadyAllocated)
		{
			runCuda(hipMalloc((void**)&trgData, dataSize * sizeof(T)));
			needToFree = true;
		}

		runCuda(hipMemcpy(trgData, data, dataSize * sizeof(T), hipMemcpyHostToDevice));

		delete[]data;
	}
}

void cudaKernel::sum_rowwise(double* d_A, double* d_B, int rows, int cols, hipStream_t& stream)
{
	dim3 blocks_size = dim3(rows / threads.x + 1);

	sum_rowwise_kernel << <blocks_size, threads, 0, stream >> > (d_A, d_B, rows, cols);
	runCuda(hipPeekAtLastError());
}

#endif