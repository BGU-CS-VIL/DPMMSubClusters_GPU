#include "hip/hip_runtime.h"
#ifndef CudaKernel_CU
#define CudaKernel_CU

//#pragma warning( disable : 2886 )
//#pragma warning( disable : 2929)

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>

#include<time.h>
#include "cudaKernel.cuh"

// function to define seed
__global__ void initCurand(hiprandState *state, unsigned long long seed, int maxIdx) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		hiprand_init(seed, idx, 0, &state[idx]);
	}
}

//__device__ void sample_by_probability(hiprandState *state, double *weight, int numClusters, int rows, int idx, int *index)
//{
//	int cluster;
//	double u;
//	do 
//	{
//		cluster = (int)(hiprand_uniform(state) * (numClusters - 0.00001));
//		u = hiprand_uniform(state);
//	} while (u > weight[cluster*rows+idx]);
//	*index = cluster + 1;
//}

__device__ void sample_by_probability(hiprandState *state, double *weight, int numClusters, int rows, int idx, int *index,
									  double *y, int *a, int *b)
{
	int i;
	int j;
	int k;
	int n = numClusters;

	
//	if (n > 1)
	{
		a[idx] = 0;
		for (i = 1; i <= n; i++)
		{
			a[i*rows + idx] = i;
		}
		a[(n+1)*rows + idx] = n + 1;

		b[idx] = 0;
		for (i = 1; i <= n; i++)
		{
			b[i*rows + idx] = i;
		}
		b[(n+1)*rows + idx] = n + 1;
		/*
		  Copy Y from X.
		  Scale the probability vector and set sentinel values at the ends.
		*/
		y[idx] = 0.0;
		for (i = 1; i <= n; i++)
		{
			y[i*rows + idx] = weight[(i-1)*rows + idx] * (double)(n);
		}
		y[(n+1)*rows + idx] = 2.0;

		i = 0;
		j = n + 1;
		for (; ; )
		{
			/*
			  Find i so Y[B[i]] needs more.
			*/
			do
			{
				i++;
			} while (y[b[i*rows + idx]*rows + idx] < 1.0);
			/*
				  Find j so Y[B[j]] wants less.
				*/
			do
			{
				j--;
			} while (1.0 <= y[b[j*rows + idx]*rows + idx]);

			if (j <= i)
			{
				break;
			}
			/*
			  Swap B[i] and B[j].
			*/
			k = b[i*rows + idx];
			b[i*rows + idx] = b[j*rows + idx];
			b[j*rows + idx] = k;
		}

		i = j;
		j++;

		while (0 < i)
		{
			/*
			  Find J such that Y[B[j]] needs more.
			*/
			while (y[b[j*rows + idx]*rows + idx] <= 1.0)
			{
				j++;
			}
			/*
			  Meanwhile, Y[B[i]] wants less.
			*/
			if (n < j)
			{
				break;
			}
			/*
			  B[i] will donate to B[j] to fix up.
			*/
			y[b[j*rows + idx]*rows + idx] = y[b[j*rows + idx]*rows + idx] - (1.0 - y[b[i*rows + idx]*rows + idx]);
			a[b[i*rows + idx]*rows + idx] = b[j*rows + idx];
			/*
			  Y[B[j]] now wants less so readjust ordering.
			*/
			if (y[b[j*rows + idx]*rows + idx] < 1.0)
			{
				k = b[i*rows + idx];
				b[i*rows + idx] = b[j*rows + idx];
				b[j*rows + idx] = k;
				j++;
			}
			else
			{
				i--;
			}
		}

		double r;
		/*
		  Let i = random uniform integer from {1,2,...N};
		*/
		i = 1 + (int)(n * hiprand_uniform(state));
		//for (int j = 0; j < 2; j++)
		{
			r = hiprand_uniform(state);

			if (y[i*rows + idx] < r)
			{
				i = a[i*rows + idx];
//				break;
			}
		}
		*index = i;
//		*index = 1;
	}
//	else
	{
//		*index = 1;
	}
}

//__device__ void sample_by_probability(hiprandState *state, double *weight, int numClusters, int rows, int idx, int *index)
//{
//	//do
//	{
//		int cluster1 = 0;
//		int cluster2 = 0;
//		if (numClusters > 1)
//		{
//			cluster2 = 1;
//		}
//
//		if (weight[cluster1*rows + idx] > weight[cluster2*rows + idx])
//		{
//			*index = cluster1 + 1;
//		}
//		else
//		{
//			*index = cluster2 + 1;
//
//		}
//	}
//}

__global__ void sample_log_cat_array_all(hiprandState *state, int *dev_sample, int maxIdx, int numClusters, double *d_log_likelihood_array, double *y, int *a, int *b)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_by_probability(&state[idx], d_log_likelihood_array, numClusters, maxIdx, idx, &dev_sample[idx], y, a, b);
	}
}

__global__ void sample_log_cat_array_sub_cluster_all(hiprandState *state, int *dev_sample, int maxIdx, int num, double *d_log_likelihood_array, int *indices, double *y, int *a, int *b)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_by_probability(&state[indices[idx]], d_log_likelihood_array, num, maxIdx, idx, &dev_sample[indices[idx]], y, a, b);
	}
}

__device__ void sample_sub_label(hiprandState *state, int *d_label)
{
	*d_label = ((int)(hiprand_uniform(state) * 2)) % 2 + 1;
}

__global__ void sample_sub_labels_all(hiprandState *state, int *d_labels, int maxIdx)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_sub_label(&state[idx], &d_labels[idx]);
	}
}

__device__ void sample_label(hiprandState *state, int *d_label, int initial_clusters, double outlier_mod)
{
	*d_label = ((int)(hiprand_uniform(state)*initial_clusters)) % initial_clusters + 1 + ((outlier_mod > 0) ? 1 : 0);
}

__global__ void sample_labels_all(hiprandState *state, int *d_labels, int maxIdx, int initial_clusters, double outlier_mod)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_label(&state[idx], &d_labels[idx], initial_clusters, outlier_mod);
	}
}

__device__ void remove_empty_clusters_worker(int *d_label, int limit)
{
	if (*d_label > limit)
	{
		*d_label -= 1;
	}
}

__global__ void remove_empty_clusters_worker_all(int *d_labels, int maxIdx, int limit)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		remove_empty_clusters_worker(&d_labels[idx], limit);
	}
}

__global__ void find_indices(int *d_labels, int maxIdx, int label, int *d_indices, int *d_indicesSize)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		if (d_labels[idx] == label)
		{
			int curIndex = atomicAdd(d_indicesSize, 1);
			d_indices[curIndex] = idx;
		}
	}
}

__device__ void split_cluster_local_worker(hiprandState *state, int *d_labels, int *d_sub_label, int index, int newIndex)
{
	if (*d_labels == index + 1)
	{
		if (*d_sub_label == 2)
		{
			*d_labels = newIndex + 1;
		}
		*d_sub_label = hiprand(state) % 2 + 1;
	}
}

__global__ void split_cluster_local_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index, int newIndex)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		split_cluster_local_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index, newIndex);
	}
}

__device__ void merge_clusters_worker(hiprandState *state, int *d_labels, int *d_sub_labels, int index, int newIndex)
{
	if (*d_labels == index + 1)
	{
		*d_sub_labels = 1;
	}
	if (*d_labels == newIndex + 1)
	{
		*d_sub_labels = 2;
	}
	if (*d_labels == newIndex + 1)
	{
		*d_labels = index + 1;
	}
}

__global__ void merge_clusters_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index, int newIndex)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		merge_clusters_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index, newIndex);
	}
}

__device__ void reset_bad_clusters_worker(hiprandState *state, int *d_labels, int *d_sub_labels, int index)
{
	if (*d_labels == index + 1)
	{
		*d_sub_labels = hiprand(state) % 2 + 1;
	}
}

__global__ void reset_bad_clusters_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		reset_bad_clusters_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index);
	}
}

__global__ void get_sub_labels_count_all(int *d_sub_labels, int maxIdx, int *l, int *r)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		if (d_sub_labels[idx] == 1)
		{
			atomicAdd(l, 1);
		}
		else if (d_sub_labels[idx] == 2)
		{
			atomicAdd(r, 1);
		}
	}
}

__global__ void create_suff_stats_dict_worker_all(
	int* d_sub_labels,
	int maxIdx,
	int *d_indices,
	int *d_indicesSize,
	double *group_pts,
	int group_pts_rows,
	double *pts,
	double *pts1,
	double *pts2,
	int *d_j1,
	int *d_j2)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < *d_indicesSize)
	{
		if (d_sub_labels[d_indices[idx]] == 1)
		{
			int curj1 = atomicAdd(d_j1, 1);
			for (int i = 0; i < group_pts_rows; i++)
			{
				double pt = group_pts[IDX2C(i, d_indices[idx], group_pts_rows)];
				pts[IDX2C(i, idx, group_pts_rows)] = pt;
				pts1[IDX2C(i, curj1, group_pts_rows)] = pt;
			}
		}
		else if (d_sub_labels[d_indices[idx]] == 2)
		{
			int curj2 = atomicAdd(d_j2, 1);
			for (int i = 0; i < group_pts_rows; i++)
			{
				double pt = group_pts[IDX2C(i, d_indices[idx], group_pts_rows)];
				pts[IDX2C(i, idx, group_pts_rows)] = pt;
				pts2[IDX2C(i, curj2, group_pts_rows)] = pt;
			}
		}
	}
}

__global__ void dcolwise_dot_all_kernel(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double sum = 0;
		for (int i = 0; i < rows; i++)
		{
			sum += d_a[idx * rows + i] * d_b[idx * rows + i];
		}

		d_r[idx] = scalar - sum / 2;
	}
}

__global__ void build_log_likelihood_array_sub_cluster_kernel(int maxIdx, double* d_log_likelihood_array, double* d_lr_weights, int rows, int cols)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double maxRow = 0;
		bool first = true;
		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] += log(d_lr_weights[j]);
			if (first || d_log_likelihood_array[IDX2C(idx, j, rows)] > maxRow)
			{
				first = false;
				maxRow = d_log_likelihood_array[IDX2C(idx, j, rows)];
			}
		}

		double sum = 0;
		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = exp(d_log_likelihood_array[IDX2C(idx, j, rows)] - maxRow);
			sum += d_log_likelihood_array[IDX2C(idx, j, rows)];
		}

		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = d_log_likelihood_array[IDX2C(idx, j, rows)] / sum;
		}
	}
}

__global__ void build_log_likelihood_array_sub_cluster_kernel_v2(int maxIdx, double* d_rl, double* d_rr, double* d_lr_weights)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double maxRow;
		double sum;

		d_rl[idx] += log(d_lr_weights[0]);
		d_rr[idx] += log(d_lr_weights[1]);
		maxRow = fmax(d_rl[idx], d_rr[idx]);

		d_rl[idx] = exp(d_rl[idx] - maxRow);
		d_rr[idx] = exp(d_rr[idx] - maxRow);
		sum = d_rl[idx] + d_rr[idx];

		d_rl[idx] = d_rl[idx] / sum;
		d_rr[idx] = d_rr[idx] / sum;
	}
}

__global__ void build_log_likelihood_array_kernel(int maxIdx, double* d_log_likelihood_array, int rows, int cols)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double maxRow = 0;
		bool first = true;
		for (int j = 0; j < cols; j++)
		{
			if (first || d_log_likelihood_array[IDX2C(idx, j, rows)] > maxRow)
			{
				first = false;
				maxRow = d_log_likelihood_array[IDX2C(idx, j, rows)];
			}
		}

		double sum = 0;
		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = exp(d_log_likelihood_array[IDX2C(idx, j, rows)] - maxRow);
			sum += d_log_likelihood_array[IDX2C(idx, j, rows)];
		}

		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = d_log_likelihood_array[IDX2C(idx, j, rows)] / sum;
		}
	}
}

void cudaKernel::init(int numLabelsIn, MatrixXd &points, unsigned long long seed)
{
	printf("Init cuda\n");

//	hipSetDevice(1);
	runCuda(hipDeviceReset());
	numLabels = numLabelsIn;
	threads = dim3(512);
	blocks = dim3(numLabels / threads.x + 1);
	runCuda(hipMalloc((void**)&devState, numLabels * sizeof(hiprandState)));
	initCurand << <blocks, threads >> > (devState, seed, numLabels);
	runCuda(hipPeekAtLastError());
	runCuda(hipPeekAtLastError());

	runCuda(hipMalloc((void**)&d_labels, numLabels * sizeof(int)));
	runCuda(hipMalloc((void**)&d_sub_labels, numLabels * sizeof(int)));
	runCuda(hipMalloc((void**)&d_points, points.size() * sizeof(double)));
	runCuda(hipMemcpy(d_points, points.data(), points.size() * sizeof(double), hipMemcpyHostToDevice));
}

void cudaKernel::release()
{
	printf("Release cuda\n");

	if (devState != NULL)
	{
		runCuda(hipFree(devState));
	}

	if (d_labels != NULL)
	{
		runCuda(hipFree(d_labels));
	}

	if (d_sub_labels != NULL)
	{
		runCuda(hipFree(d_sub_labels));
	}

	if (d_points != NULL)
	{
		runCuda(hipFree(d_points));
	}
}

int cudaKernel::sample_log_cat_array_sub_cluster(LabelType *indices, int labelsSize, Eigen::MatrixXd &log_likelihood_array, std::vector<double>& lr_weights)
{
	int *d_indices;
	double *d_y;
	int *d_a;
	int *d_b;
	double* d_lr_weights;
	
	runCuda(hipMalloc((void**)&d_indices, sizeof(int) * labelsSize));
	runCuda(hipMemcpy(d_indices, indices, sizeof(int) * labelsSize, hipMemcpyHostToDevice));
	runCuda(hipMalloc((void**)&d_lr_weights, sizeof(double) * lr_weights.size()));
	runCuda(hipMemcpy(d_lr_weights, lr_weights.data(), sizeof(double) * lr_weights.size(), hipMemcpyHostToDevice));

	dim3 blocks_size = dim3(labelsSize / threads.x + 1);
	
	int n = log_likelihood_array.size();
	double *d_log_likelihood_array;
	runCuda(hipMalloc((void **)&d_log_likelihood_array, sizeof(double)*n));
	runCuda(hipMemcpy(d_log_likelihood_array, log_likelihood_array.data(), sizeof(double)*n, hipMemcpyHostToDevice));
	runCuda(hipMalloc((void **)&d_y, sizeof(double)*labelsSize*(log_likelihood_array.cols() + 2)));
	runCuda(hipMalloc((void **)&d_a, sizeof(int)*labelsSize*(log_likelihood_array.cols() + 2)));
	runCuda(hipMalloc((void **)&d_b, sizeof(int)*labelsSize*(log_likelihood_array.cols() + 2)));

	//	int dev = 0;//GPU index. need to loop for all GPUs

	build_log_likelihood_array_sub_cluster_kernel << <blocks_size, threads >> > (log_likelihood_array.rows(), d_log_likelihood_array, d_lr_weights, log_likelihood_array.rows(), log_likelihood_array.cols());
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
	
	sample_log_cat_array_sub_cluster_all << <blocks_size, threads >> > (devState, d_sub_labels, labelsSize, log_likelihood_array.cols(), d_log_likelihood_array, d_indices, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	//free memory
	runCuda(hipFree(d_indices));
	runCuda(hipFree(d_lr_weights));
	runCuda(hipFree(d_log_likelihood_array));
	runCuda(hipFree(d_y));
	runCuda(hipFree(d_a));
	runCuda(hipFree(d_b));
	return 0;
}

int cudaKernel::sample_log_cat_array_sub_cluster_v2(double* d_r, int* d_indices, int indicesSize, std::vector<double>& lr_weights, hipStream_t& stream)
{
	double* d_y;
	int* d_a;
	int* d_b;
	double* d_lr_weights;

	runCuda(hipMalloc((void**)&d_lr_weights, sizeof(double) * lr_weights.size()));
	runCuda(hipMemcpy(d_lr_weights, lr_weights.data(), sizeof(double) * lr_weights.size(), hipMemcpyHostToDevice));

	dim3 blocks_size = dim3(indicesSize / threads.x + 1);

	//int n = log_likelihood_array.size();
	//double* d_log_likelihood_array;
	//runCuda(hipMalloc((void**)&d_log_likelihood_array, sizeof(double) * n));
	//runCuda(hipMemcpy(d_log_likelihood_array, log_likelihood_array.data(), sizeof(double) * n, hipMemcpyHostToDevice));
	runCuda(hipMalloc((void**)&d_y, sizeof(double) * indicesSize * (2 + 2)));
	runCuda(hipMalloc((void**)&d_a, sizeof(int) * indicesSize * (2 + 2)));
	runCuda(hipMalloc((void**)&d_b, sizeof(int) * indicesSize * (2 + 2)));

	//	int dev = 0;//GPU index. need to loop for all GPUs

	build_log_likelihood_array_sub_cluster_kernel_v2 << <blocks_size, threads, 0, stream >> > (indicesSize, d_r, d_r + indicesSize, d_lr_weights);
	runCuda(hipPeekAtLastError());

	sample_log_cat_array_sub_cluster_all << <blocks_size, threads, 0, stream >> > (devState, d_sub_labels, indicesSize, 2, d_r, d_indices, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());
	runCuda(hipStreamSynchronize(stream));

	//free memory
	runCuda(hipFree(d_lr_weights));
//	runCuda(hipFree(d_log_likelihood_array));
	runCuda(hipFree(d_y));
	runCuda(hipFree(d_a));
	runCuda(hipFree(d_b));
	return 0;
}

int cudaKernel::sample_log_cat_array(Eigen::MatrixXd &log_likelihood_array)
{
	int n = log_likelihood_array.size();
	double *d_log_likelihood_array;
	double *d_y;
	int *d_a;
	int *d_b;

	runCuda(hipMalloc((void **)&d_log_likelihood_array, sizeof(double)*n));
	runCuda(hipMemcpy(d_log_likelihood_array, log_likelihood_array.data(), sizeof(double)*n, hipMemcpyHostToDevice));

	runCuda(hipMalloc((void **)&d_y, sizeof(double)*numLabels*(log_likelihood_array.cols() + 2)));
	runCuda(hipMalloc((void **)&d_a, sizeof(int)*numLabels*(log_likelihood_array.cols() + 2)));
	runCuda(hipMalloc((void **)&d_b, sizeof(int)*numLabels*(log_likelihood_array.cols() + 2)));

//	int dev = 0;//GPU index. need to loop for all GPUs
	build_log_likelihood_array_kernel << <blocks, threads >> > (log_likelihood_array.rows(), d_log_likelihood_array, log_likelihood_array.rows(), log_likelihood_array.cols());
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	sample_log_cat_array_all <<<blocks, threads>>>(devState, d_labels, numLabels, log_likelihood_array.cols(), d_log_likelihood_array, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	//free memory
	runCuda(hipFree(d_log_likelihood_array));
	runCuda(hipFree(d_y));
	runCuda(hipFree(d_a));
	runCuda(hipFree(d_b));
	return 0;

}

void cudaKernel::sample_sub_clusters_worker(LabelType label, LabelType* &indices, LabelType &indicesSize)
{
	int *d_indices;
	runCuda(hipMalloc((void **)&d_indices, sizeof(int)*numLabels));

	int *d_indicesSize;
	runCuda(hipMalloc(&d_indicesSize, sizeof(int)));
	runCuda(hipMemset(d_indicesSize, 0, sizeof(int)));

	find_indices << <blocks, threads >> > (d_labels, numLabels, label, d_indices, d_indicesSize);

	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	runCuda(hipMemcpy(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(indices, d_indices, indicesSize * sizeof(int), hipMemcpyDeviceToHost));

	runCuda(hipFree(d_indicesSize));
	runCuda(hipFree(d_indices));
}

void cudaKernel::sample_sub_clusters_worker_v2(LabelType label, int* d_indices, int* d_indicesSize, int& indicesSize, hipStream_t& stream)
{
	runCuda(hipMemsetAsync(d_indicesSize, 0, sizeof(int), stream));

	find_indices << <blocks, threads, 0, stream >> > (d_labels, numLabels, label, d_indices, d_indicesSize);

	runCuda(hipPeekAtLastError());
	//	runCuda(hipDeviceSynchronize());

	runCuda(hipMemcpyAsync(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost, stream));
}

void cudaKernel::create_suff_stats_dict_worker(
	LabelType label,
	LabelType &indicesSize,
	Eigen::MatrixXd &group_pts,
	Eigen::MatrixXd* &pts,
	Eigen::MatrixXd* &pts1,
	Eigen::MatrixXd* &pts2)
{
	int *d_indices;
	runCuda(hipMalloc((void **)&d_indices, sizeof(int)*numLabels));

	int *d_indicesSize;
	runCuda(hipMalloc(&d_indicesSize, sizeof(int)));
	runCuda(hipMemset(d_indicesSize, 0, sizeof(int)));

	find_indices << <blocks, threads >> > (d_labels, numLabels, label, d_indices, d_indicesSize);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
	runCuda(hipMemcpy(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost));

	double *d_group_pts;
	runCuda(hipMalloc((void **)&d_group_pts, sizeof(double)*group_pts.size()));
	runCuda(hipMemcpy(d_group_pts, group_pts.data(), sizeof(double)*group_pts.size(), hipMemcpyHostToDevice));

	double *d_pts;
	runCuda(hipMalloc((void **)&d_pts, sizeof(double)*group_pts.rows()*indicesSize));

	double *d_pts1;
	runCuda(hipMalloc((void **)&d_pts1, sizeof(double)*group_pts.rows()*indicesSize));

	double *d_pts2;
	runCuda(hipMalloc((void **)&d_pts2, sizeof(double)*group_pts.rows()*indicesSize));

	int *d_j1;
	int *d_j2;
	runCuda(hipMalloc(&d_j1, sizeof(int)));
	runCuda(hipMemset(d_j1, 0, sizeof(int)));
	runCuda(hipMalloc(&d_j2, sizeof(int)));
	runCuda(hipMemset(d_j2, 0, sizeof(int)));

	dim3 blocks_size = dim3(indicesSize / threads.x + 1);
	create_suff_stats_dict_worker_all << <blocks_size, threads >> > (
		d_sub_labels,
		numLabels,
		d_indices,
		d_indicesSize,
		d_group_pts,
		group_pts.rows(),
		d_pts,
		d_pts1,
		d_pts2,
		d_j1,
		d_j2);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	int j1;
	int j2;
	runCuda(hipMemcpy(&j1, d_j1, sizeof(int), hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(&j2, d_j2, sizeof(int), hipMemcpyDeviceToHost));

	pts = new Eigen::MatrixXd(group_pts.rows(), indicesSize);
	pts1 = new Eigen::MatrixXd(group_pts.rows(), j1);
	pts2 = new Eigen::MatrixXd(group_pts.rows(), j2);

	runCuda(hipMemcpy(pts->data(), d_pts, sizeof(double)*group_pts.rows()*indicesSize, hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(pts1->data(), d_pts1, sizeof(double)*group_pts.rows()*j1, hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(pts2->data(), d_pts2, sizeof(double)*group_pts.rows()*j2, hipMemcpyDeviceToHost));

	runCuda(hipFree(d_j1));
	runCuda(hipFree(d_j2));
	runCuda(hipFree(d_indicesSize));
	runCuda(hipFree(d_group_pts));
	runCuda(hipFree(d_pts));
	runCuda(hipFree(d_pts1));
	runCuda(hipFree(d_pts2));
}

void cudaKernel::sample_sub_labels()
{
	sample_sub_labels_all << <blocks, threads >> > (devState, d_sub_labels, numLabels);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
}

void cudaKernel::sample_labels(int initial_clusters, double outlier_mod)
{
	sample_labels_all << <blocks, threads >> > (devState, d_labels, numLabels, initial_clusters, outlier_mod);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
}

void cudaKernel::get_sub_labels(LabelsType &subLabels)
{
	int *h_subLabels;
	h_subLabels = (int*)malloc(numLabels * sizeof(int));
	runCuda(hipMemcpy(h_subLabels, d_sub_labels, numLabels * sizeof(int), hipMemcpyDeviceToHost));

	subLabels.resize(numLabels);
	for (size_t i = 0; i < numLabels; i++)
	{
		subLabels[i] = h_subLabels[i];
	}

	free(h_subLabels);
}

void cudaKernel::get_labels(LabelsType &labels)
{
	int *h_labels;
	h_labels = (int*)malloc(numLabels * sizeof(int));
	runCuda(hipMemcpy(h_labels, d_labels, numLabels * sizeof(int), hipMemcpyDeviceToHost));

	labels.resize(numLabels);
	for (size_t i = 0; i < numLabels; i++)
	{
		labels[i] = h_labels[i];
	}

	free(h_labels);
}

void cudaKernel::update_labels(int *updateLabels, int numLabels)
{
	runCuda(hipMemcpy(d_labels, updateLabels, sizeof(int)*numLabels, hipMemcpyHostToDevice));
}

void cudaKernel::remove_empty_clusters_worker(int limit)
{
	remove_empty_clusters_worker_all << <blocks, threads >> > (d_labels, numLabels, limit);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
}

void cudaKernel::split_cluster_local_worker(LabelType index, LabelType newIndex)
{
	split_cluster_local_worker_all << <blocks, threads >> > (devState, d_labels, numLabels, d_sub_labels, index, newIndex);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
}

void cudaKernel::merge_clusters_worker(LabelType index, LabelType newIndex)
{
	merge_clusters_worker_all << <blocks, threads >> > (devState, d_labels, numLabels, d_sub_labels, index, newIndex);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
}

void cudaKernel::reset_bad_clusters_worker(LabelType index)
{
	reset_bad_clusters_worker_all << <blocks, threads >> > (devState, d_labels, numLabels, d_sub_labels, index);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
}

void cudaKernel::get_sub_labels_count(int &l, int &r)
{
	int *d_l;
	runCuda(hipMalloc((void **)&d_l, sizeof(int)));
	runCuda(hipMemset(d_l, 0, sizeof(int)));
	
	int *d_r;
	runCuda(hipMalloc((void **)&d_r, sizeof(int)));
	runCuda(hipMemset(d_r, 0, sizeof(int)));

	get_sub_labels_count_all << <blocks, threads >> > (d_sub_labels, numLabels, d_l, d_r);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	runCuda(hipMemcpy(&l, d_l, sizeof(int), hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(&r, d_r, sizeof(int), hipMemcpyDeviceToHost));

	runCuda(hipFree(d_l));
	runCuda(hipFree(d_r));
}

// C(m,k) = A(m,n) * B(n,k)
void cudaKernel::naive_matrix_multiply(const double* A, const double* B, double* C, int m, int n, int k)
{
	int lda = m, ldb = k, ldc = m;
	const double alf = 1;
	const double bet = 0;
	const double* alpha = &alf;
	const double* beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}

void cudaKernel::dcolwise_dot_all(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r)
{
	dcolwise_dot_all_kernel << <blocks, threads >> > (maxIdx, rows, d_a, d_b, scalar, d_r);
}

//void cudaKernel::dcolwise_dot(Eigen::VectorXd& r, const Eigen::MatrixXd& a, const Eigen::MatrixXd& b)
//{
//	int sizeVec = a.cols();
//
//	double* d_a;
//	double* d_b;
//	double* d_c;
//	double* d_r;
//
//	runCuda(hipMalloc((void**)&d_a, sizeof(double) * a.size()));
//	runCuda(hipMalloc((void**)&d_b, sizeof(double) * b.size()));
//	runCuda(hipMalloc((void**)&d_c, sizeof(double) * b.rows() * a.cols()));
//	runCuda(hipMemcpy(d_a, a.data(), sizeof(double) * a.size(), hipMemcpyHostToDevice));
//	runCuda(hipMemcpy(d_b, b.data(), sizeof(double) * b.size(), hipMemcpyHostToDevice));
//
//	naive_matrix_multiply(d_b, d_a, d_c, b.rows(), a.cols(), b.cols());
//
//	runCuda(hipFree(d_b));
//
//	runCuda(hipMalloc((void**)&d_r, sizeof(double)* sizeVec));
//
//	dcolwise_dot_all(sizeVec, a.rows(), d_a, d_c, d_r);
//
//	r.resize(sizeVec);
//	runCuda(hipMemcpy(r.data(), d_r, sizeof(double)* sizeVec, hipMemcpyDeviceToHost));
//	
//	runCuda(hipFree(d_a));
//	runCuda(hipFree(d_c));
//	runCuda(hipFree(d_r));
//}

typedef struct
{
	//Stream for asynchronous command execution
	hipStream_t stream;

	int* d_indices;

	int indicesSize;

	double* d_r;

	int* d_indicesSize;

} subclusters_labels_plan;

void cudaKernel::create_subclusters_labels(int numClusters, std::vector<thin_cluster_params*> &cluster_params, int dim)
//LabelType* indices, LabelType indicesSize, distribution_sample* l_dist, distribution_sample* r_dist, std::vector<double> &lr_weights)
{
	subclusters_labels_plan *plan = new subclusters_labels_plan[numClusters];
	
	for (int i = 0; i < numClusters; i++)
	{
		runCuda(hipMalloc((void**)&(plan[i].d_indices), sizeof(int) * numLabels));
		runCuda(hipMalloc(&(plan[i].d_indicesSize), sizeof(int)));
	}

	for (int i = 0; i < numClusters; i++)
	{
		runCuda(hipStreamCreate(&(plan[i].stream)));
		sample_sub_clusters_worker_v2(i + 1, plan[i].d_indices, plan[i].d_indicesSize, plan[i].indicesSize, plan[i].stream);
	}

	for (int i = 0; i < numClusters; i++)
	{
		runCuda(hipStreamSynchronize(plan[i].stream));
		runCuda(hipMalloc((void**)&(plan[i].d_r), sizeof(double) * plan[i].indicesSize * 2));
		log_likelihood_v2(plan[i].d_r, plan[i].d_indices, plan[i].indicesSize, dim, cluster_params[i]->l_dist, plan[i].stream);
		log_likelihood_v2(plan[i].d_r + plan[i].indicesSize, plan[i].d_indices, plan[i].indicesSize, dim, cluster_params[i]->r_dist, plan[i].stream);
		sample_log_cat_array_sub_cluster_v2(plan[i].d_r, plan[i].d_indices, plan[i].indicesSize, cluster_params[i]->lr_weights, plan[i].stream);
	}

	//Wait for all operations to finish
	for (int i = 0; i < numClusters; i++)
	{
		runCuda(hipStreamSynchronize(plan[i].stream));
	}

	for (int i = 0; i < numClusters; i++)
	{
		runCuda(hipStreamDestroy(plan[i].stream));

		runCuda(hipFree(plan[i].d_indices));
		runCuda(hipFree(plan[i].d_indicesSize));
		runCuda(hipFree(plan[i].d_r));
	}

	delete[]plan;
}


void cudaKernel::checkCUDAError(hipError_t err, const char* file, int line)
{
	if (hipSuccess != err)
	{
		printf("Cuda error: %s(%d):%s.\n", file, line, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

#endif