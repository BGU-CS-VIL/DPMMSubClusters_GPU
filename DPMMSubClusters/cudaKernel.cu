#include "hip/hip_runtime.h"
#ifndef CudaKernel_CU
#define CudaKernel_CU

#include <omp.h>

#include <hip/hip_runtime.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<time.h>
#include "cudaKernel.cuh"
#include "cudaKernel_plan.cuh"

#define TILE_DIM  32
#define BLOCK_ROWS  8

//Begin of kernels

// function to define seed
__global__ void initCurand(hiprandState *state, unsigned long long seed, int maxIdx) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		hiprand_init(seed, idx, 0, &state[idx]);
	}
}

__device__ void sample_by_probability(hiprandState* state, double* weight, int numClusters, int rows, int idx, int* index,
	double* y, int* a, int* b)
{
	int i;
	int j;
	int k;
	int n = numClusters;

	a[idx] = 0;
	for (i = 1; i <= n; i++)
	{
		a[i * rows + idx] = i;
	}
	a[(n + 1) * rows + idx] = n + 1;

	b[idx] = 0;
	for (i = 1; i <= n; i++)
	{
		b[i * rows + idx] = i;
	}
	b[(n + 1) * rows + idx] = n + 1;

	y[idx] = 0.0;
	for (i = 1; i <= n; i++)
	{
		y[i * rows + idx] = weight[(i - 1) * rows + idx] * n;
	}
	y[(n + 1) * rows + idx] = 2.0;

	i = 0;
	j = n + 1;
	
	while (true)
	{
		do
		{
			i++;
		} while (y[b[i * rows + idx] * rows + idx] < 1.0);

		do
		{
			j--;
		} while (1.0 <= y[b[j * rows + idx] * rows + idx]);

		if (j <= i)
		{
			break;
		}

		k = b[i * rows + idx];
		b[i * rows + idx] = b[j * rows + idx];
		b[j * rows + idx] = k;
	}

	i = j;
	j++;

	while (0 < i)
	{

		while (y[b[j * rows + idx] * rows + idx] <= 1.0)
		{
			j++;
		}

		if (n < j)
		{
			break;
		}

		y[b[j * rows + idx] * rows + idx] = y[b[j * rows + idx] * rows + idx] - (1.0 - y[b[i * rows + idx] * rows + idx]);
		a[b[i * rows + idx] * rows + idx] = b[j * rows + idx];

		if (y[b[j * rows + idx] * rows + idx] < 1.0)
		{
			k = b[i * rows + idx];
			b[i * rows + idx] = b[j * rows + idx];
			b[j * rows + idx] = k;
			j++;
		}
		else
		{
			i--;
		}
	}

	double r;

	i = (int)(hiprand_uniform(state) * (n - 1 + 0.999999) + 1);
	r = hiprand_uniform(state);

	if (y[i * rows + idx] < r)
	{
		i = a[i * rows + idx];

	}
	*index = i;
}

__global__ void sample_log_cat_array_all(hiprandState *state, int *dev_sample, int maxIdx, int numClusters, double *d_log_likelihood_array, double *y, int *a, int *b)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_by_probability(&state[idx], d_log_likelihood_array, numClusters, maxIdx, idx, &dev_sample[idx], y, a, b);
	}
}

__global__ void sample_log_cat_array_sub_cluster_all(hiprandState *state, int *dev_sample, int maxIdx, int num, double *d_log_likelihood_array, int *indices, double *y, int *a, int *b)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_by_probability(&state[indices[idx]], d_log_likelihood_array, num, maxIdx, idx, &dev_sample[indices[idx]], y, a, b);
	}
}

__device__ void sample_sub_label(hiprandState *state, int *d_label)
{
	*d_label = (int)(hiprand_uniform(state) * (2 - 1 + 0.999999) + 1);
}

__global__ void sample_sub_labels_all(hiprandState *state, int *d_labels, int maxIdx)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_sub_label(&state[idx], &d_labels[idx]);
	}
}

__device__ void sample_label(hiprandState *state, int *d_label, int initial_clusters, double outlier_mod)
{
	*d_label = (int)(hiprand_uniform(state) * (initial_clusters - 1 + 0.999999) + 1 + ((outlier_mod > 0) ? 1 : 0));
}

__global__ void sample_labels_all(hiprandState *state, int *d_labels, int maxIdx, int initial_clusters, double outlier_mod)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_label(&state[idx], &d_labels[idx], initial_clusters, outlier_mod);
	}
}

__device__ void remove_empty_clusters_worker(int *d_label, int limit)
{
	if (*d_label > limit)
	{
		*d_label -= 1;
	}
}

__global__ void remove_empty_clusters_worker_all(int *d_labels, int maxIdx, int limit)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		remove_empty_clusters_worker(&d_labels[idx], limit);
	}
}

__global__ void find_indices(int *d_labels, int maxIdx, int label, int *d_indices, int *d_indicesSize)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		if (d_labels[idx] == label)
		{
			int curIndex = atomicAdd(d_indicesSize, 1);
			d_indices[curIndex] = idx;
		}
	}
}

__device__ void split_cluster_local_worker(hiprandState *state, int *d_labels, int *d_sub_label, int index, int newIndex)
{
	if (*d_labels == index + 1)
	{
		if (*d_sub_label == 2)
		{
			*d_labels = newIndex + 1;
		}
		*d_sub_label = hiprand(state) % 2 + 1;
	}
}

__global__ void split_cluster_local_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index, int newIndex)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		split_cluster_local_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index, newIndex);
	}
}

__device__ void merge_clusters_worker(hiprandState *state, int *d_labels, int *d_sub_labels, int index, int newIndex)
{
	if (*d_labels == index + 1)
	{
		*d_sub_labels = 1;
	}
	if (*d_labels == newIndex + 1)
	{
		*d_sub_labels = 2;
	}
	if (*d_labels == newIndex + 1)
	{
		*d_labels = index + 1;
	}
}

__global__ void merge_clusters_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index, int newIndex)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		merge_clusters_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index, newIndex);
	}
}

__device__ void reset_bad_clusters_worker(hiprandState *state, int *d_labels, int *d_sub_labels, int index)
{
	if (*d_labels == index + 1)
	{
		*d_sub_labels = hiprand(state) % 2 + 1;
	}
}

__global__ void reset_bad_clusters_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		reset_bad_clusters_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index);
	}
}

__global__ void get_sub_labels_count_all(int *d_sub_labels, int maxIdx, int *l, int *r)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		if (d_sub_labels[idx] == 1)
		{
			atomicAdd(l, 1);
		}
		else if (d_sub_labels[idx] == 2)
		{
			atomicAdd(r, 1);
		}
	}
}

__global__ void create_suff_stats_dict_worker_all(
	int* d_sub_labels,
	int maxIdx,
	int *d_indices,
	int *d_indicesSize,
	double *group_pts,
	int group_pts_rows,
	double *pts,
	double *pts1,
	double *pts2,
	int *d_j1,
	int *d_j2)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < *d_indicesSize)
	{
		if (d_sub_labels[d_indices[idx]] == 1)
		{
			int curj1 = atomicAdd(d_j1, 1);
			for (int i = 0; i < group_pts_rows; i++)
			{
				double pt = group_pts[IDX2C(i, d_indices[idx], group_pts_rows)];
				pts[IDX2C(i, idx, group_pts_rows)] = pt;
				pts1[IDX2C(i, curj1, group_pts_rows)] = pt;
			}
		}
		else if (d_sub_labels[d_indices[idx]] == 2)
		{
			int curj2 = atomicAdd(d_j2, 1);
			for (int i = 0; i < group_pts_rows; i++)
			{
				double pt = group_pts[IDX2C(i, d_indices[idx], group_pts_rows)];
				pts[IDX2C(i, idx, group_pts_rows)] = pt;
				pts2[IDX2C(i, curj2, group_pts_rows)] = pt;
			}
		}
	}
}

__global__ void dcolwise_dot_all_kernel(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, int r_offset)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double sum = 0;
		for (int i = 0; i < rows; i++)
		{
			sum += d_a[idx * rows + i] * d_b[idx * rows + i];
		}

		d_r[idx + r_offset] = scalar - sum / 2;
	}
}

__global__ void dcolwise_dot_with_log_kernel(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, double weight)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double sum = 0;
		for (int i = 0; i < rows; i++)
		{
			sum += d_a[idx * rows + i] * d_b[idx * rows + i];
		}

		d_r[idx] = scalar - sum / 2 + log(weight);
	}
}

__global__ void build_log_likelihood_array_sub_cluster_kernel(int maxIdx, double* d_r, int r_offset, double* d_lr_weights)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double maxRow;
		double sum;

		d_r[idx] += log(d_lr_weights[0]);
		d_r[idx + r_offset] += log(d_lr_weights[1]);
		maxRow = fmax(d_r[idx], d_r[idx + r_offset]);

		d_r[idx] = exp(d_r[idx] - maxRow);
		d_r[idx + r_offset] = exp(d_r[idx + r_offset] - maxRow);
		sum = d_r[idx] + d_r[idx + r_offset];

		d_r[idx] = d_r[idx] / sum;
		d_r[idx + r_offset] = d_r[idx + r_offset] / sum;
	}
}

__global__ void build_log_likelihood_array_kernel(int maxIdx, double* d_log_likelihood_array, int rows, int cols)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double maxRow = 0;
		bool first = true;
		for (int j = 0; j < cols; j++)
		{
			if (first || d_log_likelihood_array[IDX2C(idx, j, rows)] > maxRow)
			{
				first = false;
				maxRow = d_log_likelihood_array[IDX2C(idx, j, rows)];
			}
		}

		double sum = 0;
		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = exp(d_log_likelihood_array[IDX2C(idx, j, rows)] - maxRow);
			sum += d_log_likelihood_array[IDX2C(idx, j, rows)];
		}

		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = d_log_likelihood_array[IDX2C(idx, j, rows)] / sum;
		}
	}
}

__global__ void update_labels_by_max_index_kernel(double* parr, int* d_labels, int maxIdx, int dim)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		double max = 0;
		bool first = true;
		int max_index = 0;
		for (int i = 0; i < dim; ++i)
		{
			if (first || max < parr[IDX2C(idx, i, maxIdx)])
			{
				first = false;
				max = parr[IDX2C(idx, i, maxIdx)];
				max_index = i;
			}
		}
		d_labels[idx] = max_index + 1;
	}
}

__global__ void gpu_matrix_mult(double* a, double* b, double* c, int m, int n, int k)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	double sum = 0;

	if (col < k && row < m)
	{
		for (int i = 0; i < n; i++)
		{
			sum += a[IDX2C(row, i, m)] * b[IDX2C(i, col, n)];
		}
		c[IDX2C(row, col, m)] = sum;
	}
}

__global__ void sum_rowwise_kernel(double* d_A, double* d_B, int rows, int cols)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < rows)
	{
		double sum = 0;
		for (int j = 0; j < cols; j++)
		{
			sum += d_A[IDX2C(idx, j, rows)];
		}
		d_B[idx] = sum;
	}
}

__global__ void nodiag_normalize(double* A, double* I, int n, int i) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n)
		if (x == i && x != y) {
			I[x * n + y] /= A[i * n + i];
			A[x * n + y] /= A[i * n + i];
		}

}

__global__ void diag_normalize(double* A, double* I, int n, int i) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n)
		if (x == y && x == i) {
			I[x * n + y] /= A[i * n + i];
			A[x * n + y] /= A[i * n + i];
		}
}

__global__ void gaussjordan(double* A, double* I, int n, int i)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n) {
		if (x != i) {
			I[x * n + y] -= I[i * n + y] * A[x * n + i];
			if (y != i) {
				A[x * n + y] -= A[i * n + y] * A[x * n + i];
			}
		}
	}

}

__global__ void set_zero(double* A, double* I, int n, int i) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n) {
		if (x != i) {
			if (y == i) {
				A[x * n + y] = 0;
			}
		}
	}
}

__global__ void transposeGPUcoalescing(double* matIn, int n, int m, double* matTran) {
	__shared__ double tile[TILE_DIM][TILE_DIM];
	int i_n = blockIdx.x * TILE_DIM + threadIdx.x;
	int i_m = blockIdx.y * TILE_DIM + threadIdx.y; // <- threadIdx.y only between 0 and 7

	// Load matrix into tile
	// Every Thread loads in this case 4 elements into tile.
	int i;
	for (i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		if (i_n < n && (i_m + i) < m) {
			tile[threadIdx.y + i][threadIdx.x] = matIn[(i_m + i) * n + i_n];
		}
	}
	__syncthreads();

	i_n = blockIdx.y * TILE_DIM + threadIdx.x;
	i_m = blockIdx.x * TILE_DIM + threadIdx.y;

	for (i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
		if (i_n < m && (i_m + i) < n) {
			matTran[(i_m + i) * m + i_n] = tile[threadIdx.x][threadIdx.y + i]; // <- multiply by m, non-squared!

		}
	}
}

//End of kernels

//cudaKernel class is the class that wraps all the calls to cuda APIs
void cudaKernel::init(int numLabelsIn, MatrixXd &points, unsigned long long seed, bool verbose, int forceKernel)
{
	use_verbose = verbose;

	if (use_verbose)
	{
		printf("Init cuda\n");
	}

	int numGPU;
	int driverVersion = 0, runtimeVersion = 0;

	lastDevice = 0;

	runCuda(hipGetDeviceCount(&numGPU));

	//Remove the following line in order to run with more than 1 GPU
	numGPU = 1;

	if (use_verbose)
	{
		printf("Number of GPUs: %i\n", numGPU);
		printf("number of host CPUs:\t%d\n", omp_get_num_procs());
	}

	for (int i = 0; i < numGPU; i++)
	{
		hipSetDevice(i);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, i);

		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);

		if (use_verbose)
		{
			printf("\nDevice %d: \"%s\"\n", i, deviceProp.name);
			printf("CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
			printf("CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);
		}

		int* dummy;
		hipStream_t stream;

		hipStreamCreate(&stream);

		hipError_t err = hipMallocAsync((void**)&dummy, sizeof(int), stream);
		if (hipSuccess != err)
		{
			hipGetLastError();
			printf("Not capable device. Can't perform asynchronous memory allocation.\n");
		}
		else
		{
			hipFreeAsync(dummy, stream);
			gpuCapabilities[i] = gpuCapability();
		}
		hipStreamSynchronize(stream);
		hipStreamDestroy(stream);
	}
		
	if (use_verbose)
	{
		printf("\nNumber of GPUs that will be used: %i\n\n", (int)gpuCapabilities.size());
	}

	numLabels = numLabelsIn;
	if (use_verbose)
	{
		printf("Number of Labels: %i\n", numLabels);
	}

	threads = dim3(512);
	blocks = dim3(numLabels / threads.x + 1);

	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		hipSetDevice(iter->first);
		runCuda(hipDeviceReset());
		runCuda(hipMalloc((void**)&(iter->second.devState), numLabels * sizeof(hiprandState)));
		initCurand << <blocks, threads >> > (iter->second.devState, seed, numLabels);
		runCuda(hipPeekAtLastError());
		runCuda(hipDeviceSynchronize());

		runCuda(hipMalloc((void**)&(iter->second.d_labels), numLabels * sizeof(int)));
		runCuda(hipMalloc((void**)&(iter->second.d_sub_labels), numLabels * sizeof(int)));
		runCuda(hipMalloc((void**)&(iter->second.d_points), points.size() * sizeof(double)));
		runCuda(hipMemcpy(iter->second.d_points, points.data(), points.size() * sizeof(double), hipMemcpyHostToDevice));
		iter->second.pointsRows = (int)points.rows();
		iter->second.pointsCols = (int)points.cols();

		optimize_kernels(iter->second, forceKernel);
	}

	if (gpuCapabilities.size() > 0)
	{
		hipSetDevice(gpuCapabilities.begin()->first);
	}
}

void cudaKernel::optimize_kernels(gpuCapability& gpu, int forceKernel)
{
	if (forceKernel == 1)
	{
		gpu.do_multiplie_matrix_by_transpose = &do_multiplie_matrix_by_transpose1;
		gpu.matrixMultiply = &do_matrixMultiply1;
	}
	else if (forceKernel == 2)
	{
		gpu.do_multiplie_matrix_by_transpose = &do_multiplie_matrix_by_transpose2;
		gpu.matrixMultiply = &do_matrixMultiply2;
	}
	else
	{
		int numIter = 5;
		MatrixXd A = MatrixXd::Random(gpu.pointsRows, numLabels / 2);

		double* d_A;
		double* d_B;
		runCuda(hipMalloc((void**)&(d_A), A.size() * sizeof(double)));
		runCuda(hipMemcpy(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice));
		runCuda(hipMalloc((void**)&(d_B), A.rows() * A.rows() * sizeof(double)));
		hipStream_t stream;
		runCuda(hipStreamCreate(&stream));

		gpu.matrixMultiply = &do_matrixMultiply1;
		do_multiplie_matrix_by_transpose1(d_A, d_B, (int)(A.rows()), (int)(A.cols()), gpu, stream, use_verbose);
		runCuda(hipStreamSynchronize(stream));

		clock_t begin = clock();
		for (int i = 0; i < numIter; i++)
		{
			do_multiplie_matrix_by_transpose1(d_A, d_B, (int)(A.rows()), (int)(A.cols()), gpu, stream, use_verbose);
		}
		runCuda(hipStreamSynchronize(stream));
		clock_t end = clock();
		double took1 = double(end - begin);

		gpu.matrixMultiply = &do_matrixMultiply2;
		do_multiplie_matrix_by_transpose2(d_A, d_B, (int)(A.rows()), (int)(A.cols()), gpu, stream, use_verbose);
		runCuda(hipStreamSynchronize(stream));

		begin = clock();
		for (int i = 0; i < numIter; i++)
		{
			do_multiplie_matrix_by_transpose2(d_A, d_B, (int)(A.rows()), (int)(A.cols()), gpu, stream, use_verbose);
		}
		runCuda(hipStreamSynchronize(stream));
		end = clock();
		double took2 = double(end - begin);

		runCuda(hipFree(d_A));
		runCuda(hipFree(d_B));
		runCuda(hipStreamDestroy(stream));

		if (use_verbose)
		{
			printf("multiplie_matrix_by_transpose kernel:\n");
			printf("  Kernel 1 took: %f\n", took1);
			printf("  Kernel 2 took: %f\n", took2);
			printf("  Using kernel: #");
		}
		if (took1 < took2)
		{
			gpu.do_multiplie_matrix_by_transpose = &do_multiplie_matrix_by_transpose1;
			gpu.matrixMultiply = &do_matrixMultiply1;
			if (use_verbose)
			{
				printf("1\n");
			}
		}
		else
		{
			gpu.do_multiplie_matrix_by_transpose = &do_multiplie_matrix_by_transpose2;
			gpu.matrixMultiply = &do_matrixMultiply2;
			if (use_verbose)
			{
				printf("2\n");
			}
		}
	}
}

void cudaKernel::release()
{
	if (use_verbose)
	{
		printf("Release cuda\n");
	}

	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		if (iter->second.devState != NULL)
		{
			runCuda(hipFree(iter->second.devState));
		}

		if (iter->second.d_labels != NULL)
		{
			runCuda(hipFree(iter->second.d_labels));
		}

		if (iter->second.d_sub_labels != NULL)
		{
			runCuda(hipFree(iter->second.d_sub_labels));
		}

		if (iter->second.d_points != NULL)
		{
			runCuda(hipFree(iter->second.d_points));
		}
	}
}

int cudaKernel::pick_first_device()
{
	int result;
	result = gpuCapabilities.begin()->first;
	hipSetDevice(result);
	return result;
}

int cudaKernel::pick_any_device()
{
	int result;
	++lastDevice;
	if (lastDevice >= gpuCapabilities.size())
		lastDevice = 0;

	int i = 0;
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); i <= lastDevice && iter != gpuCapabilities.end(); iter++, ++i)
	{
		result = iter->first;
	}
	hipSetDevice(result);
	return result;
}

void cudaKernel::sample_log_cat_array_sub_cluster(
	double* d_r,
	int r_offset,
	int* d_indices,
	int indicesSize,
	double* d_lr_weights,
	hipStream_t& stream,
	int deviceId)
{
	double* d_y;
	int* d_a;
	int* d_b;

	runCuda(hipMallocAsync((void**)&d_y, sizeof(double) * indicesSize * (2 + 2), stream));
	runCuda(hipMallocAsync((void**)&d_a, sizeof(int) * indicesSize * (2 + 2), stream));
	runCuda(hipMallocAsync((void**)&d_b, sizeof(int) * indicesSize * (2 + 2), stream));

	build_log_likelihood_array_sub_cluster_kernel << <blocks, threads, 0, stream >> > (indicesSize, d_r, r_offset, d_lr_weights);
	runCuda(hipPeekAtLastError());

	sample_log_cat_array_sub_cluster_all << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_sub_labels, indicesSize, 2, d_r, d_indices, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());

	runCuda(hipFreeAsync(d_y, stream));
	runCuda(hipFreeAsync(d_a, stream));
	runCuda(hipFreeAsync(d_b, stream));

	update_sub_labels_to_all_other_devices(deviceId, stream);
}

void cudaKernel::sample_log_cat_array(
	double* d_r,
	int dim,
	hipStream_t& stream,
	int deviceId)
{
	double* d_y;
	int* d_a;
	int* d_b;

	runCuda(hipMallocAsync((void**)&d_y, sizeof(double) * numLabels * (dim + 2), stream));
	runCuda(hipMallocAsync((void**)&d_a, sizeof(int) * numLabels * (dim + 2), stream));
	runCuda(hipMallocAsync((void**)&d_b, sizeof(int) * numLabels * (dim + 2), stream));
	build_log_likelihood_array_kernel << <blocks, threads, 0, stream >> > (numLabels, d_r, numLabels, dim);
	runCuda(hipPeekAtLastError());

	sample_log_cat_array_all << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, dim, d_r, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());

	runCuda(hipFreeAsync(d_y, stream));
	runCuda(hipFreeAsync(d_a, stream));
	runCuda(hipFreeAsync(d_b, stream));

	update_labels_to_all_other_devices(deviceId, stream);
}

void cudaKernel::sample_sub_clusters_worker(LabelType label, int* d_indices, int &indicesSize, hipStream_t& stream, int deviceId)
{
	int* d_indicesSize;
	runCuda(hipMallocAsync(&d_indicesSize, sizeof(int), stream));
	runCuda(hipMemsetAsync(d_indicesSize, 0, sizeof(int), stream));

	find_indices << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].d_labels, numLabels, label, d_indices, d_indicesSize);
	runCuda(hipMemcpyAsync(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost, stream));
	runCuda(hipFreeAsync(d_indicesSize, stream));

	runCuda(hipPeekAtLastError());
}

std::map<LabelType, std::shared_ptr<thin_suff_stats>> cudaKernel::create_sufficient_statistics(
	LabelsType& indices,
	const std::shared_ptr<hyperparams>& hyperParams,
	const std::shared_ptr<hyperparams>& posterior)
{
	std::map<LabelType, std::shared_ptr<thin_suff_stats>> suff_stats_dict;
	sufficient_statistics_plan* plan = new sufficient_statistics_plan[indices.size()];

	for (LabelType index = 0; index < indices.size(); index++)
	{
		LabelType label = indices[index] + 1;

		plan[index].deviceId = pick_any_device();
		runCuda(hipStreamCreate(&(plan[index].stream)));

		int pointsRows = gpuCapabilities[plan[index].deviceId].pointsRows;
		int* d_indices;
		runCuda(hipMallocAsync((void**)&d_indices, sizeof(int) * numLabels, plan[index].stream));

		runCuda(hipMallocAsync(&(plan[index].d_indicesSize), sizeof(int), plan[index].stream));
		runCuda(hipMemsetAsync(plan[index].d_indicesSize, 0, sizeof(int), plan[index].stream));

		find_indices << <blocks, threads, 0, plan[index].stream >> > (gpuCapabilities[plan[index].deviceId].d_labels, numLabels, label, d_indices, plan[index].d_indicesSize);
		runCuda(hipPeekAtLastError());
		runCuda(hipMemcpyAsync(&(plan[index].indicesSize), plan[index].d_indicesSize, sizeof(int), hipMemcpyDeviceToHost));

		runCuda(hipMallocAsync((void**)&(plan[index].d_pts), sizeof(double) * pointsRows * plan[index].indicesSize, plan[index].stream));

		runCuda(hipMallocAsync((void**)&(plan[index].d_pts1), sizeof(double) * pointsRows * plan[index].indicesSize, plan[index].stream));

		runCuda(hipMallocAsync((void**)&(plan[index].d_pts2), sizeof(double) * pointsRows * plan[index].indicesSize, plan[index].stream));

		runCuda(hipMallocAsync(&(plan[index].d_j1), sizeof(int), plan[index].stream));
		runCuda(hipMemsetAsync(plan[index].d_j1, 0, sizeof(int), plan[index].stream));
		runCuda(hipMallocAsync(&(plan[index].d_j2), sizeof(int), plan[index].stream));
		runCuda(hipMemsetAsync(plan[index].d_j2, 0, sizeof(int), plan[index].stream));

		dim3 blocks_size = dim3(numLabels / threads.x + 1);
		create_suff_stats_dict_worker_all << <blocks_size, threads, 0, plan[index].stream >> > (
			gpuCapabilities[plan[index].deviceId].d_sub_labels,
			numLabels,
			d_indices,
			plan[index].d_indicesSize,
			gpuCapabilities[plan[index].deviceId].d_points,
			pointsRows,
			plan[index].d_pts,
			plan[index].d_pts1,
			plan[index].d_pts2,
			plan[index].d_j1,
			plan[index].d_j2);
		runCuda(hipPeekAtLastError());

		runCuda(hipFreeAsync(d_indices, plan[index].stream));
	}
	
	for (LabelType index = 0; index < indices.size(); index++)
	{
		int pointsRows = gpuCapabilities[plan[index].deviceId].pointsRows;
		plan[index].tss = std::make_shared<thin_suff_stats>();

		runCuda(hipStreamSynchronize(plan[index].stream));

		runCuda(hipStreamCreate(&(plan[index].stream1)));
		do_create_sufficient_statistics(plan[index].d_pts1, pointsRows, plan[index].d_j1, hyperParams, posterior, plan[index].stream1, plan[index].tss->l_suff, plan[index].deviceId);

		runCuda(hipStreamCreate(&(plan[index].stream2)));
		do_create_sufficient_statistics(plan[index].d_pts2, pointsRows, plan[index].d_j2, hyperParams, posterior, plan[index].stream2, plan[index].tss->r_suff, plan[index].deviceId);

		runCuda(hipStreamCreate(&(plan[index].stream3)));
		do_create_sufficient_statistics(plan[index].d_pts, pointsRows, plan[index].d_indicesSize, hyperParams, posterior, plan[index].stream3, plan[index].tss->cluster_suff, plan[index].deviceId);

	}

	for (LabelType index = 0; index < indices.size(); index++)
	{
		runCuda(hipStreamSynchronize(plan[index].stream1));
		runCuda(hipStreamSynchronize(plan[index].stream2));
		runCuda(hipStreamSynchronize(plan[index].stream3));

		runCuda(hipStreamDestroy(plan[index].stream1));
		runCuda(hipStreamDestroy(plan[index].stream2));
		runCuda(hipStreamDestroy(plan[index].stream3));

		runCuda(hipFreeAsync(plan[index].d_indicesSize, plan[index].stream));
		runCuda(hipFreeAsync(plan[index].d_pts, plan[index].stream));
		runCuda(hipFreeAsync(plan[index].d_pts1, plan[index].stream));
		runCuda(hipFreeAsync(plan[index].d_pts2, plan[index].stream));
		runCuda(hipFreeAsync(plan[index].d_j1, plan[index].stream));
		runCuda(hipFreeAsync(plan[index].d_j2, plan[index].stream));
		runCuda(hipStreamDestroy(plan[index].stream));

		suff_stats_dict[indices[index]] = plan[index].tss;
	}

	delete[]plan;
	return suff_stats_dict;
}

// A -> (N x M) 
//Call to the best kernel best of the dimensions that are needed in this run
void cudaKernel::multiplie_matrix_by_transpose(double* d_A, double* d_B, int N, int M, int deviceId, hipStream_t& stream)
{
	gpuCapabilities[deviceId].do_multiplie_matrix_by_transpose(d_A, d_B, N, M, gpuCapabilities[deviceId], stream, use_verbose);
}

// A -> (N x M) 
//This method is good from low dimensions
void cudaKernel::do_multiplie_matrix_by_transpose1(double* d_A, double* d_B, int N, int M, gpuCapability &gpu, hipStream_t& stream, bool use_verbose)
{
	dim3 blocks_size = dim3(N / TILE_DIM + 1, M / TILE_DIM + 1);
	dim3 threads = dim3(TILE_DIM, BLOCK_ROWS);

	double* d_A_T;
	runCuda(hipMallocAsync(&d_A_T, sizeof(double) * N * M, stream));
	transposeGPUcoalescing << <blocks_size, threads, 0, stream >> > (d_A, N, M, d_A_T);
	runCuda(hipPeekAtLastError());
	gpu.matrixMultiply(d_A, d_A_T, d_B, N, M, N, stream, use_verbose);
	runCuda(hipFreeAsync(d_A_T, stream));
}

// A -> (N x M) 
//This method is good from high dimensions
void cudaKernel::do_multiplie_matrix_by_transpose2(double* d_A, double* d_B, int N, int M, gpuCapability& gpu, hipStream_t &stream, bool use_verbose)
{
	hipblasHandle_t handle;
	runCuda(hipblasCreate(&handle));
	runCuda(hipblasSetStream(handle, stream));
	double alpha = 1.0;
	double beta = 0.0;
	runCuda(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, M, &alpha, d_A, N, d_A, N, &beta, d_B, N));

	runCuda(hipblasDestroy(handle));
}

//C = A*B * (A* B)T
void cudaKernel::multiplie_matrix_for_inverseWishart(const MatrixXd& A, const MatrixXd& B, MatrixXd& C)
{
	hipblasHandle_t handle;
	runCuda(hipblasCreate(&handle));
	double alpha = 1.0;
	double beta = 0.0;
	double* d_A;
	double* d_B;
	double* d_C;
	double* d_temp;

	runCuda(hipMalloc(&d_A, sizeof(double) * A.size()));
	runCuda(hipMemcpy(d_A, A.data(), sizeof(double) * A.size(), hipMemcpyHostToDevice));
	runCuda(hipMalloc(&d_B, sizeof(double) * B.size()));
	runCuda(hipMemcpy(d_B, B.data(), sizeof(double) * B.size(), hipMemcpyHostToDevice));
	runCuda(hipMalloc(&d_temp, sizeof(double) * A.rows() * B.cols()));
	runCuda(hipMalloc(&d_C, sizeof(double) * A.rows() * A.rows()));

	// temp(m,k) = A(m,n) * B(n,k)
	int m = (int)(A.rows());
	int n = (int)(A.cols());
	int k = (int)(B.cols());
	runCuda(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &alpha, d_A, m, d_B, n, &beta, d_temp, m));

	// C(m,m) = temp(m,k) * temp(m,k)'
	runCuda(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, m, k, &alpha, d_temp, m, d_temp, m, &beta, d_C, m));

	runCuda(hipblasDestroy(handle));

	C.resize(A.rows(), A.rows());
	runCuda(hipMemcpy(C.data(), d_C, sizeof(double) * A.rows() * A.rows(), hipMemcpyDeviceToHost));

	runCuda(hipFree(d_A));
	runCuda(hipFree(d_B));
	runCuda(hipFree(d_temp));
	runCuda(hipFree(d_C));
}

void cudaKernel::create_suff_stats_dict_worker(
	LabelType label,
	LabelType& indicesSize,
	Eigen::MatrixXd& pts,
	Eigen::MatrixXd& pts1,
	Eigen::MatrixXd& pts2)
{
	int deviceId = pick_first_device();
	int pointsRows = gpuCapabilities[deviceId].pointsRows;
	int* d_indices;
	runCuda(hipMalloc((void**)&d_indices, sizeof(int) * numLabels));

	int* d_indicesSize;
	runCuda(hipMalloc(&d_indicesSize, sizeof(int)));
	runCuda(hipMemset(d_indicesSize, 0, sizeof(int)));

	find_indices << <blocks, threads >> > (gpuCapabilities[deviceId].d_labels, numLabels, label, d_indices, d_indicesSize);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
	runCuda(hipMemcpy(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost));

	double* d_pts;
	runCuda(hipMalloc((void**)&d_pts, sizeof(double) * pointsRows * indicesSize));

	double* d_pts1;
	runCuda(hipMalloc((void**)&d_pts1, sizeof(double) * pointsRows * indicesSize));

	double* d_pts2;
	runCuda(hipMalloc((void**)&d_pts2, sizeof(double) * pointsRows * indicesSize));

	int* d_j1;
	int* d_j2;
	runCuda(hipMalloc(&d_j1, sizeof(int)));
	runCuda(hipMemset(d_j1, 0, sizeof(int)));
	runCuda(hipMalloc(&d_j2, sizeof(int)));
	runCuda(hipMemset(d_j2, 0, sizeof(int)));

	dim3 blocks_size = dim3(indicesSize / threads.x + 1);
	create_suff_stats_dict_worker_all << <blocks_size, threads >> > (
		gpuCapabilities[deviceId].d_sub_labels,
		numLabels,
		d_indices,
		d_indicesSize,
		gpuCapabilities[deviceId].d_points,
		pointsRows,
		d_pts,
		d_pts1,
		d_pts2,
		d_j1,
		d_j2);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	int j1;
	int j2;
	runCuda(hipMemcpy(&j1, d_j1, sizeof(int), hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(&j2, d_j2, sizeof(int), hipMemcpyDeviceToHost));

	pts.resize(pointsRows, indicesSize);
	pts1.resize(pointsRows, j1);
	pts2.resize(pointsRows, j2);

	runCuda(hipMemcpy(pts.data(), d_pts, sizeof(double) * pointsRows * indicesSize, hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(pts1.data(), d_pts1, sizeof(double) * pointsRows * j1, hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(pts2.data(), d_pts2, sizeof(double) * pointsRows * j2, hipMemcpyDeviceToHost));

	runCuda(hipFree(d_j1));
	runCuda(hipFree(d_j2));
	runCuda(hipFree(d_indices));
	runCuda(hipFree(d_indicesSize));
	runCuda(hipFree(d_pts));
	runCuda(hipFree(d_pts1));
	runCuda(hipFree(d_pts2));
}

void cudaKernel::sample_sub_labels()
{
	int deviceId = pick_first_device();
	sample_sub_labels_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_sub_labels, numLabels);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	update_sub_labels_to_all_other_devices(deviceId);
}

void cudaKernel::sample_labels(int initial_clusters, double outlier_mod)
{
	int deviceId = pick_first_device();
	sample_labels_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, initial_clusters, outlier_mod);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	update_labels_to_all_other_devices(deviceId);
}

void cudaKernel::get_sub_labels(std::shared_ptr<LabelsType> &subLabels)
{
	int deviceId = pick_first_device();
	int *h_subLabels;
	h_subLabels = (int*)malloc(numLabels * sizeof(int));
	runCuda(hipMemcpy(h_subLabels, gpuCapabilities[deviceId].d_sub_labels, numLabels * sizeof(int), hipMemcpyDeviceToHost));

	subLabels->resize(numLabels);
	for (size_t i = 0; i < numLabels; i++)
	{
		(*subLabels)[i] = h_subLabels[i];
	}

	free(h_subLabels);
}

void cudaKernel::get_labels(std::shared_ptr<LabelsType> &labels)
{
	int deviceId = pick_first_device();
	int *h_labels;
	h_labels = (int*)malloc(numLabels * sizeof(int));
	runCuda(hipMemcpy(h_labels, gpuCapabilities[deviceId].d_labels, numLabels * sizeof(int), hipMemcpyDeviceToHost));

	labels->resize(numLabels);
	for (size_t i = 0; i < numLabels; i++)
	{
		(*labels)[i] = h_labels[i];
	}

	free(h_labels);
}

void cudaKernel::update_labels(int *updateLabels, int numLabels, int deviceId)
{
	runCuda(hipMemcpy(gpuCapabilities[deviceId].d_labels, updateLabels, sizeof(int)*numLabels, hipMemcpyHostToDevice));
}

void cudaKernel::update_labels_to_all_other_devices(int srcDeviceId)
{
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		bool needToFree;
		device_to_device_copy(srcDeviceId, iter->first, numLabels, gpuCapabilities[srcDeviceId].d_labels, iter->second.d_labels, true, needToFree);
	}
}

void cudaKernel::update_labels_to_all_other_devices(int srcDeviceId, hipStream_t& stream)
{
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		bool needToFree;
		device_to_device_copy(srcDeviceId, iter->first, numLabels, gpuCapabilities[srcDeviceId].d_labels, iter->second.d_labels, true, needToFree, stream);
	}
}

void cudaKernel::update_sub_labels_to_all_other_devices(int srcDeviceId)
{
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		bool needToFree;
		device_to_device_copy(srcDeviceId, iter->first, numLabels, gpuCapabilities[srcDeviceId].d_sub_labels, iter->second.d_sub_labels, true, needToFree);
	}
}

void cudaKernel::update_sub_labels_to_all_other_devices(int srcDeviceId, hipStream_t& stream)
{
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		bool needToFree;
		device_to_device_copy(srcDeviceId, iter->first, numLabels, gpuCapabilities[srcDeviceId].d_sub_labels, iter->second.d_sub_labels, true, needToFree, stream);
	}
}

void cudaKernel::update_labels_by_max_index(double* parr, int dim, hipStream_t& stream, int deviceId)
{
	update_labels_by_max_index_kernel << <blocks, threads, 0, stream >> > (parr, gpuCapabilities[deviceId].d_labels, numLabels, dim);

	update_labels_to_all_other_devices(deviceId, stream);
}


void cudaKernel::remove_empty_clusters_worker(int limit)
{
	int deviceId = pick_first_device();
	remove_empty_clusters_worker_all << <blocks, threads >> > (gpuCapabilities[deviceId].d_labels, numLabels, limit);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	update_labels_to_all_other_devices(deviceId);
}

void cudaKernel::split_cluster_local_worker(LabelType index, LabelType newIndex)
{
	int deviceId = pick_first_device();
	split_cluster_local_worker_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, gpuCapabilities[deviceId].d_sub_labels, index, newIndex);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	update_labels_to_all_other_devices(deviceId);
	update_sub_labels_to_all_other_devices(deviceId);
}

void cudaKernel::merge_clusters_worker(LabelType index, LabelType newIndex)
{
	int deviceId = pick_first_device();
	merge_clusters_worker_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, gpuCapabilities[deviceId].d_sub_labels, index, newIndex);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	update_labels_to_all_other_devices(deviceId);
	update_sub_labels_to_all_other_devices(deviceId);
}

void cudaKernel::reset_bad_clusters_worker(LabelType index)
{
	int deviceId = pick_first_device();

	reset_bad_clusters_worker_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, gpuCapabilities[deviceId].d_sub_labels, index);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	update_sub_labels_to_all_other_devices(deviceId);
}

void cudaKernel::get_sub_labels_count(int &l, int &r)
{
	int deviceId = pick_first_device();
	int *d_l;
	runCuda(hipMalloc((void **)&d_l, sizeof(int)));
	runCuda(hipMemset(d_l, 0, sizeof(int)));
	
	int *d_r;
	runCuda(hipMalloc((void **)&d_r, sizeof(int)));
	runCuda(hipMemset(d_r, 0, sizeof(int)));

	get_sub_labels_count_all << <blocks, threads >> > (gpuCapabilities[deviceId].d_sub_labels, numLabels, d_l, d_r);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	runCuda(hipMemcpy(&l, d_l, sizeof(int), hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(&r, d_r, sizeof(int), hipMemcpyDeviceToHost));

	runCuda(hipFree(d_l));
	runCuda(hipFree(d_r));
}

// C(m,k) = A(m,n) * B(n,k)
void cudaKernel::do_matrixMultiply1(double* d_A, double* d_B, double* d_C, int m, int n, int k, hipStream_t& stream, bool use_verbose)
{
	const int BlockSize = 16;

	unsigned int grid_rows = (m + BlockSize - 1) / BlockSize;
	unsigned int grid_cols = (k + BlockSize - 1) / BlockSize;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BlockSize, BlockSize);

	if (k > 0)
	{
		gpu_matrix_mult << <dimGrid, dimBlock, 0, stream >> > (d_A, d_B, d_C, m, n, k);
		runCuda(hipPeekAtLastError());
	}
}

// C(m,k) = A(m,n) * B(n,k)
void cudaKernel::do_matrixMultiply2(double* d_A, double* d_B, double* d_C, int m, int n, int k, hipStream_t& stream, bool use_verbose)
{
	hipblasHandle_t handle;
	runCuda(hipblasCreate(&handle));
	runCuda(hipblasSetStream(handle, stream));
	double alpha = 1.0;
	double beta = 0.0;
	runCuda(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, k, n, &alpha, d_A, m, d_B, n, &beta, d_C, m));

	runCuda(hipblasDestroy(handle));
}

void cudaKernel::dcolwise_dot_all_sub_labels(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, int r_offset, hipStream_t& stream)
{
	dcolwise_dot_all_kernel << <blocks, threads, 0, stream >> > (maxIdx, rows, d_a, d_b, scalar, d_r, r_offset);
}

void cudaKernel::dcolwise_dot_all_labels(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, double weight, hipStream_t& stream)
{
	dcolwise_dot_with_log_kernel << <blocks, threads, 0, stream >> > (maxIdx, rows, d_a, d_b, scalar, d_r, weight);
}

void cudaKernel::create_subclusters_labels(int numClusters, std::vector<std::shared_ptr<thin_cluster_params>>& cluster_params, int dim)
{
	subclusters_labels_plan* plan = new subclusters_labels_plan[numClusters];

	//Allocate memory for all streams
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel
	for (int i = 0; i < numClusters; i++)
	{
		plan[i].deviceId = pick_any_device();
		runCuda(hipStreamCreate(&(plan[i].stream)));
		runCuda(hipMallocAsync((void**)&(plan[i].d_indices), sizeof(int) * numLabels, plan[i].stream));

		//Both
		runCuda(hipMallocAsync((void**)&(plan[i].d_lr_weights), sizeof(double) * cluster_params[i]->lr_weights.size(), plan[i].stream));
		runCuda(hipMemcpyAsync(plan[i].d_lr_weights, cluster_params[i]->lr_weights.data(), sizeof(double) * cluster_params[i]->lr_weights.size(), hipMemcpyHostToDevice, plan[i].stream));
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel
	for (int i = 0; i < numClusters; i++)
	{
		//Find indices
		//Can be used on any GPU
		sample_sub_clusters_worker(i + 1, plan[i].d_indices, plan[i].indicesSize, plan[i].stream, plan[i].deviceId);

		//Return the likelihood in r vector.
		//Can be used on any GPU
		runCuda(hipMallocAsync((void**)&(plan[i].d_r), sizeof(double) * plan[i].indicesSize * 2, plan[i].stream));

		log_likelihood_sub_labels(plan[i].d_r, 0, plan[i].d_indices, plan[i].indicesSize, dim, cluster_params[i]->l_dist, plan[i].stream, plan[i].deviceId);
		log_likelihood_sub_labels(plan[i].d_r, plan[i].indicesSize, plan[i].d_indices, plan[i].indicesSize, dim, cluster_params[i]->r_dist, plan[i].stream, plan[i].deviceId);
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		runCuda(hipStreamSynchronize(plan[i].stream));
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[0].deviceId);

		int* d_indices;
		bool needToFree_d_indices;
		device_to_device_copy(plan[i].deviceId, plan[0].deviceId, plan[i].indicesSize, plan[i].d_indices, d_indices, false, needToFree_d_indices);

		double* d_lr_weights;
		bool needToFree_d_lr_weights;
		device_to_device_copy(plan[i].deviceId, plan[0].deviceId, (int)cluster_params[i]->lr_weights.size(), plan[i].d_lr_weights, d_lr_weights, false, needToFree_d_lr_weights);

		double* d_r;
		bool needToFree_d_r;
		device_to_device_copy(plan[i].deviceId, plan[0].deviceId, plan[i].indicesSize * 2, plan[i].d_r, d_r, false, needToFree_d_r);

		sample_log_cat_array_sub_cluster(d_r, plan[i].indicesSize, d_indices, plan[i].indicesSize, d_lr_weights, plan[0].stream, plan[0].deviceId);

		if (needToFree_d_indices)
		{
			runCuda(hipFreeAsync(d_indices, plan[0].stream));
		}
		if (needToFree_d_lr_weights)
		{
			runCuda(hipFreeAsync(d_lr_weights, plan[0].stream));
		}
		if (needToFree_d_r)
		{
			runCuda(hipFreeAsync(d_r, plan[0].stream));
		}
	}

	//Wait for all operations to finish
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipStreamSynchronize(plan[i].stream));
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipFreeAsync(plan[i].d_indices, plan[i].stream));
		runCuda(hipFreeAsync(plan[i].d_r, plan[i].stream));

		runCuda(hipFreeAsync(plan[i].d_lr_weights, plan[i].stream));

		runCuda(hipStreamDestroy(plan[i].stream));
	}
	delete[]plan;
}

void cudaKernel::create_clusters_labels(int numClusters, std::vector<std::shared_ptr<thin_cluster_params>>& cluster_params, std::vector<double>& weights, bool bFinal)
{
	int masterDevice = -1;
	clusters_labels_plan* plan = new clusters_labels_plan[numClusters];

	//Allocate memory for all streams
	for (int i = 0; i < numClusters; i++)
	{
		plan[i].deviceId = pick_any_device();

		runCuda(hipStreamCreate(&(plan[i].stream)));

		if (i == 0)
		{
			masterDevice = plan[i].deviceId;
			runCuda(hipMalloc((void**)&(plan[i].d_r), sizeof(double) * numLabels * numClusters));
		}
		else if (masterDevice != plan[i].deviceId)
		{
			runCuda(hipMalloc((void**)&(plan[i].d_r), sizeof(double) * numLabels));
		}
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		if (masterDevice == plan[i].deviceId)
		{
			log_likelihood_labels(plan[0].d_r + i * numLabels, weights[i], cluster_params[i]->cluster_dist, plan[i].stream, plan[i].deviceId);
		}
		else
		{
			log_likelihood_labels(plan[i].d_r, weights[i], cluster_params[i]->cluster_dist, plan[i].stream, plan[i].deviceId);
		}
	}

	//Wait for all operations to finish
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipStreamSynchronize(plan[i].stream));
	}

	//Copy d_r from all streams and GPU to one
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		if (masterDevice != plan[i].deviceId)
		{
			double* r = new double[numLabels];

			hipSetDevice(plan[i].deviceId);
			runCuda(hipMemcpy(r, plan[i].d_r, numLabels * sizeof(double), hipMemcpyDeviceToHost));

			hipSetDevice(plan[0].deviceId);
			runCuda(hipMemcpy(plan[0].d_r + i * numLabels, r, numLabels * sizeof(double), hipMemcpyHostToDevice));

			delete []r;
		}
	}

	hipSetDevice(plan[0].deviceId);
	if (bFinal)
	{
		update_labels_by_max_index(plan[0].d_r, numClusters, plan[0].stream, plan[0].deviceId);
	}
	else
	{
		sample_log_cat_array(plan[0].d_r, numClusters, plan[0].stream, plan[0].deviceId);
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipStreamSynchronize(plan[i].stream));
		runCuda(hipStreamDestroy(plan[i].stream));
		if (i == 0 || masterDevice != plan[i].deviceId)
		{
			runCuda(hipFree(plan[i].d_r));
		}
	}

	delete[]plan;
}

void cudaKernel::checkCUDAError(hipError_t err, const char* file, int line)
{
	if (hipSuccess != err)
	{
		printf("Cuda error: %s(%d):%s.\n", file, line, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void cudaKernel::checkCUDAError(hipblasStatus_t err, const char* file, int line)
{
	if (HIPBLAS_STATUS_SUCCESS != err)
	{
		printf("Cuda error: %s(%d):%d.\n", file, line, err);
		exit(EXIT_FAILURE);
	}
}

template<typename T>
void cudaKernel::device_to_device_copy(int srcDeviceId, int trgDeviceId, int dataSize, T* srcData, T*& trgData, bool alreadyAllocated, bool& needToFree, hipStream_t& stream)
{
	if (srcDeviceId != trgDeviceId)
	{
		runCuda(hipStreamSynchronize(stream));
	}
	device_to_device_copy(srcDeviceId, trgDeviceId, dataSize, srcData, trgData, alreadyAllocated, needToFree);
}

template<typename T>
void cudaKernel::device_to_device_copy(int srcDeviceId, int trgDeviceId, int dataSize, T* srcData, T* &trgData, bool alreadyAllocated, bool &needToFree)
{
	needToFree = false;

	if (srcDeviceId == trgDeviceId)
	{
		trgData = srcData;
	}
	else
	{
		T* data = new T[dataSize];

		hipSetDevice(srcDeviceId);
		runCuda(hipMemcpy(data, srcData, dataSize * sizeof(T), hipMemcpyDeviceToHost));

		hipSetDevice(trgDeviceId);

		if (!alreadyAllocated)
		{
			runCuda(hipMalloc((void**)&trgData, dataSize * sizeof(T)));
			needToFree = true;
		}

		runCuda(hipMemcpy(trgData, data, dataSize * sizeof(T), hipMemcpyHostToDevice));

		delete[]data;
	}
}

void cudaKernel::sum_rowwise(double* d_A, double* d_B, int rows, int cols, hipStream_t& stream)
{
	dim3 blocks_size = dim3(rows / threads.x + 1);

	sum_rowwise_kernel << <blocks_size, threads, 0, stream >> > (d_A, d_B, rows, cols);
	runCuda(hipPeekAtLastError());
}

void cudaKernel::inverse_matrix(const MatrixXd &A, MatrixXd& B)
{
	int blocksize = 8;
	int n = (int)(A.rows());
	int ddsize = n * n * sizeof(double);
	double* d_A, * d_B, *I;

	I = new double[n * n];

	for (int i = 0; i < n; i++) 
	{
		for (int j = 0; j < n; j++) {
			if (i == j) I[i * n + i] = 1.0;
			else I[i * n + j] = 0.0;
		}
	}

	dim3 threadsPerBlock(blocksize, blocksize);
	dim3 numBlocks((n + blocksize - 1) / blocksize, (n + blocksize - 1) / blocksize);
	runCuda(hipMalloc((void**)&d_A, ddsize));
	runCuda(hipMemcpy(d_A, A.data(), ddsize, hipMemcpyHostToDevice));

	runCuda(hipMalloc((void**)&d_B, ddsize));
	runCuda(hipMemcpy(d_B, I, ddsize, hipMemcpyHostToDevice));

	for (int i = 0; i < n; i++) 
	{
		nodiag_normalize << <numBlocks, threadsPerBlock >> > (d_A, d_B, n, i);
		diag_normalize << <numBlocks, threadsPerBlock >> > (d_A, d_B, n, i);
		gaussjordan << <numBlocks, threadsPerBlock >> > (d_A, d_B, n, i);
		set_zero << <numBlocks, threadsPerBlock >> > (d_A, d_B, n, i);
	}

	B.resize(n, n);
	runCuda(hipDeviceSynchronize());

	runCuda(hipMemcpy(B.data(), d_B, ddsize, hipMemcpyDeviceToHost));

	runCuda(hipFree(d_A));
	runCuda(hipFree(d_B));
}

#endif