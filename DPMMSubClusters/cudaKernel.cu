#include "hip/hip_runtime.h"
#ifndef CudaKernel_CU
#define CudaKernel_CU

//#pragma warning( disable : 2886 )
//#pragma warning( disable : 2929)

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>

#include<time.h>
#include "cudaKernel.cuh"
#include "distributions/mv_gaussian.h"

// function to define seed
__global__ void initCurand(hiprandState *state, unsigned long long seed, int maxIdx) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		hiprand_init(seed, idx, 0, &state[idx]);
	}
}

//__device__ void sample_by_probability(hiprandState *state, double *weight, int numClusters, int rows, int idx, int *index)
//{
//	int cluster;
//	double u;
//	do 
//	{
//		cluster = (int)(hiprand_uniform(state) * (numClusters - 0.00001));
//		u = hiprand_uniform(state);
//	} while (u > weight[cluster*rows+idx]);
//	*index = cluster + 1;
//}

__device__ void sample_by_probability(hiprandState *state, double *weight, int numClusters, int rows, int idx, int *index,
									  double *y, int *a, int *b)
{
	int i;
	int j;
	int k;
	int n = numClusters;

	
//	if (n > 1)
	{
		a[idx] = 0;
		for (i = 1; i <= n; i++)
		{
			a[i*rows + idx] = i;
		}
		a[(n+1)*rows + idx] = n + 1;

		b[idx] = 0;
		for (i = 1; i <= n; i++)
		{
			b[i*rows + idx] = i;
		}
		b[(n+1)*rows + idx] = n + 1;
		/*
		  Copy Y from X.
		  Scale the probability vector and set sentinel values at the ends.
		*/
		y[idx] = 0.0;
		for (i = 1; i <= n; i++)
		{
			y[i*rows + idx] = weight[(i-1)*rows + idx] * (double)(n);
		}
		y[(n+1)*rows + idx] = 2.0;

		i = 0;
		j = n + 1;
		for (; ; )
		{
			/*
			  Find i so Y[B[i]] needs more.
			*/
			do
			{
				i++;
			} while (y[b[i*rows + idx]*rows + idx] < 1.0);
			/*
				  Find j so Y[B[j]] wants less.
				*/
			do
			{
				j--;
			} while (1.0 <= y[b[j*rows + idx]*rows + idx]);

			if (j <= i)
			{
				break;
			}
			/*
			  Swap B[i] and B[j].
			*/
			k = b[i*rows + idx];
			b[i*rows + idx] = b[j*rows + idx];
			b[j*rows + idx] = k;
		}

		i = j;
		j++;

		while (0 < i)
		{
			/*
			  Find J such that Y[B[j]] needs more.
			*/
			while (y[b[j*rows + idx]*rows + idx] <= 1.0)
			{
				j++;
			}
			/*
			  Meanwhile, Y[B[i]] wants less.
			*/
			if (n < j)
			{
				break;
			}
			/*
			  B[i] will donate to B[j] to fix up.
			*/
			y[b[j*rows + idx]*rows + idx] = y[b[j*rows + idx]*rows + idx] - (1.0 - y[b[i*rows + idx]*rows + idx]);
			a[b[i*rows + idx]*rows + idx] = b[j*rows + idx];
			/*
			  Y[B[j]] now wants less so readjust ordering.
			*/
			if (y[b[j*rows + idx]*rows + idx] < 1.0)
			{
				k = b[i*rows + idx];
				b[i*rows + idx] = b[j*rows + idx];
				b[j*rows + idx] = k;
				j++;
			}
			else
			{
				i--;
			}
		}

		double r;
		/*
		  Let i = random uniform integer from {1,2,...N};
		*/
		i = 1 + (int)(n * hiprand_uniform(state));
		//for (int j = 0; j < 2; j++)
		{
			r = hiprand_uniform(state);

			if (y[i*rows + idx] < r)
			{
				i = a[i*rows + idx];
//				break;
			}
		}
		*index = i;
//		*index = 1;
	}
//	else
	{
//		*index = 1;
	}
}

//__device__ void sample_by_probability(hiprandState *state, double *weight, int numClusters, int rows, int idx, int *index)
//{
//	//do
//	{
//		int cluster1 = 0;
//		int cluster2 = 0;
//		if (numClusters > 1)
//		{
//			cluster2 = 1;
//		}
//
//		if (weight[cluster1*rows + idx] > weight[cluster2*rows + idx])
//		{
//			*index = cluster1 + 1;
//		}
//		else
//		{
//			*index = cluster2 + 1;
//
//		}
//	}
//}

__global__ void sample_log_cat_array_all(hiprandState *state, int *dev_sample, int maxIdx, int numClusters, double *d_log_likelihood_array, double *y, int *a, int *b)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_by_probability(&state[idx], d_log_likelihood_array, numClusters, maxIdx, idx, &dev_sample[idx], y, a, b);
	}
}

__global__ void sample_log_cat_array_sub_cluster_all(hiprandState *state, int *dev_sample, int* d_maxIdx, int num, double *d_log_likelihood_array, int *indices, double *y, int *a, int *b)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < *d_maxIdx)
	{
		sample_by_probability(&state[indices[idx]], d_log_likelihood_array, num, *d_maxIdx, idx, &dev_sample[indices[idx]], y, a, b);
	}
}

__device__ void sample_sub_label(hiprandState *state, int *d_label)
{
	*d_label = ((int)(hiprand_uniform(state) * 2)) % 2 + 1;
}

__global__ void sample_sub_labels_all(hiprandState *state, int *d_labels, int maxIdx)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_sub_label(&state[idx], &d_labels[idx]);
	}
}

__device__ void sample_label(hiprandState *state, int *d_label, int initial_clusters, double outlier_mod)
{
	*d_label = ((int)(hiprand_uniform(state)*initial_clusters)) % initial_clusters + 1 + ((outlier_mod > 0) ? 1 : 0);
}

__global__ void sample_labels_all(hiprandState *state, int *d_labels, int maxIdx, int initial_clusters, double outlier_mod)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_label(&state[idx], &d_labels[idx], initial_clusters, outlier_mod);
	}
}

__device__ void remove_empty_clusters_worker(int *d_label, int limit)
{
	if (*d_label > limit)
	{
		*d_label -= 1;
	}
}

__global__ void remove_empty_clusters_worker_all(int *d_labels, int maxIdx, int limit)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		remove_empty_clusters_worker(&d_labels[idx], limit);
	}
}

__global__ void find_indices(int *d_labels, int maxIdx, int label, int *d_indices, int *d_indicesSize)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		if (d_labels[idx] == label)
		{
			int curIndex = atomicAdd(d_indicesSize, 1);
			d_indices[curIndex] = idx;
		}
	}
}

__device__ void split_cluster_local_worker(hiprandState *state, int *d_labels, int *d_sub_label, int index, int newIndex)
{
	if (*d_labels == index + 1)
	{
		if (*d_sub_label == 2)
		{
			*d_labels = newIndex + 1;
		}
		*d_sub_label = hiprand(state) % 2 + 1;
	}
}

__global__ void split_cluster_local_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index, int newIndex)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		split_cluster_local_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index, newIndex);
	}
}

__device__ void merge_clusters_worker(hiprandState *state, int *d_labels, int *d_sub_labels, int index, int newIndex)
{
	if (*d_labels == index + 1)
	{
		*d_sub_labels = 1;
	}
	if (*d_labels == newIndex + 1)
	{
		*d_sub_labels = 2;
	}
	if (*d_labels == newIndex + 1)
	{
		*d_labels = index + 1;
	}
}

__global__ void merge_clusters_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index, int newIndex)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		merge_clusters_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index, newIndex);
	}
}

__device__ void reset_bad_clusters_worker(hiprandState *state, int *d_labels, int *d_sub_labels, int index)
{
	if (*d_labels == index + 1)
	{
		*d_sub_labels = hiprand(state) % 2 + 1;
	}
}

__global__ void reset_bad_clusters_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		reset_bad_clusters_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index);
	}
}

__global__ void get_sub_labels_count_all(int *d_sub_labels, int maxIdx, int *l, int *r)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		if (d_sub_labels[idx] == 1)
		{
			atomicAdd(l, 1);
		}
		else if (d_sub_labels[idx] == 2)
		{
			atomicAdd(r, 1);
		}
	}
}

__global__ void create_suff_stats_dict_worker_all(
	int* d_sub_labels,
	int maxIdx,
	int *d_indices,
	int *d_indicesSize,
	double *group_pts,
	int group_pts_rows,
	double *pts,
	double *pts1,
	double *pts2,
	int *d_j1,
	int *d_j2)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < *d_indicesSize)
	{
		if (d_sub_labels[d_indices[idx]] == 1)
		{
			int curj1 = atomicAdd(d_j1, 1);
			for (int i = 0; i < group_pts_rows; i++)
			{
				double pt = group_pts[IDX2C(i, d_indices[idx], group_pts_rows)];
				pts[IDX2C(i, idx, group_pts_rows)] = pt;
				pts1[IDX2C(i, curj1, group_pts_rows)] = pt;
			}
		}
		else if (d_sub_labels[d_indices[idx]] == 2)
		{
			int curj2 = atomicAdd(d_j2, 1);
			for (int i = 0; i < group_pts_rows; i++)
			{
				double pt = group_pts[IDX2C(i, d_indices[idx], group_pts_rows)];
				pts[IDX2C(i, idx, group_pts_rows)] = pt;
				pts2[IDX2C(i, curj2, group_pts_rows)] = pt;
			}
		}
	}
}

__global__ void dcolwise_dot_all_kernel(int* d_maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, int* d_r_offset)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < *d_maxIdx)
	{
		double sum = 0;
		for (int i = 0; i < rows; i++)
		{
			sum += d_a[idx * rows + i] * d_b[idx * rows + i];
		}

		d_r[idx + *d_r_offset] = scalar - sum / 2;
	}
}

__global__ void dcolwise_dot_with_log_kernel(int* d_maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, double weight)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < *d_maxIdx)
	{
		double sum = 0;
		for (int i = 0; i < rows; i++)
		{
			sum += d_a[idx * rows + i] * d_b[idx * rows + i];
		}

		d_r[idx] = scalar - sum / 2 + log(weight);
	}
}

__global__ void build_log_likelihood_array_sub_cluster_kernel(int maxIdx, double* d_log_likelihood_array, double* d_lr_weights, int rows, int cols)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double maxRow = 0;
		bool first = true;
		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] += log(d_lr_weights[j]);
			if (first || d_log_likelihood_array[IDX2C(idx, j, rows)] > maxRow)
			{
				first = false;
				maxRow = d_log_likelihood_array[IDX2C(idx, j, rows)];
			}
		}

		double sum = 0;
		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = exp(d_log_likelihood_array[IDX2C(idx, j, rows)] - maxRow);
			sum += d_log_likelihood_array[IDX2C(idx, j, rows)];
		}

		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = d_log_likelihood_array[IDX2C(idx, j, rows)] / sum;
		}
	}
}

__global__ void build_log_likelihood_array_sub_cluster_kernel_v2(int *d_maxIdx, double* d_r, int* d_r_offset_r, double* d_lr_weights)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < *d_maxIdx)
	{
		double maxRow;
		double sum;

		d_r[idx] += log(d_lr_weights[0]);
		d_r[idx + *d_r_offset_r] += log(d_lr_weights[1]);
		maxRow = fmax(d_r[idx], d_r[idx + *d_r_offset_r]);

		d_r[idx] = exp(d_r[idx] - maxRow);
		d_r[idx + *d_r_offset_r] = exp(d_r[idx + *d_r_offset_r] - maxRow);
		sum = d_r[idx] + d_r[idx + *d_r_offset_r];

		d_r[idx] = d_r[idx] / sum;
		d_r[idx + *d_r_offset_r] = d_r[idx + *d_r_offset_r] / sum;
	}
}

__global__ void build_log_likelihood_array_kernel(int maxIdx, double* d_log_likelihood_array, int rows, int cols)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double maxRow = 0;
		bool first = true;
		for (int j = 0; j < cols; j++)
		{
			if (first || d_log_likelihood_array[IDX2C(idx, j, rows)] > maxRow)
			{
				first = false;
				maxRow = d_log_likelihood_array[IDX2C(idx, j, rows)];
			}
		}

		double sum = 0;
		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = exp(d_log_likelihood_array[IDX2C(idx, j, rows)] - maxRow);
			sum += d_log_likelihood_array[IDX2C(idx, j, rows)];
		}

		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = d_log_likelihood_array[IDX2C(idx, j, rows)] / sum;
		}
	}
}

__global__ void update_labels_by_max_index_kernel(double* parr, int* d_labels, int maxIdx, int dim)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		double max = 0;
		bool first = true;
		int max_index = 0;
		for (int i = 0; i < dim; ++i)
		{
			if (first || max < parr[IDX2C(idx, i, maxIdx)])
			{
				first = false;
				max = parr[IDX2C(idx, i, maxIdx)];
				max_index = i;
			}
		}
		d_labels[idx] = max_index + 1;
	}
}

void cudaKernel::init(int numLabelsIn, MatrixXd &points, unsigned long long seed)
{
	printf("Init cuda\n");

//	hipSetDevice(1);
	runCuda(hipDeviceReset());
	numLabels = numLabelsIn;
	threads = dim3(512);
	blocks = dim3(numLabels / threads.x + 1);
	runCuda(hipMalloc((void**)&devState, numLabels * sizeof(hiprandState)));
	initCurand << <blocks, threads >> > (devState, seed, numLabels);
	runCuda(hipPeekAtLastError());
	runCuda(hipPeekAtLastError());

	runCuda(hipMalloc((void**)&d_labels, numLabels * sizeof(int)));
	runCuda(hipMalloc((void**)&d_sub_labels, numLabels * sizeof(int)));
	runCuda(hipMalloc((void**)&d_points, points.size() * sizeof(double)));
	runCuda(hipMemcpy(d_points, points.data(), points.size() * sizeof(double), hipMemcpyHostToDevice));
}

void cudaKernel::release()
{
	printf("Release cuda\n");

	if (devState != NULL)
	{
		runCuda(hipFree(devState));
	}

	if (d_labels != NULL)
	{
		runCuda(hipFree(d_labels));
	}

	if (d_sub_labels != NULL)
	{
		runCuda(hipFree(d_sub_labels));
	}

	if (d_points != NULL)
	{
		runCuda(hipFree(d_points));
	}
}

int cudaKernel::sample_log_cat_array_sub_cluster(LabelType *indices, int labelsSize, Eigen::MatrixXd &log_likelihood_array, std::vector<double>& lr_weights)
{
	int *d_indices;
	double *d_y;
	int *d_a;
	int *d_b;
	double* d_lr_weights;
	int* d_labelsSize;
	
	runCuda(hipMalloc((void**)&d_indices, sizeof(int) * labelsSize));
	runCuda(hipMemcpy(d_indices, indices, sizeof(int) * labelsSize, hipMemcpyHostToDevice));
	runCuda(hipMalloc((void**)&d_lr_weights, sizeof(double) * lr_weights.size()));
	runCuda(hipMemcpy(d_lr_weights, lr_weights.data(), sizeof(double) * lr_weights.size(), hipMemcpyHostToDevice));

	dim3 blocks_size = dim3(labelsSize / threads.x + 1);
	
	int n = log_likelihood_array.size();
	double *d_log_likelihood_array;
	runCuda(hipMalloc((void **)&d_log_likelihood_array, sizeof(double)*n));
	runCuda(hipMemcpy(d_log_likelihood_array, log_likelihood_array.data(), sizeof(double)*n, hipMemcpyHostToDevice));
	runCuda(hipMalloc((void **)&d_y, sizeof(double)*labelsSize*(log_likelihood_array.cols() + 2)));
	runCuda(hipMalloc((void **)&d_a, sizeof(int)*labelsSize*(log_likelihood_array.cols() + 2)));
	runCuda(hipMalloc((void **)&d_b, sizeof(int)*labelsSize*(log_likelihood_array.cols() + 2)));

	runCuda(hipMalloc((void**)&d_labelsSize, sizeof(int)));
	runCuda(hipMemcpy(d_labelsSize, &labelsSize, sizeof(int), hipMemcpyHostToDevice));

	//	int dev = 0;//GPU index. need to loop for all GPUs

	build_log_likelihood_array_sub_cluster_kernel << <blocks_size, threads >> > (log_likelihood_array.rows(), d_log_likelihood_array, d_lr_weights, log_likelihood_array.rows(), log_likelihood_array.cols());
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
	
	sample_log_cat_array_sub_cluster_all << <blocks_size, threads >> > (devState, d_sub_labels, d_labelsSize, log_likelihood_array.cols(), d_log_likelihood_array, d_indices, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	//free memory
	runCuda(hipFree(d_indices));
	runCuda(hipFree(d_lr_weights));
	runCuda(hipFree(d_log_likelihood_array));
	runCuda(hipFree(d_y));
	runCuda(hipFree(d_a));
	runCuda(hipFree(d_b));
	runCuda(hipFree(d_labelsSize));

	return 0;
}

void cudaKernel::sample_log_cat_array_sub_cluster_v2(
	double* d_r,
	int* d_r_offset,
	double* d_y,
	int* d_a,
	int* d_b,
	int* d_indices,
	int* d_indicesSize,
	double* d_lr_weights,
	hipStream_t& stream)
{
	build_log_likelihood_array_sub_cluster_kernel_v2 << <blocks, threads, 0, stream >> > (d_indicesSize, d_r, d_r_offset, d_lr_weights);
	runCuda(hipPeekAtLastError());

	//TODO - Can we remove d_y, d_a, d_b?
	sample_log_cat_array_sub_cluster_all << <blocks, threads, 0, stream >> > (devState, d_sub_labels, d_indicesSize, 2, d_r, d_indices, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());
}

void cudaKernel::sample_log_cat_array_v2(
	double* d_r,
	double* d_y,
	int* d_a,
	int* d_b,
	int dim,
	hipStream_t& stream)
{
	build_log_likelihood_array_kernel << <blocks, threads, 0, stream >> > (numLabels, d_r, numLabels, dim);
	runCuda(hipPeekAtLastError());

	sample_log_cat_array_all << <blocks, threads, 0, stream >> > (devState, d_labels, numLabels, dim, d_r, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());
}

int cudaKernel::sample_log_cat_array(Eigen::MatrixXd &log_likelihood_array)
{
	int n = log_likelihood_array.size();
	double *d_log_likelihood_array;
	double *d_y;
	int *d_a;
	int *d_b;

	runCuda(hipMalloc((void **)&d_log_likelihood_array, sizeof(double)*n));
	runCuda(hipMemcpy(d_log_likelihood_array, log_likelihood_array.data(), sizeof(double)*n, hipMemcpyHostToDevice));

	runCuda(hipMalloc((void **)&d_y, sizeof(double)*numLabels*(log_likelihood_array.cols() + 2)));
	runCuda(hipMalloc((void **)&d_a, sizeof(int)*numLabels*(log_likelihood_array.cols() + 2)));
	runCuda(hipMalloc((void **)&d_b, sizeof(int)*numLabels*(log_likelihood_array.cols() + 2)));

//	int dev = 0;//GPU index. need to loop for all GPUs
	build_log_likelihood_array_kernel << <blocks, threads >> > (log_likelihood_array.rows(), d_log_likelihood_array, log_likelihood_array.rows(), log_likelihood_array.cols());
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	sample_log_cat_array_all <<<blocks, threads>>>(devState, d_labels, numLabels, log_likelihood_array.cols(), d_log_likelihood_array, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	//free memory
	runCuda(hipFree(d_log_likelihood_array));
	runCuda(hipFree(d_y));
	runCuda(hipFree(d_a));
	runCuda(hipFree(d_b));
	return 0;

}

void cudaKernel::sample_sub_clusters_worker(LabelType label, LabelType* &indices, LabelType &indicesSize)
{
	int *d_indices;
	runCuda(hipMalloc((void **)&d_indices, sizeof(int)*numLabels));

	int *d_indicesSize;
	runCuda(hipMalloc(&d_indicesSize, sizeof(int)));
	runCuda(hipMemset(d_indicesSize, 0, sizeof(int)));

	find_indices << <blocks, threads >> > (d_labels, numLabels, label, d_indices, d_indicesSize);

	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	runCuda(hipMemcpy(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(indices, d_indices, indicesSize * sizeof(int), hipMemcpyDeviceToHost));

	runCuda(hipFree(d_indicesSize));
	runCuda(hipFree(d_indices));
}

void cudaKernel::sample_sub_clusters_worker_v2(LabelType label, int* d_indices, int* d_indicesSize, hipStream_t& stream)
{
	runCuda(hipMemsetAsync(d_indicesSize, 0, sizeof(int), stream));

	find_indices << <blocks, threads, 0, stream >> > (d_labels, numLabels, label, d_indices, d_indicesSize);

	runCuda(hipPeekAtLastError());
}

void cudaKernel::create_suff_stats_dict_worker(
	LabelType label,
	LabelType &indicesSize,
	Eigen::MatrixXd &group_pts,
	Eigen::MatrixXd* &pts,
	Eigen::MatrixXd* &pts1,
	Eigen::MatrixXd* &pts2)
{
	int *d_indices;
	runCuda(hipMalloc((void **)&d_indices, sizeof(int)*numLabels));

	int *d_indicesSize;
	runCuda(hipMalloc(&d_indicesSize, sizeof(int)));
	runCuda(hipMemset(d_indicesSize, 0, sizeof(int)));

	find_indices << <blocks, threads >> > (d_labels, numLabels, label, d_indices, d_indicesSize);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
	runCuda(hipMemcpy(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost));

	double *d_group_pts;
	runCuda(hipMalloc((void **)&d_group_pts, sizeof(double)*group_pts.size()));
	runCuda(hipMemcpy(d_group_pts, group_pts.data(), sizeof(double)*group_pts.size(), hipMemcpyHostToDevice));

	double *d_pts;
	runCuda(hipMalloc((void **)&d_pts, sizeof(double)*group_pts.rows()*indicesSize));

	double *d_pts1;
	runCuda(hipMalloc((void **)&d_pts1, sizeof(double)*group_pts.rows()*indicesSize));

	double *d_pts2;
	runCuda(hipMalloc((void **)&d_pts2, sizeof(double)*group_pts.rows()*indicesSize));

	int *d_j1;
	int *d_j2;
	runCuda(hipMalloc(&d_j1, sizeof(int)));
	runCuda(hipMemset(d_j1, 0, sizeof(int)));
	runCuda(hipMalloc(&d_j2, sizeof(int)));
	runCuda(hipMemset(d_j2, 0, sizeof(int)));

	dim3 blocks_size = dim3(indicesSize / threads.x + 1);
	create_suff_stats_dict_worker_all << <blocks_size, threads >> > (
		d_sub_labels,
		numLabels,
		d_indices,
		d_indicesSize,
		d_group_pts,
		group_pts.rows(),
		d_pts,
		d_pts1,
		d_pts2,
		d_j1,
		d_j2);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	int j1;
	int j2;
	runCuda(hipMemcpy(&j1, d_j1, sizeof(int), hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(&j2, d_j2, sizeof(int), hipMemcpyDeviceToHost));

	pts = new Eigen::MatrixXd(group_pts.rows(), indicesSize);
	pts1 = new Eigen::MatrixXd(group_pts.rows(), j1);
	pts2 = new Eigen::MatrixXd(group_pts.rows(), j2);

	runCuda(hipMemcpy(pts->data(), d_pts, sizeof(double)*group_pts.rows()*indicesSize, hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(pts1->data(), d_pts1, sizeof(double)*group_pts.rows()*j1, hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(pts2->data(), d_pts2, sizeof(double)*group_pts.rows()*j2, hipMemcpyDeviceToHost));

	runCuda(hipFree(d_j1));
	runCuda(hipFree(d_j2));
	runCuda(hipFree(d_indicesSize));
	runCuda(hipFree(d_group_pts));
	runCuda(hipFree(d_pts));
	runCuda(hipFree(d_pts1));
	runCuda(hipFree(d_pts2));
}

void cudaKernel::sample_sub_labels()
{
	sample_sub_labels_all << <blocks, threads >> > (devState, d_sub_labels, numLabels);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
}

void cudaKernel::sample_labels(int initial_clusters, double outlier_mod)
{
	sample_labels_all << <blocks, threads >> > (devState, d_labels, numLabels, initial_clusters, outlier_mod);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
}

void cudaKernel::get_sub_labels(LabelsType &subLabels)
{
	int *h_subLabels;
	h_subLabels = (int*)malloc(numLabels * sizeof(int));
	runCuda(hipMemcpy(h_subLabels, d_sub_labels, numLabels * sizeof(int), hipMemcpyDeviceToHost));

	subLabels.resize(numLabels);
	for (size_t i = 0; i < numLabels; i++)
	{
		subLabels[i] = h_subLabels[i];
	}

	free(h_subLabels);
}

void cudaKernel::get_labels(LabelsType &labels)
{
	int *h_labels;
	h_labels = (int*)malloc(numLabels * sizeof(int));
	runCuda(hipMemcpy(h_labels, d_labels, numLabels * sizeof(int), hipMemcpyDeviceToHost));

	labels.resize(numLabels);
	for (size_t i = 0; i < numLabels; i++)
	{
		labels[i] = h_labels[i];
	}

	free(h_labels);
}

void cudaKernel::update_labels(int *updateLabels, int numLabels)
{
	runCuda(hipMemcpy(d_labels, updateLabels, sizeof(int)*numLabels, hipMemcpyHostToDevice));
}

void cudaKernel::update_labels_by_max_index(double* parr, int dim, hipStream_t& stream)
{
	update_labels_by_max_index_kernel << <blocks, threads, 0, stream >> > (parr, d_labels, numLabels, dim);
}


void cudaKernel::remove_empty_clusters_worker(int limit)
{
	remove_empty_clusters_worker_all << <blocks, threads >> > (d_labels, numLabels, limit);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
}

void cudaKernel::split_cluster_local_worker(LabelType index, LabelType newIndex)
{
	split_cluster_local_worker_all << <blocks, threads >> > (devState, d_labels, numLabels, d_sub_labels, index, newIndex);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
}

void cudaKernel::merge_clusters_worker(LabelType index, LabelType newIndex)
{
	merge_clusters_worker_all << <blocks, threads >> > (devState, d_labels, numLabels, d_sub_labels, index, newIndex);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
}

void cudaKernel::reset_bad_clusters_worker(LabelType index)
{
	reset_bad_clusters_worker_all << <blocks, threads >> > (devState, d_labels, numLabels, d_sub_labels, index);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
}

void cudaKernel::get_sub_labels_count(int &l, int &r)
{
	int *d_l;
	runCuda(hipMalloc((void **)&d_l, sizeof(int)));
	runCuda(hipMemset(d_l, 0, sizeof(int)));
	
	int *d_r;
	runCuda(hipMalloc((void **)&d_r, sizeof(int)));
	runCuda(hipMemset(d_r, 0, sizeof(int)));

	get_sub_labels_count_all << <blocks, threads >> > (d_sub_labels, numLabels, d_l, d_r);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	runCuda(hipMemcpy(&l, d_l, sizeof(int), hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(&r, d_r, sizeof(int), hipMemcpyDeviceToHost));

	runCuda(hipFree(d_l));
	runCuda(hipFree(d_r));
}

// C(m,k) = A(m,n) * B(n,k)
void cudaKernel::naive_matrix_multiply(const double* A, const double* B, double* C, int m, int n, int k)
{
	int lda = m, ldb = k, ldc = m;
	const double alf = 1;
	const double bet = 0;
	const double* alpha = &alf;
	const double* beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}

// C(m,k) = A(m,n) * B(n,k)
void cudaKernel::naive_matrix_multiply_v2(const double* A, const double* B, double* C, int m, int* d_n, int k, hipStream_t& stream)
{
	int lda = m, ldb = k, ldc = m;
	const double alf = 1;
	const double bet = 0;
	const double* alpha = &alf;
	const double* beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetStream(handle, stream);
	// Do the actual multiplication
	int n;
	runCuda(hipMemcpyAsync(&n, d_n, sizeof(int), hipMemcpyDeviceToHost, stream));
	//TODO - can we use d_n here?
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}

void cudaKernel::dcolwise_dot_all(int* d_maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, int* d_r_offset)
{
	dcolwise_dot_all_kernel << <blocks, threads>> > (d_maxIdx, rows, d_a, d_b, scalar, d_r, d_r_offset);
}

void cudaKernel::dcolwise_dot_all_v2(int* d_maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, int* d_r_offset, hipStream_t& stream)
{
	dcolwise_dot_all_kernel << <blocks, threads, 0, stream >> > (d_maxIdx, rows, d_a, d_b, scalar, d_r, d_r_offset);
}

void cudaKernel::dcolwise_dot_all_v3(int* d_maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, double weight, hipStream_t& stream)
{
	dcolwise_dot_with_log_kernel << <blocks, threads, 0, stream >> > (d_maxIdx, rows, d_a, d_b, scalar, d_r, weight);
}

//void cudaKernel::dcolwise_dot(Eigen::VectorXd& r, const Eigen::MatrixXd& a, const Eigen::MatrixXd& b)
//{
//	int sizeVec = a.cols();
//
//	double* d_a;
//	double* d_b;
//	double* d_c;
//	double* d_r;
//
//	runCuda(hipMalloc((void**)&d_a, sizeof(double) * a.size()));
//	runCuda(hipMalloc((void**)&d_b, sizeof(double) * b.size()));
//	runCuda(hipMalloc((void**)&d_c, sizeof(double) * b.rows() * a.cols()));
//	runCuda(hipMemcpy(d_a, a.data(), sizeof(double) * a.size(), hipMemcpyHostToDevice));
//	runCuda(hipMemcpy(d_b, b.data(), sizeof(double) * b.size(), hipMemcpyHostToDevice));
//
//	naive_matrix_multiply(d_b, d_a, d_c, b.rows(), a.cols(), b.cols());
//
//	runCuda(hipFree(d_b));
//
//	runCuda(hipMalloc((void**)&d_r, sizeof(double)* sizeVec));
//
//	dcolwise_dot_all(sizeVec, a.rows(), d_a, d_c, d_r);
//
//	r.resize(sizeVec);
//	runCuda(hipMemcpy(r.data(), d_r, sizeof(double)* sizeVec, hipMemcpyDeviceToHost));
//	
//	runCuda(hipFree(d_a));
//	runCuda(hipFree(d_c));
//	runCuda(hipFree(d_r));
//}

typedef struct
{
	//Stream for asynchronous command execution
	hipStream_t stream;

	int* d_indices;

	double* d_r;

	int* d_indicesSize;

	double* d_z_l;
	double* d_z_r;

	double* d_mu_l;
	double* d_mu_r;

	double* d_b_l;
	double* d_b_r;

	double* d_c_l;
	double* d_c_r;

	double* d_lr_weights;
	double* d_y2;
	int* d_a2;
	int* d_b2;

} subclusters_labels_plan;

void cudaKernel::create_subclusters_labels(int numClusters, std::vector<thin_cluster_params*>& cluster_params, int dim)
//LabelType* indices, LabelType indicesSize, distribution_sample* l_dist, distribution_sample* r_dist, std::vector<double> &lr_weights)
{
	subclusters_labels_plan* plan = new subclusters_labels_plan[numClusters];
	int* d_zero;

	//Allocate memory for all streams
	for (int i = 0; i < numClusters; i++)
	{
		runCuda(hipStreamCreate(&(plan[i].stream)));
		runCuda(hipMallocAsync((void**)&d_zero, sizeof(int), plan[i].stream));
		runCuda(hipMemsetAsync(d_zero, 0, sizeof(int), plan[i].stream));
		runCuda(hipMallocAsync((void**)&(plan[i].d_indices), sizeof(int) * numLabels, plan[i].stream));
		runCuda(hipMallocAsync(&(plan[i].d_indicesSize), sizeof(int), plan[i].stream));

		//The following allocation could be used less memory and also later when it's really needed since the upper bound is number of labels but the actual is smaller.
		//However in order to run in stream we needed to allocate all of them in advance so we use the upper bounder.
		//Actual memory that required below is indicesSize instead of numLabels.
		runCuda(hipMallocAsync((void**)&(plan[i].d_r), sizeof(double) * numLabels * 2, plan[i].stream));

		//Left
		const mv_gaussian* ds_l = (mv_gaussian*)(cluster_params[i]->l_dist);
		runCuda(hipMallocAsync((void**)&(plan[i].d_z_l), sizeof(double) * dim * numLabels, plan[i].stream));
		runCuda(hipMallocAsync((void**)&(plan[i].d_mu_l), sizeof(double) * ds_l->mu.size(), plan[i].stream));
		runCuda(hipMallocAsync((void**)&(plan[i].d_b_l), sizeof(double) * ds_l->invSigma.size(), plan[i].stream));
		runCuda(hipMallocAsync((void**)&(plan[i].d_c_l), sizeof(double) * ds_l->invSigma.rows() * numLabels, plan[i].stream));

		//Right
		const mv_gaussian* ds_r = (mv_gaussian*)(cluster_params[i]->r_dist);
		runCuda(hipMallocAsync((void**)&(plan[i].d_z_r), sizeof(double) * dim * numLabels, plan[i].stream));
		runCuda(hipMallocAsync((void**)&(plan[i].d_mu_r), sizeof(double) * ds_r->mu.size(), plan[i].stream));
		runCuda(hipMallocAsync((void**)&(plan[i].d_b_r), sizeof(double) * ds_r->invSigma.size(), plan[i].stream));
		runCuda(hipMallocAsync((void**)&(plan[i].d_c_r), sizeof(double) * ds_r->invSigma.rows() * numLabels, plan[i].stream));

		//Both
		runCuda(hipMallocAsync((void**)&(plan[i].d_lr_weights), sizeof(double) * cluster_params[i]->lr_weights.size(), plan[i].stream));
		runCuda(hipMemcpyAsync(plan[i].d_lr_weights, cluster_params[i]->lr_weights.data(), sizeof(double) * cluster_params[i]->lr_weights.size(), hipMemcpyHostToDevice, plan[i].stream));
		runCuda(hipMallocAsync((void**)&(plan[i].d_y2), sizeof(double) * numLabels * (2 + 2), plan[i].stream));
		runCuda(hipMallocAsync((void**)&(plan[i].d_a2), sizeof(int) * numLabels * (2 + 2), plan[i].stream));
		runCuda(hipMallocAsync((void**)&(plan[i].d_b2), sizeof(int) * numLabels * (2 + 2), plan[i].stream));
	}

	for (int i = 0; i < numClusters; i++)
	{
		//Find indices
		//Can be used on any GPU
		sample_sub_clusters_worker_v2(i + 1, plan[i].d_indices, plan[i].d_indicesSize, plan[i].stream);

		//Return the likelihhod in r vector.
		//Can be used on any GPU
		log_likelihood_v2(plan[i].d_r, d_zero, plan[i].d_b_l, plan[i].d_c_l, plan[i].d_z_l, plan[i].d_mu_l, plan[i].d_indices, plan[i].d_indicesSize, dim, cluster_params[i]->l_dist, plan[i].stream);
		log_likelihood_v2(plan[i].d_r, plan[i].d_indicesSize, plan[i].d_b_r, plan[i].d_c_r, plan[i].d_z_r, plan[i].d_mu_r, plan[i].d_indices, plan[i].d_indicesSize, dim, cluster_params[i]->r_dist, plan[i].stream);
	}

	for (int i = 0; i < numClusters; i++)
	{
		//Change the sub labels. Should run on one GPU
		sample_log_cat_array_sub_cluster_v2(plan[i].d_r, plan[i].d_indicesSize, plan[i].d_y2, plan[i].d_a2, plan[i].d_b2, plan[i].d_indices, plan[i].d_indicesSize, plan[i].d_lr_weights, plan[i].stream);
	}

	//Wait for all operations to finish
	for (int i = 0; i < numClusters; i++)
	{
		runCuda(hipStreamSynchronize(plan[i].stream));
	}

	for (int i = 0; i < numClusters; i++)
	{
		runCuda(hipFreeAsync(plan[i].d_indices, plan[i].stream));
		runCuda(hipFreeAsync(plan[i].d_indicesSize, plan[i].stream));
		runCuda(hipFreeAsync(plan[i].d_r, plan[i].stream));

		runCuda(hipFreeAsync(plan[i].d_z_l, plan[i].stream));
		runCuda(hipFreeAsync(plan[i].d_z_r, plan[i].stream));

		runCuda(hipFreeAsync(plan[i].d_mu_l, plan[i].stream));
		runCuda(hipFreeAsync(plan[i].d_mu_r, plan[i].stream));

		runCuda(hipFreeAsync(plan[i].d_b_l, plan[i].stream));
		runCuda(hipFreeAsync(plan[i].d_b_r, plan[i].stream));

		runCuda(hipFreeAsync(plan[i].d_c_r, plan[i].stream));
		runCuda(hipFreeAsync(plan[i].d_c_l, plan[i].stream));

		runCuda(hipFreeAsync(plan[i].d_lr_weights, plan[i].stream));
		runCuda(hipFreeAsync(plan[i].d_y2, plan[i].stream));
		runCuda(hipFreeAsync(plan[i].d_a2, plan[i].stream));
		runCuda(hipFreeAsync(plan[i].d_b2, plan[i].stream));

		runCuda(hipStreamDestroy(plan[i].stream));
	}

	delete[]plan;
}

typedef struct
{
	//Stream for asynchronous command execution
	hipStream_t stream;

	int* d_indicesSize;

	double* d_z;

	double* d_mu;

	double* d_b;

	double* d_c;

	

} clusters_labels_plan;

void cudaKernel::create_clusters_labels(int numClusters, std::vector<thin_cluster_params*>& cluster_params, std::vector<double>& weights, bool bFinal)
{	
	clusters_labels_plan* plan = new clusters_labels_plan[numClusters];
	int* d_zero;
	double* d_r;
	double* d_y2;
	int* d_a2;
	int* d_b2;

	const mv_gaussian* ds = (mv_gaussian*)(cluster_params[0]->cluster_dist);
	int dim = ds->invSigma.rows();

	//printf("Need %ld,  sizeof(double):%ld,  numLabels:%ld,  numClusters:%ld\n", sizeof(double) * numLabels * numClusters, sizeof(double) , numLabels , numClusters);
	runCuda(hipMalloc((void**)&d_r, sizeof(double) * numLabels * numClusters));
	runCuda(hipMalloc((void**)&d_y2, sizeof(double) * numLabels * (numClusters + 2)));
	runCuda(hipMalloc((void**)&d_a2, sizeof(int) * numLabels * (numClusters + 2)));
	runCuda(hipMalloc((void**)&d_b2, sizeof(int) * numLabels * (numClusters + 2)));

	//Allocate memory for all streams
	for (int i = 0; i < numClusters; i++)
	{
		runCuda(hipStreamCreate(&(plan[i].stream)));
		runCuda(hipMallocAsync((void**)&d_zero, sizeof(int), plan[i].stream));
		runCuda(hipMemsetAsync(d_zero, 0, sizeof(int), plan[i].stream));
//		runCuda(hipMalloc((void**)&(plan[i].d_indices), sizeof(int) * numLabels));
		runCuda(hipMallocAsync(&(plan[i].d_indicesSize), sizeof(int), plan[i].stream));
		runCuda(hipMemcpyAsync(plan[i].d_indicesSize, &numLabels, sizeof(int), hipMemcpyHostToDevice, plan[i].stream));

		const mv_gaussian* ds = (mv_gaussian*)(cluster_params[i]->cluster_dist);
		runCuda(hipMallocAsync((void**)&(plan[i].d_z), sizeof(double) * numClusters * numLabels, plan[i].stream));
		runCuda(hipMallocAsync((void**)&(plan[i].d_mu), sizeof(double) * ds->mu.size(), plan[i].stream));
		runCuda(hipMallocAsync((void**)&(plan[i].d_b), sizeof(double) * ds->invSigma.size(), plan[i].stream));

	//	printf("ds->invSigma.rows():%ld, ds->invSigma.cols():%ld, numLabels:%ld\n", ds->invSigma.rows(), ds->invSigma.cols(), numLabels);
		runCuda(hipMallocAsync((void**)&(plan[i].d_c), sizeof(double) * ds->invSigma.rows() * numLabels, plan[i].stream));
	}

	for (int i = 0; i < numClusters; i++)
	{
		log_likelihood_v3(d_r + i * numLabels, d_zero, plan[i].d_b, plan[i].d_c, plan[i].d_z, plan[i].d_mu, plan[i].d_indicesSize, dim, weights[i], cluster_params[i]->cluster_dist, plan[i].stream);
	}

	//Wait for all operations to finish
	for (int i = 0; i < numClusters; i++)
	{
		runCuda(hipStreamSynchronize(plan[i].stream));
	}

	if (bFinal)
	{
		update_labels_by_max_index(d_r, numClusters, plan[0].stream);
	}
	else
	{
		//Change the sub labels. Should run on one GPU
		sample_log_cat_array_v2(d_r, d_y2, d_a2, d_b2, numClusters, plan[0].stream);
	}
	runCuda(hipStreamSynchronize(plan[0].stream));

	for (int i = 0; i < numClusters; i++)
	{
		runCuda(hipFreeAsync(plan[i].d_indicesSize, plan[i].stream));

		runCuda(hipFreeAsync(plan[i].d_z, plan[i].stream));

		runCuda(hipFreeAsync(plan[i].d_mu, plan[i].stream));

		runCuda(hipFreeAsync(plan[i].d_b, plan[i].stream));

		runCuda(hipFreeAsync(plan[i].d_c, plan[i].stream));

		runCuda(hipStreamDestroy(plan[i].stream));
	}
	runCuda(hipFree(d_r));
	runCuda(hipFree(d_y2));
	runCuda(hipFree(d_a2));
	runCuda(hipFree(d_b2));

	delete[]plan;
}

void cudaKernel::checkCUDAError(hipError_t err, const char* file, int line)
{
	if (hipSuccess != err)
	{
		printf("Cuda error: %s(%d):%s.\n", file, line, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

#endif