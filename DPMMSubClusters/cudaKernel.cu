#include "hip/hip_runtime.h"
#ifndef CudaKernel_CU
#define CudaKernel_CU

//#pragma warning( disable : 2886 )
//#pragma warning( disable : 2929)
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>

#include<time.h>
#include "cudaKernel.cuh"
#include "distributions/mv_gaussian.h"

// function to define seed
__global__ void initCurand(hiprandState *state, unsigned long long seed, int maxIdx) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		hiprand_init(seed, idx, 0, &state[idx]);
	}
}

__device__ void sample_by_probability(hiprandState *state, double *weight, int numClusters, int rows, int idx, int *index,
									  double *y, int *a, int *b)
{
	int i;
	int j;
	int k;
	int n = numClusters;

	
//	if (n > 1)
	{
		a[idx] = 0;
		for (i = 1; i <= n; i++)
		{
			a[i*rows + idx] = i;
		}
		a[(n+1)*rows + idx] = n + 1;

		b[idx] = 0;
		for (i = 1; i <= n; i++)
		{
			b[i*rows + idx] = i;
		}
		b[(n+1)*rows + idx] = n + 1;
		/*
		  Copy Y from X.
		  Scale the probability vector and set sentinel values at the ends.
		*/
		y[idx] = 0.0;
		for (i = 1; i <= n; i++)
		{
			y[i*rows + idx] = weight[(i-1)*rows + idx] * (double)(n);
		}
		y[(n+1)*rows + idx] = 2.0;

		i = 0;
		j = n + 1;
		for (; ; )
		{
			/*
			  Find i so Y[B[i]] needs more.
			*/
			do
			{
				i++;
			} while (y[b[i*rows + idx]*rows + idx] < 1.0);
			/*
				  Find j so Y[B[j]] wants less.
				*/
			do
			{
				j--;
			} while (1.0 <= y[b[j*rows + idx]*rows + idx]);

			if (j <= i)
			{
				break;
			}
			/*
			  Swap B[i] and B[j].
			*/
			k = b[i*rows + idx];
			b[i*rows + idx] = b[j*rows + idx];
			b[j*rows + idx] = k;
		}

		i = j;
		j++;

		while (0 < i)
		{
			/*
			  Find J such that Y[B[j]] needs more.
			*/
			while (y[b[j*rows + idx]*rows + idx] <= 1.0)
			{
				j++;
			}
			/*
			  Meanwhile, Y[B[i]] wants less.
			*/
			if (n < j)
			{
				break;
			}
			/*
			  B[i] will donate to B[j] to fix up.
			*/
			y[b[j*rows + idx]*rows + idx] = y[b[j*rows + idx]*rows + idx] - (1.0 - y[b[i*rows + idx]*rows + idx]);
			a[b[i*rows + idx]*rows + idx] = b[j*rows + idx];
			/*
			  Y[B[j]] now wants less so readjust ordering.
			*/
			if (y[b[j*rows + idx]*rows + idx] < 1.0)
			{
				k = b[i*rows + idx];
				b[i*rows + idx] = b[j*rows + idx];
				b[j*rows + idx] = k;
				j++;
			}
			else
			{
				i--;
			}
		}

		double r;
		/*
		  Let i = random uniform integer from {1,2,...N};
		*/
		i = 1 + (int)(n * hiprand_uniform(state));
		//for (int j = 0; j < 2; j++)
		{
			r = hiprand_uniform(state);

			if (y[i*rows + idx] < r)
			{
				i = a[i*rows + idx];
//				break;
			}
		}
		*index = i;
//		*index = 1;
	}
//	else
	{
//		*index = 1;
	}
}

//__device__ void sample_by_probability(hiprandState *state, double *weight, int numClusters, int rows, int idx, int *index)
//{
//	//do
//	{
//		int cluster1 = 0;
//		int cluster2 = 0;
//		if (numClusters > 1)
//		{
//			cluster2 = 1;
//		}
//
//		if (weight[cluster1*rows + idx] > weight[cluster2*rows + idx])
//		{
//			*index = cluster1 + 1;
//		}
//		else
//		{
//			*index = cluster2 + 1;
//
//		}
//	}
//}

__global__ void sample_log_cat_array_all(hiprandState *state, int *dev_sample, int maxIdx, int numClusters, double *d_log_likelihood_array, double *y, int *a, int *b)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_by_probability(&state[idx], d_log_likelihood_array, numClusters, maxIdx, idx, &dev_sample[idx], y, a, b);
	}
}

__global__ void sample_log_cat_array_sub_cluster_all(hiprandState *state, int *dev_sample, int maxIdx, int num, double *d_log_likelihood_array, int *indices, double *y, int *a, int *b)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_by_probability(&state[indices[idx]], d_log_likelihood_array, num, maxIdx, idx, &dev_sample[indices[idx]], y, a, b);
	}
}

__device__ void sample_sub_label(hiprandState *state, int *d_label)
{
	*d_label = ((int)(hiprand_uniform(state) * 2)) % 2 + 1;
}

__global__ void sample_sub_labels_all(hiprandState *state, int *d_labels, int maxIdx)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_sub_label(&state[idx], &d_labels[idx]);
	}
}

__device__ void sample_label(hiprandState *state, int *d_label, int initial_clusters, double outlier_mod)
{
	*d_label = ((int)(hiprand_uniform(state)*initial_clusters)) % initial_clusters + 1 + ((outlier_mod > 0) ? 1 : 0);
}

__global__ void sample_labels_all(hiprandState *state, int *d_labels, int maxIdx, int initial_clusters, double outlier_mod)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		sample_label(&state[idx], &d_labels[idx], initial_clusters, outlier_mod);
	}
}

__device__ void remove_empty_clusters_worker(int *d_label, int limit)
{
	if (*d_label > limit)
	{
		*d_label -= 1;
	}
}

__global__ void remove_empty_clusters_worker_all(int *d_labels, int maxIdx, int limit)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		remove_empty_clusters_worker(&d_labels[idx], limit);
	}
}

__global__ void find_indices(int *d_labels, int maxIdx, int label, int *d_indices, int *d_indicesSize)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		if (d_labels[idx] == label)
		{
			int curIndex = atomicAdd(d_indicesSize, 1);
			d_indices[curIndex] = idx;
		}
	}
}

__device__ void split_cluster_local_worker(hiprandState *state, int *d_labels, int *d_sub_label, int index, int newIndex)
{
	if (*d_labels == index + 1)
	{
		if (*d_sub_label == 2)
		{
			*d_labels = newIndex + 1;
		}
		*d_sub_label = hiprand(state) % 2 + 1;
	}
}

__global__ void split_cluster_local_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index, int newIndex)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		split_cluster_local_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index, newIndex);
	}
}

__device__ void merge_clusters_worker(hiprandState *state, int *d_labels, int *d_sub_labels, int index, int newIndex)
{
	if (*d_labels == index + 1)
	{
		*d_sub_labels = 1;
	}
	if (*d_labels == newIndex + 1)
	{
		*d_sub_labels = 2;
	}
	if (*d_labels == newIndex + 1)
	{
		*d_labels = index + 1;
	}
}

__global__ void merge_clusters_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index, int newIndex)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		merge_clusters_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index, newIndex);
	}
}

__device__ void reset_bad_clusters_worker(hiprandState *state, int *d_labels, int *d_sub_labels, int index)
{
	if (*d_labels == index + 1)
	{
		*d_sub_labels = hiprand(state) % 2 + 1;
	}
}

__global__ void reset_bad_clusters_worker_all(hiprandState *state, int *d_labels, int maxIdx, int *d_sub_labels, int index)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		reset_bad_clusters_worker(&state[idx], &d_labels[idx], &d_sub_labels[idx], index);
	}
}

__global__ void get_sub_labels_count_all(int *d_sub_labels, int maxIdx, int *l, int *r)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		if (d_sub_labels[idx] == 1)
		{
			atomicAdd(l, 1);
		}
		else if (d_sub_labels[idx] == 2)
		{
			atomicAdd(r, 1);
		}
	}
}

__global__ void create_suff_stats_dict_worker_all(
	int* d_sub_labels,
	int maxIdx,
	int *d_indices,
	int *d_indicesSize,
	double *group_pts,
	int group_pts_rows,
	double *pts,
	double *pts1,
	double *pts2,
	int *d_j1,
	int *d_j2)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < *d_indicesSize)
	{
		if (d_sub_labels[d_indices[idx]] == 1)
		{
			int curj1 = atomicAdd(d_j1, 1);
			for (int i = 0; i < group_pts_rows; i++)
			{
				double pt = group_pts[IDX2C(i, d_indices[idx], group_pts_rows)];
				pts[IDX2C(i, idx, group_pts_rows)] = pt;
				pts1[IDX2C(i, curj1, group_pts_rows)] = pt;
			}
		}
		else if (d_sub_labels[d_indices[idx]] == 2)
		{
			int curj2 = atomicAdd(d_j2, 1);
			for (int i = 0; i < group_pts_rows; i++)
			{
				double pt = group_pts[IDX2C(i, d_indices[idx], group_pts_rows)];
				pts[IDX2C(i, idx, group_pts_rows)] = pt;
				pts2[IDX2C(i, curj2, group_pts_rows)] = pt;
			}
		}
	}
}

__global__ void dcolwise_dot_all_kernel(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, int r_offset)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double sum = 0;
		for (int i = 0; i < rows; i++)
		{
			sum += d_a[idx * rows + i] * d_b[idx * rows + i];
		}

		d_r[idx + r_offset] = scalar - sum / 2;
	}
}

__global__ void dcolwise_dot_with_log_kernel(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, double weight)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double sum = 0;
		for (int i = 0; i < rows; i++)
		{
			sum += d_a[idx * rows + i] * d_b[idx * rows + i];
		}

		d_r[idx] = scalar - sum / 2 + __logf(weight);
	}
}

__global__ void build_log_likelihood_array_sub_cluster_kernel(int maxIdx, double* d_log_likelihood_array, double* d_lr_weights, int rows, int cols)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double maxRow = 0;
		bool first = true;
		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] += log(d_lr_weights[j]);
			if (first || d_log_likelihood_array[IDX2C(idx, j, rows)] > maxRow)
			{
				first = false;
				maxRow = d_log_likelihood_array[IDX2C(idx, j, rows)];
			}
		}

		double sum = 0;
		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = exp(d_log_likelihood_array[IDX2C(idx, j, rows)] - maxRow);
			sum += d_log_likelihood_array[IDX2C(idx, j, rows)];
		}

		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = d_log_likelihood_array[IDX2C(idx, j, rows)] / sum;
		}
	}
}

__global__ void build_log_likelihood_array_sub_cluster_kernel_v2(int maxIdx, double* d_r, int r_offset, double* d_lr_weights)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double maxRow;
		double sum;

		d_r[idx] += __logf(d_lr_weights[0]);
		d_r[idx + r_offset] += __logf(d_lr_weights[1]);
		maxRow = fmax(d_r[idx], d_r[idx + r_offset]);

		d_r[idx] = __expf(d_r[idx] - maxRow);
		d_r[idx + r_offset] = __expf(d_r[idx + r_offset] - maxRow);
		sum = d_r[idx] + d_r[idx + r_offset];

		d_r[idx] = d_r[idx] / sum;
		d_r[idx + r_offset] = d_r[idx + r_offset] / sum;
	}
}

__global__ void build_log_likelihood_array_kernel(int maxIdx, double* d_log_likelihood_array, int rows, int cols)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < maxIdx)
	{
		double maxRow = 0;
		bool first = true;
		for (int j = 0; j < cols; j++)
		{
			if (first || d_log_likelihood_array[IDX2C(idx, j, rows)] > maxRow)
			{
				first = false;
				maxRow = d_log_likelihood_array[IDX2C(idx, j, rows)];
			}
		}

		double sum = 0;
		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = exp(d_log_likelihood_array[IDX2C(idx, j, rows)] - maxRow);
			sum += d_log_likelihood_array[IDX2C(idx, j, rows)];
		}

		for (int j = 0; j < cols; j++)
		{
			d_log_likelihood_array[IDX2C(idx, j, rows)] = d_log_likelihood_array[IDX2C(idx, j, rows)] / sum;
		}
	}
}

__global__ void update_labels_by_max_index_kernel(double* parr, int* d_labels, int maxIdx, int dim)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < maxIdx)
	{
		double max = 0;
		bool first = true;
		int max_index = 0;
		for (int i = 0; i < dim; ++i)
		{
			if (first || max < parr[IDX2C(idx, i, maxIdx)])
			{
				first = false;
				max = parr[IDX2C(idx, i, maxIdx)];
				max_index = i;
			}
		}
		d_labels[idx] = max_index + 1;
	}
}

__global__ void gpu_matrix_mult(double* a, double* b, double* c, int m, int n, int k)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	double sum = 0;
//	printf("row=%d, col=%d\n", row, col);
	if (col < k && row < m)
	{
		for (int i = 0; i < n; i++)
		{
			//sum += a[row * n + i] * b[i * k + col];
			sum += a[IDX2C(row, i,m)] * b[IDX2C(i, col,n)];
			//printf("cuSum=%f, sum=%f\n", a[IDX2C(row, i, m)] * b[IDX2C(i, col, n)], sum);
//			printf("cuSum=%f, sum=%f, a=%f, b=%f\n", a[IDX2C(row, i, m)] * b[IDX2C(i, col, n)], sum, a[IDX2C(row, i, n)], b[IDX2C(i, col, k)]);

//			printf("row=%d, col=%d, i=%d, a=%f, b=%f\n", row, col,i, a[IDX2C(row, i, n)], b[IDX2C(i, col, k)]);

			
		}
		//c[row * k + col] = sum;
		c[IDX2C(row, col,m)] = sum;

	}
}

void cudaKernel::init(int numLabelsIn, MatrixXd &points, unsigned long long seed)
{
	printf("Init cuda\n");

	int numGPU;
	int driverVersion = 0, runtimeVersion = 0;

	lastDevice = 0;

	runCuda(hipGetDeviceCount(&numGPU));
	numGPU = 1;

	printf("Number of GPUs: %i\n", numGPU);
	printf("number of host CPUs:\t%d\n", omp_get_num_procs());

	for (int i = 0; i < numGPU; i++)
	{
		hipSetDevice(i);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, i);

		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);

		printf("\nDevice %d: \"%s\"\n", i, deviceProp.name);
		printf("CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
		printf("CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

		int* dummy;
		hipStream_t stream;

		hipStreamCreate(&stream);

		hipError_t err = hipMallocAsync((void**)&dummy, sizeof(int), stream);
		if (hipSuccess != err)
		{
			hipGetLastError();
			printf("Not capable device. Can't perform asynchronous memory allocation.\n");
		}
		else
		{
			gpuCapabilities[i] = gpuCapability();
		}
		hipStreamSynchronize(stream);
		hipStreamDestroy(stream);
	}
		
	printf("\nNumber of GPUs that will be used: %i\n\n", gpuCapabilities.size());

	numLabels = numLabelsIn;
	threads = dim3(512);
	blocks = dim3(numLabels / threads.x + 1);

	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		hipSetDevice(iter->first);
		runCuda(hipDeviceReset());
		runCuda(hipMalloc((void**)&(iter->second.devState), numLabels * sizeof(hiprandState)));
		initCurand << <blocks, threads >> > (iter->second.devState, seed, numLabels);
		runCuda(hipPeekAtLastError());

		runCuda(hipMalloc((void**)&(iter->second.d_labels), numLabels * sizeof(int)));
		runCuda(hipMalloc((void**)&(iter->second.d_sub_labels), numLabels * sizeof(int)));
		runCuda(hipMalloc((void**)&(iter->second.d_points), points.size() * sizeof(double)));
		runCuda(hipMemcpy(iter->second.d_points, points.data(), points.size() * sizeof(double), hipMemcpyHostToDevice));
	}

	if (gpuCapabilities.size() > 0)
	{
		hipSetDevice(gpuCapabilities.begin()->first);
	}
}

void cudaKernel::release()
{
	printf("Release cuda\n");

	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		if (iter->second.devState != NULL)
		{
			runCuda(hipFree(iter->second.devState));
		}

		if (iter->second.d_labels != NULL)
		{
			runCuda(hipFree(iter->second.d_labels));
		}

		if (iter->second.d_sub_labels != NULL)
		{
			runCuda(hipFree(iter->second.d_sub_labels));
		}

		if (iter->second.d_points != NULL)
		{
			runCuda(hipFree(iter->second.d_points));
		}
	}
}

int cudaKernel::peak_device()
{
	int result;
	//++lastDevice;
	//if (lastDevice >= gpuCapabilities.size())
	//	lastDevice = 0;

	//int i = 0;
	//for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); i <= lastDevice && iter != gpuCapabilities.end(); iter++, ++i)
	//{
	//	result = iter->first;
	//}
	//hipSetDevice(result);
	//return result;
	result = gpuCapabilities.begin()->first;
	hipSetDevice(result);
	return result;
}

int cudaKernel::peak_device_v2()
{
	int result;
	++lastDevice;
	if (lastDevice >= gpuCapabilities.size())
		lastDevice = 0;

	int i = 0;
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); i <= lastDevice && iter != gpuCapabilities.end(); iter++, ++i)
	{
		result = iter->first;
	}
	hipSetDevice(result);
	return result;
	//return 0;
}

int cudaKernel::sample_log_cat_array_sub_cluster(LabelType *indices, int labelsSize, Eigen::MatrixXd &log_likelihood_array, std::vector<double>& lr_weights, int deviceId)
{
	int *d_indices;
	double *d_y;
	int *d_a;
	int *d_b;
	double* d_lr_weights;
	
	runCuda(hipMalloc((void**)&d_indices, sizeof(int) * labelsSize));
	runCuda(hipMemcpy(d_indices, indices, sizeof(int) * labelsSize, hipMemcpyHostToDevice));
	runCuda(hipMalloc((void**)&d_lr_weights, sizeof(double) * lr_weights.size()));
	runCuda(hipMemcpy(d_lr_weights, lr_weights.data(), sizeof(double) * lr_weights.size(), hipMemcpyHostToDevice));

	dim3 blocks_size = dim3(labelsSize / threads.x + 1);
	
	int n = log_likelihood_array.size();
	double *d_log_likelihood_array;
	runCuda(hipMalloc((void **)&d_log_likelihood_array, sizeof(double)*n));
	runCuda(hipMemcpy(d_log_likelihood_array, log_likelihood_array.data(), sizeof(double)*n, hipMemcpyHostToDevice));
	runCuda(hipMalloc((void **)&d_y, sizeof(double)*labelsSize*(log_likelihood_array.cols() + 2)));
	runCuda(hipMalloc((void **)&d_a, sizeof(int)*labelsSize*(log_likelihood_array.cols() + 2)));
	runCuda(hipMalloc((void **)&d_b, sizeof(int)*labelsSize*(log_likelihood_array.cols() + 2)));

	//	int dev = 0;//GPU index. need to loop for all GPUs

	build_log_likelihood_array_sub_cluster_kernel << <blocks_size, threads >> > (log_likelihood_array.rows(), d_log_likelihood_array, d_lr_weights, log_likelihood_array.rows(), log_likelihood_array.cols());
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
	
	sample_log_cat_array_sub_cluster_all << <blocks_size, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_sub_labels, labelsSize, log_likelihood_array.cols(), d_log_likelihood_array, d_indices, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	//free memory
	runCuda(hipFree(d_indices));
	runCuda(hipFree(d_lr_weights));
	runCuda(hipFree(d_log_likelihood_array));
	runCuda(hipFree(d_y));
	runCuda(hipFree(d_a));
	runCuda(hipFree(d_b));

	return 0;
}

void cudaKernel::sample_log_cat_array_sub_cluster_v2(
	double* d_r,
	int r_offset,
	int* d_indices,
	int indicesSize,
	double* d_lr_weights,
	hipStream_t& stream,
	int deviceId)
{
	double* d_y;
	int* d_a;
	int* d_b;

	runCuda(hipMallocAsync((void**)&d_y, sizeof(double) * indicesSize * (2 + 2), stream));
	runCuda(hipMallocAsync((void**)&d_a, sizeof(int) * indicesSize * (2 + 2), stream));
	runCuda(hipMallocAsync((void**)&d_b, sizeof(int) * indicesSize * (2 + 2), stream));

	build_log_likelihood_array_sub_cluster_kernel_v2 << <blocks, threads, 0, stream >> > (indicesSize, d_r, r_offset, d_lr_weights);
	runCuda(hipPeekAtLastError());

	//TODO - Can we remove d_y, d_a, d_b?
	sample_log_cat_array_sub_cluster_all << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_sub_labels, indicesSize, 2, d_r, d_indices, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());

	runCuda(hipFreeAsync(d_y, stream));
	runCuda(hipFreeAsync(d_a, stream));
	runCuda(hipFreeAsync(d_b, stream));

	update_sub_labels_to_all_other_devices(deviceId, stream);
}

void cudaKernel::sample_log_cat_array_v2(
	double* d_r,
	int dim,
	hipStream_t& stream,
	int deviceId)
{
	double* d_y;
	int* d_a;
	int* d_b;

	runCuda(hipMallocAsync((void**)&d_y, sizeof(double) * numLabels * (dim + 2), stream));
	runCuda(hipMallocAsync((void**)&d_a, sizeof(int) * numLabels * (dim + 2), stream));
	runCuda(hipMallocAsync((void**)&d_b, sizeof(int) * numLabels * (dim + 2), stream));
	build_log_likelihood_array_kernel << <blocks, threads, 0, stream >> > (numLabels, d_r, numLabels, dim);
	runCuda(hipPeekAtLastError());

	sample_log_cat_array_all << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, dim, d_r, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());

	runCuda(hipFreeAsync(d_y, stream));
	runCuda(hipFreeAsync(d_a, stream));
	runCuda(hipFreeAsync(d_b, stream));

	update_labels_to_all_other_devices(deviceId, stream);
}

int cudaKernel::sample_log_cat_array(Eigen::MatrixXd &log_likelihood_array,	int deviceId)
{
	int n = log_likelihood_array.size();
	double *d_log_likelihood_array;
	double *d_y;
	int *d_a;
	int *d_b;

	runCuda(hipMalloc((void **)&d_log_likelihood_array, sizeof(double)*n));
	runCuda(hipMemcpy(d_log_likelihood_array, log_likelihood_array.data(), sizeof(double)*n, hipMemcpyHostToDevice));

	runCuda(hipMalloc((void **)&d_y, sizeof(double)*numLabels*(log_likelihood_array.cols() + 2)));
	runCuda(hipMalloc((void **)&d_a, sizeof(int)*numLabels*(log_likelihood_array.cols() + 2)));
	runCuda(hipMalloc((void **)&d_b, sizeof(int)*numLabels*(log_likelihood_array.cols() + 2)));

//	int dev = 0;//GPU index. need to loop for all GPUs
	build_log_likelihood_array_kernel << <blocks, threads >> > (log_likelihood_array.rows(), d_log_likelihood_array, log_likelihood_array.rows(), log_likelihood_array.cols());
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	sample_log_cat_array_all <<<blocks, threads>>>(gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, log_likelihood_array.cols(), d_log_likelihood_array, d_y, d_a, d_b);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	//free memory
	runCuda(hipFree(d_log_likelihood_array));
	runCuda(hipFree(d_y));
	runCuda(hipFree(d_a));
	runCuda(hipFree(d_b));
	return 0;

}

//void cudaKernel::sample_sub_clusters_worker(LabelType label, LabelType* &indices, LabelType &indicesSize, int deviceId)
//{
//	int *d_indices;
//	runCuda(hipMalloc((void **)&d_indices, sizeof(int)*numLabels));
//
//	int *d_indicesSize;
//	runCuda(hipMalloc(&d_indicesSize, sizeof(int)));
//	runCuda(hipMemset(d_indicesSize, 0, sizeof(int)));
//
//	find_indices << <blocks, threads >> > (gpuCapabilities[deviceId].d_labels, numLabels, label, d_indices, d_indicesSize);
//
//	runCuda(hipPeekAtLastError());
//	runCuda(hipDeviceSynchronize());
//
//	runCuda(hipMemcpy(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost));
//	runCuda(hipMemcpy(indices, d_indices, indicesSize * sizeof(int), hipMemcpyDeviceToHost));
//
//	runCuda(hipFree(d_indicesSize));
//	runCuda(hipFree(d_indices));
//}

void cudaKernel::sample_sub_clusters_worker_v2(LabelType label, int* d_indices, int &indicesSize, hipStream_t& stream, int deviceId)
{
	int* d_indicesSize;
	runCuda(hipMallocAsync(&d_indicesSize, sizeof(int), stream));
	runCuda(hipMemsetAsync(d_indicesSize, 0, sizeof(int), stream));

	find_indices << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].d_labels, numLabels, label, d_indices, d_indicesSize);
	runCuda(hipMemcpyAsync(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost, stream));

	runCuda(hipPeekAtLastError());
}

void cudaKernel::create_suff_stats_dict_worker(
	LabelType label,
	LabelType &indicesSize,
	Eigen::MatrixXd &group_pts,
	Eigen::MatrixXd* &pts,
	Eigen::MatrixXd* &pts1,
	Eigen::MatrixXd* &pts2)
{
	int deviceId = peak_device();
	int *d_indices;
	runCuda(hipMalloc((void **)&d_indices, sizeof(int)*numLabels));

	int *d_indicesSize;
	runCuda(hipMalloc(&d_indicesSize, sizeof(int)));
	runCuda(hipMemset(d_indicesSize, 0, sizeof(int)));

	find_indices << <blocks, threads >> > (gpuCapabilities[deviceId].d_labels, numLabels, label, d_indices, d_indicesSize);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());
	runCuda(hipMemcpy(&indicesSize, d_indicesSize, sizeof(int), hipMemcpyDeviceToHost));

	double *d_group_pts;
	runCuda(hipMalloc((void **)&d_group_pts, sizeof(double)*group_pts.size()));
	runCuda(hipMemcpy(d_group_pts, group_pts.data(), sizeof(double)*group_pts.size(), hipMemcpyHostToDevice));

	double *d_pts;
	runCuda(hipMalloc((void **)&d_pts, sizeof(double)*group_pts.rows()*indicesSize));

	double *d_pts1;
	runCuda(hipMalloc((void **)&d_pts1, sizeof(double)*group_pts.rows()*indicesSize));

	double *d_pts2;
	runCuda(hipMalloc((void **)&d_pts2, sizeof(double)*group_pts.rows()*indicesSize));

	int *d_j1;
	int *d_j2;
	runCuda(hipMalloc(&d_j1, sizeof(int)));
	runCuda(hipMemset(d_j1, 0, sizeof(int)));
	runCuda(hipMalloc(&d_j2, sizeof(int)));
	runCuda(hipMemset(d_j2, 0, sizeof(int)));

	dim3 blocks_size = dim3(indicesSize / threads.x + 1);
	create_suff_stats_dict_worker_all << <blocks_size, threads >> > (
		gpuCapabilities[deviceId].d_sub_labels,
		numLabels,
		d_indices,
		d_indicesSize,
		d_group_pts,
		group_pts.rows(),
		d_pts,
		d_pts1,
		d_pts2,
		d_j1,
		d_j2);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	int j1;
	int j2;
	runCuda(hipMemcpy(&j1, d_j1, sizeof(int), hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(&j2, d_j2, sizeof(int), hipMemcpyDeviceToHost));

	pts = new Eigen::MatrixXd(group_pts.rows(), indicesSize);
	pts1 = new Eigen::MatrixXd(group_pts.rows(), j1);
	pts2 = new Eigen::MatrixXd(group_pts.rows(), j2);

	runCuda(hipMemcpy(pts->data(), d_pts, sizeof(double)*group_pts.rows()*indicesSize, hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(pts1->data(), d_pts1, sizeof(double)*group_pts.rows()*j1, hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(pts2->data(), d_pts2, sizeof(double)*group_pts.rows()*j2, hipMemcpyDeviceToHost));

	runCuda(hipFree(d_j1));
	runCuda(hipFree(d_j2));
	runCuda(hipFree(d_indicesSize));
	runCuda(hipFree(d_group_pts));
	runCuda(hipFree(d_pts));
	runCuda(hipFree(d_pts1));
	runCuda(hipFree(d_pts2));
}
//
//void cudaKernel::create_suff_stats_dict_worker_v2(MatrixXd& group_pts, hyperparams* hyper_params, LabelsType& indices)
//{
//	std::map<LabelType, thin_suff_stats*> suff_stats_dict;
//
//	for (LabelType index = 0; index < indices.size(); index++)
//	{
//		LabelType indicesLabelsSize = 0;
//
//		MatrixXd* pts;
//		MatrixXd* pts1;
//		MatrixXd* pts2;
//
//		create_suff_stats_dict_worker(indices[index] + 1,
//			indicesLabelsSize,
//			group_pts,
//			pts,
//			pts1,
//			pts2);
//
//		thin_suff_stats* tss = new thin_suff_stats();
//		prior* cpl_suff = utils::create_sufficient_statistics(hyper_params, &(tss->l_suff), globalParams, *pts1);
//		delete cpl_suff;
//		prior* cpr_suff = utils::create_sufficient_statistics(hyper_params, &(tss->r_suff), globalParams, *pts2);
//		delete cpr_suff;
//		prior* cp_suff = utils::create_sufficient_statistics(hyper_params, &(tss->cluster_suff), globalParams, *pts);
//		delete cp_suff;
//
//		suff_stats_dict[index] = tss;
//
//		delete pts;
//		delete pts1;
//		delete pts2;
//	}
//
//	return suff_stats_dict;
//}

void cudaKernel::sample_sub_labels()
{
	int deviceId = peak_device();
	sample_sub_labels_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_sub_labels, numLabels);
	runCuda(hipPeekAtLastError());

	update_sub_labels_to_all_other_devices(deviceId);
}

void cudaKernel::sample_labels(int initial_clusters, double outlier_mod)
{
	int deviceId = peak_device();
	sample_labels_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, initial_clusters, outlier_mod);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	update_labels_to_all_other_devices(deviceId);
}

void cudaKernel::get_sub_labels(LabelsType &subLabels)
{
	int deviceId = peak_device();
	int *h_subLabels;
	h_subLabels = (int*)malloc(numLabels * sizeof(int));
	runCuda(hipMemcpy(h_subLabels, gpuCapabilities[deviceId].d_sub_labels, numLabels * sizeof(int), hipMemcpyDeviceToHost));

	subLabels.resize(numLabels);
	for (size_t i = 0; i < numLabels; i++)
	{
		subLabels[i] = h_subLabels[i];
	}

	free(h_subLabels);
}

void cudaKernel::get_labels(LabelsType &labels)
{
	int deviceId = peak_device();
	int *h_labels;
	h_labels = (int*)malloc(numLabels * sizeof(int));
	runCuda(hipMemcpy(h_labels, gpuCapabilities[deviceId].d_labels, numLabels * sizeof(int), hipMemcpyDeviceToHost));

	labels.resize(numLabels);
	for (size_t i = 0; i < numLabels; i++)
	{
		labels[i] = h_labels[i];
	}

	free(h_labels);
}

void cudaKernel::update_labels(int *updateLabels, int numLabels, int deviceId)
{
	runCuda(hipMemcpy(gpuCapabilities[deviceId].d_labels, updateLabels, sizeof(int)*numLabels, hipMemcpyHostToDevice));
}

void cudaKernel::update_labels_to_all_other_devices(int srcDeviceId)
{
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		//printf("20\n");
		bool needToFree;
		device_to_device_copy(srcDeviceId, iter->first, numLabels, gpuCapabilities[srcDeviceId].d_labels, iter->second.d_labels, true, needToFree);
	}
}

void cudaKernel::update_labels_to_all_other_devices(int srcDeviceId, hipStream_t& stream)
{
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
//		printf("21\n");
		bool needToFree;
		device_to_device_copy(srcDeviceId, iter->first, numLabels, gpuCapabilities[srcDeviceId].d_labels, iter->second.d_labels, true, needToFree, stream);
	}
}

void cudaKernel::update_sub_labels_to_all_other_devices(int srcDeviceId)
{
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		//printf("22\n");

		bool needToFree;
		device_to_device_copy(srcDeviceId, iter->first, numLabels, gpuCapabilities[srcDeviceId].d_sub_labels, iter->second.d_sub_labels, true, needToFree);
	}
}

void cudaKernel::update_sub_labels_to_all_other_devices(int srcDeviceId, hipStream_t& stream)
{
	for (std::map<int, gpuCapability>::iterator iter = gpuCapabilities.begin(); iter != gpuCapabilities.end(); iter++)
	{
		//printf("23\n");

		bool needToFree;
		device_to_device_copy(srcDeviceId, iter->first, numLabels, gpuCapabilities[srcDeviceId].d_sub_labels, iter->second.d_sub_labels, true, needToFree, stream);
	}
}

void cudaKernel::update_labels_by_max_index(double* parr, int dim, hipStream_t& stream, int deviceId)
{
	update_labels_by_max_index_kernel << <blocks, threads, 0, stream >> > (parr, gpuCapabilities[deviceId].d_labels, numLabels, dim);

	update_labels_to_all_other_devices(deviceId, stream);
}


void cudaKernel::remove_empty_clusters_worker(int limit)
{
	int deviceId = peak_device();
	remove_empty_clusters_worker_all << <blocks, threads >> > (gpuCapabilities[deviceId].d_labels, numLabels, limit);
	runCuda(hipPeekAtLastError());

	update_labels_to_all_other_devices(deviceId);
}

void cudaKernel::split_cluster_local_worker(LabelType index, LabelType newIndex)
{
	int deviceId = peak_device();
	split_cluster_local_worker_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, gpuCapabilities[deviceId].d_sub_labels, index, newIndex);
	runCuda(hipPeekAtLastError());

	update_labels_to_all_other_devices(deviceId);
	update_sub_labels_to_all_other_devices(deviceId);
}

void cudaKernel::merge_clusters_worker(LabelType index, LabelType newIndex)
{
	int deviceId = peak_device();
	merge_clusters_worker_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, gpuCapabilities[deviceId].d_sub_labels, index, newIndex);
	runCuda(hipPeekAtLastError());

	update_labels_to_all_other_devices(deviceId);
	update_sub_labels_to_all_other_devices(deviceId);
}

void cudaKernel::reset_bad_clusters_worker(LabelType index)
{
	int deviceId = peak_device();

	reset_bad_clusters_worker_all << <blocks, threads >> > (gpuCapabilities[deviceId].devState, gpuCapabilities[deviceId].d_labels, numLabels, gpuCapabilities[deviceId].d_sub_labels, index);
	runCuda(hipPeekAtLastError());

	update_sub_labels_to_all_other_devices(deviceId);
}

void cudaKernel::get_sub_labels_count(int &l, int &r)
{
	int deviceId = peak_device();
	int *d_l;
	runCuda(hipMalloc((void **)&d_l, sizeof(int)));
	runCuda(hipMemset(d_l, 0, sizeof(int)));
	
	int *d_r;
	runCuda(hipMalloc((void **)&d_r, sizeof(int)));
	runCuda(hipMemset(d_r, 0, sizeof(int)));

	get_sub_labels_count_all << <blocks, threads >> > (gpuCapabilities[deviceId].d_sub_labels, numLabels, d_l, d_r);
	runCuda(hipPeekAtLastError());
	runCuda(hipDeviceSynchronize());

	runCuda(hipMemcpy(&l, d_l, sizeof(int), hipMemcpyDeviceToHost));
	runCuda(hipMemcpy(&r, d_r, sizeof(int), hipMemcpyDeviceToHost));

	runCuda(hipFree(d_l));
	runCuda(hipFree(d_r));
}

// C(m,k) = A(m,n) * B(n,k)
void cudaKernel::naive_matrix_multiply(const double* A, const double* B, double* C, int m, int n, int k)
{
	int lda = m, ldb = k, ldc = m;
	const double alf = 1;
	const double bet = 0;
	const double* alpha = &alf;
	const double* beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Do the actual multiplication
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}

// C(m,k) = A(m,n) * B(n,k)
void cudaKernel::naive_matrix_multiply_v2(const double* A, const double* B, double* C, int m, int n, int k, hipStream_t& stream)
{
	int lda = m, ldb = k, ldc = m;
	const double alf = 1;
	const double bet = 0;
	const double* alpha = &alf;
	const double* beta = &bet;

	// Create a handle for CUBLAS
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSetStream(handle, stream);
	// Do the actual multiplication
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

	// Destroy the handle
	hipblasDestroy(handle);
}

void cudaKernel::naive_matrix_multiply_v3(double* A, double* B, double* C, int m, int n, int k, hipStream_t& stream)
{
	const int BlockSize = 16;

	unsigned int grid_rows = (m + BlockSize - 1) / BlockSize;
	unsigned int grid_cols = (k + BlockSize - 1) / BlockSize;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BlockSize, BlockSize);


//	printf("m=%d, n=%d, k=%d\n", m, n, k);
	if (k > 0)
	{
		gpu_matrix_mult << <dimGrid, dimBlock, 0, stream >> > (A, B, C, m, n, k);
		runCuda(hipPeekAtLastError());
	}


//	runCuda(hipMemcpy(A2, A, sizeof(double)* sizeVec, hipMemcpyDeviceToHost));




}

void cudaKernel::dcolwise_dot_all(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, int r_offset)
{
	dcolwise_dot_all_kernel << <blocks, threads>> > (maxIdx, rows, d_a, d_b, scalar, d_r, r_offset);
}

void cudaKernel::dcolwise_dot_all_v2(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, int r_offset, hipStream_t& stream)
{
	dcolwise_dot_all_kernel << <blocks, threads, 0, stream >> > (maxIdx, rows, d_a, d_b, scalar, d_r, r_offset);
}

void cudaKernel::dcolwise_dot_all_v3(int maxIdx, int rows, double* d_a, double* d_b, double scalar, double* d_r, double weight, hipStream_t& stream)
{
	dcolwise_dot_with_log_kernel << <blocks, threads, 0, stream >> > (maxIdx, rows, d_a, d_b, scalar, d_r, weight);
}

//void cudaKernel::dcolwise_dot(Eigen::VectorXd& r, const Eigen::MatrixXd& a, const Eigen::MatrixXd& b)
//{
//	int sizeVec = a.cols();
//
//	double* d_a;
//	double* d_b;
//	double* d_c;
//	double* d_r;
//
//	runCuda(hipMalloc((void**)&d_a, sizeof(double) * a.size()));
//	runCuda(hipMalloc((void**)&d_b, sizeof(double) * b.size()));
//	runCuda(hipMalloc((void**)&d_c, sizeof(double) * b.rows() * a.cols()));
//	runCuda(hipMemcpy(d_a, a.data(), sizeof(double) * a.size(), hipMemcpyHostToDevice));
//	runCuda(hipMemcpy(d_b, b.data(), sizeof(double) * b.size(), hipMemcpyHostToDevice));
//
//	naive_matrix_multiply(d_b, d_a, d_c, b.rows(), a.cols(), b.cols());
//
//	runCuda(hipFree(d_b));
//
//	runCuda(hipMalloc((void**)&d_r, sizeof(double)* sizeVec));
//
//	dcolwise_dot_all(sizeVec, a.rows(), d_a, d_c, d_r);
//
//	r.resize(sizeVec);
//	runCuda(hipMemcpy(r.data(), d_r, sizeof(double)* sizeVec, hipMemcpyDeviceToHost));
//	
//	runCuda(hipFree(d_a));
//	runCuda(hipFree(d_c));
//	runCuda(hipFree(d_r));
//}

typedef struct
{
	//Stream for asynchronous command execution
	hipStream_t stream;

	int* d_indices;
	int indicesSize;
	double* d_r;
	double* d_lr_weights;
	int deviceId;
} subclusters_labels_plan;

void cudaKernel::create_subclusters_labels(int numClusters, std::vector<thin_cluster_params*>& cluster_params, int dim)
//LabelType* indices, LabelType indicesSize, distribution_sample* l_dist, distribution_sample* r_dist, std::vector<double> &lr_weights)
{
//	omp_set_num_threads(20);
//	#pragma omp parallel
	{
//		unsigned int i = omp_get_thread_num();

//		printf("**** i=%d\n", i);
	}

	subclusters_labels_plan* plan = new subclusters_labels_plan[numClusters];

	//Allocate memory for all streams
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel
	for (int i = 0; i < numClusters; i++)
	{
//		unsigned int i = omp_get_thread_num();

		plan[i].deviceId = peak_device_v2();
		runCuda(hipStreamCreate(&(plan[i].stream)));
		runCuda(hipMallocAsync((void**)&(plan[i].d_indices), sizeof(int) * numLabels, plan[i].stream));
		//printf("10\n");

		//Both
		runCuda(hipMallocAsync((void**)&(plan[i].d_lr_weights), sizeof(double) * cluster_params[i]->lr_weights.size(), plan[i].stream));
		runCuda(hipMemcpyAsync(plan[i].d_lr_weights, cluster_params[i]->lr_weights.data(), sizeof(double) * cluster_params[i]->lr_weights.size(), hipMemcpyHostToDevice, plan[i].stream));
	}

	//printf("11\n");
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel
	for (int i = 0; i < numClusters; i++)
	{
//		unsigned int i = omp_get_thread_num();

//		printf("i=%d, start log_likelihood_v2\n", i);
		hipSetDevice(plan[i].deviceId);
		//Find indices
		//Can be used on any GPU
		//printf("12\n");
		sample_sub_clusters_worker_v2(i + 1, plan[i].d_indices, plan[i].indicesSize, plan[i].stream, plan[i].deviceId);
		//printf("13\n");

		//Return the likelihood in r vector.
		//Can be used on any GPU
		runCuda(hipMallocAsync((void**)&(plan[i].d_r), sizeof(double) * plan[i].indicesSize * 2, plan[i].stream));

		//printf("14\n");
		log_likelihood_v2(plan[i].d_r, 0, plan[i].d_indices, plan[i].indicesSize, dim, cluster_params[i]->l_dist, plan[i].stream, plan[i].deviceId);
		log_likelihood_v2(plan[i].d_r, plan[i].indicesSize, plan[i].d_indices, plan[i].indicesSize, dim, cluster_params[i]->r_dist, plan[i].stream, plan[i].deviceId);
//		printf("i=%d, end log_likelihood_v2\n", i);
		//printf("15\n");
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipStreamSynchronize(plan[i].stream));
	}
	//printf("16\n");

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		//run on one GPU (plan[0].deviceId) - maybe could be optimized
		hipSetDevice(plan[0].deviceId);
		
		//printf("1\n");
		int* d_indices;
		bool needToFree_d_indices;
		device_to_device_copy(plan[i].deviceId, plan[0].deviceId, plan[i].indicesSize, plan[i].d_indices, d_indices, false, needToFree_d_indices);

		//printf("2\n");
		double* d_lr_weights;
		bool needToFree_d_lr_weights;
		device_to_device_copy(plan[i].deviceId, plan[0].deviceId, cluster_params[i]->lr_weights.size(), plan[i].d_lr_weights, d_lr_weights, false, needToFree_d_lr_weights);

		//printf("3\n");
		double* d_r;
		bool needToFree_d_r;
		device_to_device_copy(plan[i].deviceId, plan[0].deviceId, plan[i].indicesSize * 2, plan[i].d_r, d_r, false, needToFree_d_r);

		sample_log_cat_array_sub_cluster_v2(d_r, plan[i].indicesSize, d_indices, plan[i].indicesSize, d_lr_weights, plan[0].stream, plan[0].deviceId);

		if (needToFree_d_indices)
		{
			runCuda(hipFreeAsync(d_indices, plan[0].stream));
		}
		if (needToFree_d_lr_weights)
		{
			runCuda(hipFreeAsync(d_lr_weights, plan[0].stream));
		}
		if (needToFree_d_r)
		{
			runCuda(hipFreeAsync(d_r, plan[0].stream));
		}
	}

	//Wait for all operations to finish
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipStreamSynchronize(plan[i].stream));
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipFreeAsync(plan[i].d_indices, plan[i].stream));
		runCuda(hipFreeAsync(plan[i].d_r, plan[i].stream));

		runCuda(hipFreeAsync(plan[i].d_lr_weights, plan[i].stream));

		runCuda(hipStreamDestroy(plan[i].stream));
	}

	delete[]plan;
}

typedef struct
{
	//Stream for asynchronous command execution
	hipStream_t stream;
	int deviceId;
	double* d_r;
} clusters_labels_plan;

void cudaKernel::create_clusters_labels(int numClusters, std::vector<thin_cluster_params*>& cluster_params, std::vector<double>& weights, bool bFinal)
{
	int masterDevice = -1;
	clusters_labels_plan* plan = new clusters_labels_plan[numClusters];
	const mv_gaussian* ds = (mv_gaussian*)(cluster_params[0]->cluster_dist);
	int dim = ds->invSigma.rows();

	//printf("Need %ld,  sizeof(double):%ld,  numLabels:%ld,  numClusters:%ld\n", sizeof(double) * numLabels * numClusters, sizeof(double) , numLabels , numClusters);

	//Allocate memory for all streams
	for (int i = 0; i < numClusters; i++)
	{
		plan[i].deviceId = peak_device();
		runCuda(hipStreamCreate(&(plan[i].stream)));

		if (i == 0)
		{
			masterDevice = plan[i].deviceId;
			runCuda(hipMalloc((void**)&(plan[i].d_r), sizeof(double) * numLabels * numClusters));
		}
		else if (masterDevice != plan[i].deviceId)
		{
			runCuda(hipMalloc((void**)&(plan[i].d_r), sizeof(double) * numLabels));
		}

		//		printf("ds->invSigma.rows():%ld, ds->invSigma.cols():%ld, numLabels:%ld\n", ds->invSigma.rows(), ds->invSigma.cols(), numLabels);
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
//		printf("i=%d, start log_likelihood_v3\n",i);
		hipSetDevice(plan[i].deviceId);
		if (masterDevice == plan[i].deviceId)
		{
			log_likelihood_v3(plan[0].d_r + i * numLabels, dim, weights[i], cluster_params[i]->cluster_dist, plan[i].stream, plan[i].deviceId);
		}
		else
		{
			log_likelihood_v3(plan[i].d_r, dim, weights[i], cluster_params[i]->cluster_dist, plan[i].stream, plan[i].deviceId);
		}
//		printf("i=%d, end log_likelihood_v3\n", i);
	}

	//Wait for all operations to finish
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipStreamSynchronize(plan[i].stream));
	}

	//Copy d_r from all streams and GPU to one
	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		if (masterDevice != plan[i].deviceId)
		{
			//For improvement maybe worth to check if hipDeviceCanAccessPeer
			double* r = new double[numLabels];

			hipSetDevice(plan[i].deviceId);
			runCuda(hipMemcpy(r, plan[i].d_r, numLabels * sizeof(double), hipMemcpyDeviceToHost));

			hipSetDevice(plan[0].deviceId);
			runCuda(hipMemcpy(plan[0].d_r + i * numLabels, r, numLabels * sizeof(double), hipMemcpyHostToDevice));

			delete []r;
		}
	}

	hipSetDevice(plan[0].deviceId);
	if (bFinal)
	{
		update_labels_by_max_index(plan[0].d_r, numClusters, plan[0].stream, plan[0].deviceId);
	}
	else
	{
		sample_log_cat_array_v2(plan[0].d_r, numClusters, plan[0].stream, plan[0].deviceId);
	}

	//omp_set_num_threads(numClusters);
	//#pragma omp parallel for
	for (int i = 0; i < numClusters; i++)
	{
		hipSetDevice(plan[i].deviceId);
		runCuda(hipStreamSynchronize(plan[i].stream));
		runCuda(hipStreamDestroy(plan[i].stream));
		if (i == 0 || masterDevice != plan[i].deviceId)
		{
			runCuda(hipFree(plan[i].d_r));
		}
	}

	delete[]plan;
}

void cudaKernel::checkCUDAError(hipError_t err, const char* file, int line)
{
	if (hipSuccess != err)
	{
		printf("Cuda error: %s(%d):%s.\n", file, line, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

template<typename T>
void cudaKernel::device_to_device_copy(int srcDeviceId, int trgDeviceId, int dataSize, T* srcData, T*& trgData, bool alreadyAllocated, bool& needToFree, hipStream_t& stream)
{
	if (srcDeviceId != trgDeviceId)
	{
		runCuda(hipStreamSynchronize(stream));
	}
	device_to_device_copy(srcDeviceId, trgDeviceId, dataSize, srcData, trgData, alreadyAllocated, needToFree);
}

template<typename T>
void cudaKernel::device_to_device_copy(int srcDeviceId, int trgDeviceId, int dataSize, T* srcData, T* &trgData, bool alreadyAllocated, bool &needToFree)
{
	needToFree = false;
//	printf("alreadyAllocated=%d\n", alreadyAllocated);

	if (srcDeviceId == trgDeviceId)
	{
		trgData = srcData;
	}
	else
	{
		T* data = new T[dataSize];

		hipSetDevice(srcDeviceId);
		runCuda(hipMemcpy(data, srcData, dataSize * sizeof(T), hipMemcpyDeviceToHost));

		hipSetDevice(trgDeviceId);

		if (!alreadyAllocated)
		{
			runCuda(hipMalloc((void**)&trgData, dataSize * sizeof(T)));
			needToFree = true;
		}

		runCuda(hipMemcpy(trgData, data, dataSize * sizeof(T), hipMemcpyHostToDevice));

		delete[]data;
	}
}

#endif