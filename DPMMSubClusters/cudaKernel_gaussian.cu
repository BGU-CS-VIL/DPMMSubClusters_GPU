#include "hip/hip_runtime.h"
#ifndef CudaKernel_gaussian_CU
#define CudaKernel_gaussian_CU

#include "cudaKernel_gaussian.cuh"

void cudaKernel_gaussian::log_likelihood(Eigen::VectorXd& r, const Eigen::MatrixXd& x, const distribution_sample* distribution_sample)
{
	const mv_gaussian* pDistribution_sample = (mv_gaussian*)distribution_sample;
	MatrixXd z = x.colwise() - pDistribution_sample->mu;

	int sizeVec = z.cols();

	double* d_z;
	double* d_b;
	double* d_c;
	double* d_r;

	runCuda(hipMalloc((void**)&d_z, sizeof(double) * z.size()));
	runCuda(hipMalloc((void**)&d_b, sizeof(double) * pDistribution_sample->invSigma.size()));
	runCuda(hipMalloc((void**)&d_c, sizeof(double) * pDistribution_sample->invSigma.rows() * z.cols()));
	runCuda(hipMemcpy(d_z, z.data(), sizeof(double) * z.size(), hipMemcpyHostToDevice));
	runCuda(hipMemcpy(d_b, pDistribution_sample->invSigma.data(), sizeof(double) * pDistribution_sample->invSigma.size(), hipMemcpyHostToDevice));

	naive_matrix_multiply(d_b, d_z, d_c, pDistribution_sample->invSigma.rows(), z.cols(), pDistribution_sample->invSigma.cols());

	runCuda(hipFree(d_b));

	runCuda(hipMalloc((void**)&d_r, sizeof(double) * sizeVec));

	double scalar = -((pDistribution_sample->sigma.size() * log(2 * EIGEN_PI) + pDistribution_sample->logdetSigma) / 2);
	dcolwise_dot_all(sizeVec, z.rows(), d_z, d_c, scalar, d_r);

	r.resize(sizeVec);
	runCuda(hipMemcpy(r.data(), d_r, sizeof(double) * sizeVec, hipMemcpyDeviceToHost));

	runCuda(hipFree(d_z));
	runCuda(hipFree(d_c));
	runCuda(hipFree(d_r));
}

//__global__ void get_mat_from_indices(double* d_points, double* d_x, int* d_indices, int dim, int indicesSize)
//{
//	int idx = threadIdx.x + blockIdx.x * blockDim.x;
//	if (idx < indicesSize)
//	{
//		for (int j = 0; j < dim; j++)
//		{
//			d_x[IDX2C(j, idx, dim)] = d_points[IDX2C(j, d_indices[idx], dim)];
//		}
//	}
//}


__global__ void divide_points_by_mu(double* d_points, int* d_indices, int dim, int indicesSize, double* d_mu, double* d_z)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < indicesSize)
	{
		for (int j = 0; j < dim; j++)
		{
			d_z[IDX2C(j, idx, dim)] = d_points[IDX2C(j, d_indices[idx], dim)] - d_mu[j];
		}
	}
}

void cudaKernel_gaussian::log_likelihood_v2(double* d_r, int* d_indices, int indicesSize, int dim, const distribution_sample* distribution_sample, hipStream_t& stream)
{
	const mv_gaussian* pDistribution_sample = (mv_gaussian*)distribution_sample;

	//TODO - remove for perf.
	//double* d_x;
	//runCuda(hipMalloc((void**)&d_x, sizeof(double) * dim* indicesSize));
	//get_mat_from_indices << <blocks, threads >> > (d_points, d_x, d_indices, dim, indicesSize);
	//runCuda(hipPeekAtLastError());
	//runCuda(hipDeviceSynchronize());

	//MatrixXd x(dim, indicesSize);
	//runCuda(hipMemcpy(x.data(), d_x, sizeof(double) * dim * indicesSize, hipMemcpyDeviceToHost));
	//End - remove for perf.

//	MatrixXd z = x.colwise() - pDistribution_sample->mu;
	double* d_z;
	double* d_mu;
	runCuda(hipMalloc((void**)&d_z, sizeof(double) * dim * indicesSize));
	runCuda(hipMalloc((void**)&d_mu, sizeof(double) * pDistribution_sample->mu.size()));
	runCuda(hipMemcpy(d_mu, pDistribution_sample->mu.data(), sizeof(double) * pDistribution_sample->mu.size(), hipMemcpyHostToDevice));
	divide_points_by_mu << <blocks, threads , 0, stream>> > (d_points, d_indices, dim, indicesSize, d_mu, d_z);

	double* d_b;
	double* d_c;
	
	runCuda(hipMalloc((void**)&d_b, sizeof(double) * pDistribution_sample->invSigma.size()));
	runCuda(hipMalloc((void**)&d_c, sizeof(double) * pDistribution_sample->invSigma.rows() * indicesSize));
//	runCuda(hipMemcpy(d_z, z.data(), sizeof(double) * dim * indicesSize, hipMemcpyHostToDevice));
	runCuda(hipMemcpy(d_b, pDistribution_sample->invSigma.data(), sizeof(double) * pDistribution_sample->invSigma.size(), hipMemcpyHostToDevice));

	hipStreamSynchronize(stream);
	naive_matrix_multiply(d_b, d_z, d_c, pDistribution_sample->invSigma.rows(), indicesSize, pDistribution_sample->invSigma.cols());

	runCuda(hipFree(d_b));

	double scalar = -((pDistribution_sample->sigma.size() * log(2 * EIGEN_PI) + pDistribution_sample->logdetSigma) / 2);
	dcolwise_dot_all(indicesSize, dim, d_z, d_c, scalar, d_r);

	runCuda(hipFree(d_mu));
	runCuda(hipFree(d_z));
	runCuda(hipFree(d_c));
}


#endif //CudaKernel_gaussian_CU