#include "hip/hip_runtime.h"
#ifndef CudaKernel_gaussian_CU
#define CudaKernel_gaussian_CU

#include "cudaKernel_gaussian.cuh"

void cudaKernel_gaussian::log_likelihood(Eigen::VectorXd& r, const Eigen::MatrixXd& x, const distribution_sample* distribution_sample)
{
	const mv_gaussian* pDistribution_sample = (mv_gaussian*)distribution_sample;
	MatrixXd z = x.colwise() - pDistribution_sample->mu;

	int sizeVec = z.cols();

	double* d_z;
	double* d_b;
	double* d_c;
	double* d_r;

	runCuda(hipMalloc((void**)&d_z, sizeof(double) * z.size()));
	runCuda(hipMalloc((void**)&d_b, sizeof(double) * pDistribution_sample->invSigma.size()));
	runCuda(hipMalloc((void**)&d_c, sizeof(double) * pDistribution_sample->invSigma.rows() * z.cols()));
	runCuda(hipMemcpy(d_z, z.data(), sizeof(double) * z.size(), hipMemcpyHostToDevice));
	runCuda(hipMemcpy(d_b, pDistribution_sample->invSigma.data(), sizeof(double) * pDistribution_sample->invSigma.size(), hipMemcpyHostToDevice));

	naive_matrix_multiply(d_b, d_z, d_c, pDistribution_sample->invSigma.rows(), z.cols(), pDistribution_sample->invSigma.cols());

	runCuda(hipFree(d_b));

	runCuda(hipMalloc((void**)&d_r, sizeof(double) * sizeVec));

	double scalar = -((pDistribution_sample->sigma.size() * log(2 * EIGEN_PI) + pDistribution_sample->logdetSigma) / 2);

	dcolwise_dot_all(sizeVec, z.rows(), d_z, d_c, scalar, d_r, 0);

	r.resize(sizeVec);
	runCuda(hipMemcpy(r.data(), d_r, sizeof(double) * sizeVec, hipMemcpyDeviceToHost));

	runCuda(hipFree(d_z));
	runCuda(hipFree(d_c));
	runCuda(hipFree(d_r));
}

//__global__ void get_mat_from_indices(double* d_points, double* d_x, int* d_indices, int dim, int indicesSize)
//{
//	int idx = threadIdx.x + blockIdx.x * blockDim.x;
//	if (idx < indicesSize)
//	{
//		for (int j = 0; j < dim; j++)
//		{
//			d_x[IDX2C(j, idx, dim)] = d_points[IDX2C(j, d_indices[idx], dim)];
//		}
//	}
//}


__global__ void divide_points_by_mu(double* d_points, int* d_indices, int dim, int indicesSize, double* d_mu, double* d_z)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < indicesSize)
	{
		for (int j = 0; j < dim; j++)
		{
			d_z[IDX2C(j, idx, dim)] = d_points[IDX2C(j, d_indices[idx], dim)] - d_mu[j];
		}
	}
}

__global__ void divide_points_by_mu_all(double* d_points, int dim, int indicesSize, double* d_mu, double* d_z)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < indicesSize)
	{
		for (int j = 0; j < dim; j++)
		{
			d_z[IDX2C(j, idx, dim)] = d_points[IDX2C(j, idx, dim)] - d_mu[j];
		}
	}
}

void cudaKernel_gaussian::log_likelihood_v2(
	double* d_r,
	int r_offset,
	int* d_indices,
	int indicesSize,
	int dim, 
	const distribution_sample* distribution_sample, 
	hipStream_t& stream,
	int deviceId)
{
	const mv_gaussian* pDistribution_sample = (mv_gaussian*)distribution_sample;
	double* d_b;
	double* d_c;
	double* d_z;
	double* d_mu;

	//TODO - remove for perf.
	//double* d_x;
	//runCuda(hipMalloc((void**)&d_x, sizeof(double) * dim* indicesSize));
	//get_mat_from_indices << <blocks, threads >> > (d_points, d_x, d_indices, dim, indicesSize);
	//runCuda(hipPeekAtLastError());
	//runCuda(hipDeviceSynchronize());

	//MatrixXd x(dim, indicesSize);
	//runCuda(hipMemcpy(x.data(), d_x, sizeof(double) * dim * indicesSize, hipMemcpyDeviceToHost));
	//End - remove for perf.

//	MatrixXd z = x.colwise() - pDistribution_sample->mu;
	runCuda(hipMallocAsync((void**)&d_mu, sizeof(double) * pDistribution_sample->mu.size(), stream));
	runCuda(hipMemcpyAsync(d_mu, pDistribution_sample->mu.data(), sizeof(double) * pDistribution_sample->mu.size(), hipMemcpyHostToDevice, stream));
	runCuda(hipMallocAsync((void**)&d_z, sizeof(double) * dim* indicesSize, stream));
	runCuda(hipMallocAsync((void**)&d_c, sizeof(double) * pDistribution_sample->invSigma.rows() * indicesSize, stream));

	divide_points_by_mu << <blocks, threads , 0, stream>> > (gpuCapabilities[deviceId].d_points, d_indices, dim, indicesSize, d_mu, d_z);
	runCuda(hipFreeAsync(d_mu, stream));

	runCuda(hipMallocAsync((void**)&d_b, sizeof(double) * pDistribution_sample->invSigma.size(), stream));
	runCuda(hipMemcpyAsync(d_b, pDistribution_sample->invSigma.data(), sizeof(double) * pDistribution_sample->invSigma.size(), hipMemcpyHostToDevice, stream));

	//naive_matrix_multiply_v2(d_b, d_z, d_c, pDistribution_sample->invSigma.rows(), indicesSize, pDistribution_sample->invSigma.cols(), stream);
	naive_matrix_multiply_v3(d_b, d_z, d_c, pDistribution_sample->invSigma.rows(), pDistribution_sample->invSigma.cols(), indicesSize, stream);
	runCuda(hipFreeAsync(d_b,stream));

	double scalar = -((pDistribution_sample->sigma.size() * log(2 * EIGEN_PI) + pDistribution_sample->logdetSigma) / 2);
	dcolwise_dot_all_v2(indicesSize, dim, d_z, d_c, scalar, d_r, r_offset, stream);
	runCuda(hipFreeAsync(d_z, stream));
	runCuda(hipFreeAsync(d_c, stream));
}

void cudaKernel_gaussian::log_likelihood_v3(
	double* d_r,
	int dim,
	double weight, 
	const distribution_sample* distribution_sample,
	hipStream_t& stream,
	int deviceId)
{
	const mv_gaussian* pDistribution_sample = (mv_gaussian*)distribution_sample;
	double* d_b;
	double* d_c;
	double* d_z;
	double* d_mu;

	runCuda(hipMallocAsync((void**)&d_mu, sizeof(double) * pDistribution_sample->mu.size(), stream));
	runCuda(hipMemcpyAsync(d_mu, pDistribution_sample->mu.data(), sizeof(double) * pDistribution_sample->mu.size(), hipMemcpyHostToDevice, stream));
	runCuda(hipMallocAsync((void**)&d_z, sizeof(double) * dim * numLabels, stream));
	runCuda(hipMallocAsync((void**)&d_c, sizeof(double) * pDistribution_sample->invSigma.rows() * numLabels, stream));
	divide_points_by_mu_all << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].d_points, dim, numLabels, d_mu, d_z);
	runCuda(hipFreeAsync(d_mu, stream));

	runCuda(hipMallocAsync((void**)&d_b, sizeof(double) * pDistribution_sample->invSigma.size(), stream));
	runCuda(hipMemcpyAsync(d_b, pDistribution_sample->invSigma.data(), sizeof(double) * pDistribution_sample->invSigma.size(), hipMemcpyHostToDevice, stream));

	//naive_matrix_multiply_v2(d_b, d_z, d_c, pDistribution_sample->invSigma.rows(), numLabels, pDistribution_sample->invSigma.cols(), stream);
	naive_matrix_multiply_v3(d_b, d_z, d_c, pDistribution_sample->invSigma.rows(), pDistribution_sample->invSigma.cols(), numLabels, stream);
	runCuda(hipFreeAsync(d_b, stream));

	double scalar = -((pDistribution_sample->sigma.size() * log(2 * EIGEN_PI) + pDistribution_sample->logdetSigma) / 2);
	dcolwise_dot_all_v3(numLabels, dim, d_z, d_c, scalar, d_r, weight, stream);
	runCuda(hipFreeAsync(d_z, stream));
	runCuda(hipFreeAsync(d_c, stream));
}


#endif //CudaKernel_gaussian_CU