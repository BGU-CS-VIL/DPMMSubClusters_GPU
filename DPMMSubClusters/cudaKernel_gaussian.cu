#include "hip/hip_runtime.h"
#ifndef CudaKernel_gaussian_CU
#define CudaKernel_gaussian_CU

#include "cudaKernel_gaussian.cuh"

void cudaKernel_gaussian::log_likelihood(Eigen::VectorXd& r, const Eigen::MatrixXd& x, const distribution_sample* distribution_sample)
{
	const mv_gaussian* pDistribution_sample = (mv_gaussian*)distribution_sample;
	MatrixXd z = x.colwise() - pDistribution_sample->mu;

	int sizeVec = z.cols();

	double* d_z;
	double* d_b;
	double* d_c;
	double* d_r;

	runCuda(hipMalloc((void**)&d_z, sizeof(double) * z.size()));
	runCuda(hipMalloc((void**)&d_b, sizeof(double) * pDistribution_sample->invSigma.size()));
	runCuda(hipMalloc((void**)&d_c, sizeof(double) * pDistribution_sample->invSigma.rows() * z.cols()));
	runCuda(hipMemcpy(d_z, z.data(), sizeof(double) * z.size(), hipMemcpyHostToDevice));
	runCuda(hipMemcpy(d_b, pDistribution_sample->invSigma.data(), sizeof(double) * pDistribution_sample->invSigma.size(), hipMemcpyHostToDevice));

	naive_matrix_multiply(d_b, d_z, d_c, pDistribution_sample->invSigma.rows(), z.cols(), pDistribution_sample->invSigma.cols());

	runCuda(hipFree(d_b));

	runCuda(hipMalloc((void**)&d_r, sizeof(double) * sizeVec));

	double scalar = -((pDistribution_sample->sigma.size() * log(2 * EIGEN_PI) + pDistribution_sample->logdetSigma) / 2);
	int* d_zero;
	runCuda(hipMalloc((void**)&d_zero, sizeof(int)));
	runCuda(hipMemset(d_zero, 0, sizeof(int)));
	int* d_sizeVec;
	runCuda(hipMalloc((void**)&d_sizeVec, sizeof(int)));
	runCuda(hipMemcpy(d_sizeVec, &sizeVec, sizeof(int), hipMemcpyHostToDevice));

	dcolwise_dot_all(d_sizeVec, z.rows(), d_z, d_c, scalar, d_r, d_zero);

	r.resize(sizeVec);
	runCuda(hipMemcpy(r.data(), d_r, sizeof(double) * sizeVec, hipMemcpyDeviceToHost));

	runCuda(hipFree(d_zero));
	runCuda(hipFree(d_sizeVec));
	runCuda(hipFree(d_z));
	runCuda(hipFree(d_c));
	runCuda(hipFree(d_r));
}

//__global__ void get_mat_from_indices(double* d_points, double* d_x, int* d_indices, int dim, int indicesSize)
//{
//	int idx = threadIdx.x + blockIdx.x * blockDim.x;
//	if (idx < indicesSize)
//	{
//		for (int j = 0; j < dim; j++)
//		{
//			d_x[IDX2C(j, idx, dim)] = d_points[IDX2C(j, d_indices[idx], dim)];
//		}
//	}
//}


__global__ void divide_points_by_mu(double* d_points, int* d_indices, int dim, int* d_indicesSize, double* d_mu, double* d_z)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < *d_indicesSize)
	{
		for (int j = 0; j < dim; j++)
		{
			d_z[IDX2C(j, idx, dim)] = d_points[IDX2C(j, d_indices[idx], dim)] - d_mu[j];
		}
	}
}

__global__ void divide_points_by_mu_all(double* d_points, int dim, int* d_indicesSize, double* d_mu, double* d_z)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < *d_indicesSize)
	{
		for (int j = 0; j < dim; j++)
		{
			d_z[IDX2C(j, idx, dim)] = d_points[IDX2C(j, idx, dim)] - d_mu[j];
		}
	}
}

void cudaKernel_gaussian::log_likelihood_v2(
	double* d_r,
	int *d_r_offset,
	double* d_b,
	double* d_c,
	double* d_z,
	double* d_mu,
	int* d_indices,
	int * d_indicesSize,
	int dim, 
	const distribution_sample* distribution_sample, 
	hipStream_t& stream)
{
	const mv_gaussian* pDistribution_sample = (mv_gaussian*)distribution_sample;

	//TODO - remove for perf.
	//double* d_x;
	//runCuda(hipMalloc((void**)&d_x, sizeof(double) * dim* indicesSize));
	//get_mat_from_indices << <blocks, threads >> > (d_points, d_x, d_indices, dim, indicesSize);
	//runCuda(hipPeekAtLastError());
	//runCuda(hipDeviceSynchronize());

	//MatrixXd x(dim, indicesSize);
	//runCuda(hipMemcpy(x.data(), d_x, sizeof(double) * dim * indicesSize, hipMemcpyDeviceToHost));
	//End - remove for perf.

//	MatrixXd z = x.colwise() - pDistribution_sample->mu;
	runCuda(hipMemcpyAsync(d_mu, pDistribution_sample->mu.data(), sizeof(double) * pDistribution_sample->mu.size(), hipMemcpyHostToDevice, stream));
	divide_points_by_mu << <blocks, threads , 0, stream>> > (d_points, d_indices, dim, d_indicesSize, d_mu, d_z);

	runCuda(hipMemcpyAsync(d_b, pDistribution_sample->invSigma.data(), sizeof(double) * pDistribution_sample->invSigma.size(), hipMemcpyHostToDevice, stream));

	naive_matrix_multiply_v2(d_b, d_z, d_c, pDistribution_sample->invSigma.rows(), d_indicesSize, pDistribution_sample->invSigma.cols(), stream);

	double scalar = -((pDistribution_sample->sigma.size() * log(2 * EIGEN_PI) + pDistribution_sample->logdetSigma) / 2);
	dcolwise_dot_all_v2(d_indicesSize, dim, d_z, d_c, scalar, d_r, d_r_offset, stream);
}

void cudaKernel_gaussian::log_likelihood_v3(
	double* d_r,
	int* d_r_offset,
	double* d_b,
	double* d_c,
	double* d_z,
	double* d_mu,
	int* d_indicesSize,
	int dim,
	double weight,
	const distribution_sample* distribution_sample,
	hipStream_t& stream)
{
	const mv_gaussian* pDistribution_sample = (mv_gaussian*)distribution_sample;

	runCuda(hipMemcpyAsync(d_mu, pDistribution_sample->mu.data(), sizeof(double) * pDistribution_sample->mu.size(), hipMemcpyHostToDevice, stream));
	divide_points_by_mu_all << <blocks, threads, 0, stream >> > (d_points, dim, d_indicesSize, d_mu, d_z);
//	runCuda(hipStreamSynchronize(stream));

	runCuda(hipMemcpyAsync(d_b, pDistribution_sample->invSigma.data(), sizeof(double) * pDistribution_sample->invSigma.size(), hipMemcpyHostToDevice, stream));
//	runCuda(hipStreamSynchronize(stream));

	naive_matrix_multiply_v2(d_b, d_z, d_c, pDistribution_sample->invSigma.rows(), d_indicesSize, pDistribution_sample->invSigma.cols(), stream);
//	runCuda(hipStreamSynchronize(stream));

	double scalar = -((pDistribution_sample->sigma.size() * log(2 * EIGEN_PI) + pDistribution_sample->logdetSigma) / 2);
	dcolwise_dot_all_v3(d_indicesSize, dim, d_z, d_c, scalar, d_r, weight, stream);
//	runCuda(hipStreamSynchronize(stream));
}


#endif //CudaKernel_gaussian_CU