#include "hip/hip_runtime.h"
#ifndef CudaKernel_gaussian_CU
#define CudaKernel_gaussian_CU

#include "cudaKernel_gaussian.cuh"
#include "niw_hyperparams.h"
#include "niw_sufficient_statistics.h"
#include "check_time.h"

__global__ void divide_points_by_mu_kernel(double* d_points, int* d_indices, int dim, int indicesSize, double* d_mu, double* d_z)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < indicesSize)
	{
		for (int j = 0; j < dim; j++)
		{
			d_z[IDX2C(j, idx, dim)] = d_points[IDX2C(j, d_indices[idx], dim)] - d_mu[j];
		}
	}
}

__global__ void divide_points_by_mu_all_kernel(double* d_points, int dim, int indicesSize, double* d_mu, double* d_z)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < indicesSize)
	{
		for (int j = 0; j < dim; j++)
		{
			d_z[IDX2C(j, idx, dim)] = d_points[IDX2C(j, idx, dim)] - d_mu[j];
		}
	}
}
__global__ void mul_scalar_sum_A_AT_kernel(double* d_A, double* d_B, int n, double scalar)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	double sum = 0;

	if (col < n && row < n)
	{
		d_B[IDX2C(row, col, n)] = scalar * (d_A[IDX2C(row, col, n)] + d_A[IDX2C(col, row, n)]);
	}
}

void cudaKernel_gaussian::log_likelihood_sub_labels(
	double* d_r,
	int r_offset,
	int* d_indices,
	int indicesSize,
	int dim, 
	const std::shared_ptr<distribution_sample>& distribution_sample,
	hipStream_t& stream,
	int deviceId)
{
	mv_gaussian* pDistribution_sample = dynamic_cast<mv_gaussian*>(distribution_sample.get());

	double* d_b;
	double* d_c;
	double* d_z;
	double* d_mu;

	runCuda(hipMallocAsync((void**)&d_mu, sizeof(double) * pDistribution_sample->mu.size(), stream));
	runCuda(hipMemcpyAsync(d_mu, pDistribution_sample->mu.data(), sizeof(double) * pDistribution_sample->mu.size(), hipMemcpyHostToDevice, stream));
	runCuda(hipMallocAsync((void**)&d_z, sizeof(double) * dim* indicesSize, stream));
	runCuda(hipMallocAsync((void**)&d_c, sizeof(double) * pDistribution_sample->invSigma.rows() * indicesSize, stream));

	divide_points_by_mu_kernel << <blocks, threads , 0, stream>> > (gpuCapabilities[deviceId].d_points, d_indices, dim, indicesSize, d_mu, d_z);
	runCuda(hipPeekAtLastError());
	runCuda(hipFreeAsync(d_mu, stream));

	runCuda(hipMallocAsync((void**)&d_b, sizeof(double) * pDistribution_sample->invSigma.size(), stream));
	runCuda(hipMemcpyAsync(d_b, pDistribution_sample->invSigma.data(), sizeof(double) * pDistribution_sample->invSigma.size(), hipMemcpyHostToDevice, stream));

	naive_matrix_multiply(d_b, d_z, d_c, (int)pDistribution_sample->invSigma.rows(), (int)pDistribution_sample->invSigma.cols(), indicesSize, stream);
	runCuda(hipFreeAsync(d_b,stream));

	double scalar = -((pDistribution_sample->sigma.size() * log(2 * EIGEN_PI) + pDistribution_sample->logdetSigma) / 2);
	dcolwise_dot_all_sub_labels(indicesSize, dim, d_z, d_c, scalar, d_r, r_offset, stream);
	runCuda(hipFreeAsync(d_z, stream));
	runCuda(hipFreeAsync(d_c, stream));
}

//OUTPUT: d_r - log_likelihhod allocated in device memroy
void cudaKernel_gaussian::log_likelihood_labels(
	double* d_r,
	double weight, 
	const std::shared_ptr<distribution_sample>& distribution_sample,
	hipStream_t& stream,
	int deviceId)
{
	mv_gaussian* pDistribution_sample = dynamic_cast<mv_gaussian*>(distribution_sample.get());

	double* d_b;
	double* d_c;
	double* d_z;
	int dim = gpuCapabilities[deviceId].pointsRows;

	runCuda(hipMallocAsync((void**)&d_z, sizeof(double) * dim * numLabels, stream));
	divide_points_by_mu_all(dim, pDistribution_sample, d_z, stream, deviceId);
	hipStreamSynchronize(stream);
	MatrixXd data;
	data.resize(dim, numLabels);
	hipMemcpy(data.data(), d_z, sizeof(double) * data.size(), hipMemcpyDeviceToHost);

	runCuda(hipMallocAsync((void**)&d_c, sizeof(double) * pDistribution_sample->invSigma.rows() * numLabels, stream));
	runCuda(hipMallocAsync((void**)&d_b, sizeof(double) * pDistribution_sample->invSigma.size(), stream));
	runCuda(hipMemcpyAsync(d_b, pDistribution_sample->invSigma.data(), sizeof(double) * pDistribution_sample->invSigma.size(), hipMemcpyHostToDevice, stream));

	naive_matrix_multiply(d_b, d_z, d_c, (int)pDistribution_sample->invSigma.rows(), (int)pDistribution_sample->invSigma.cols(), numLabels, stream);
	runCuda(hipFreeAsync(d_b, stream));

	hipStreamSynchronize(stream);
	data.resize(dim , numLabels);
	hipMemcpy(data.data(), d_z, sizeof(double) * data.size(), hipMemcpyDeviceToHost);

	data.resize(pDistribution_sample->invSigma.rows(), numLabels);
	hipMemcpy(data.data(), d_c, sizeof(double) * data.size(), hipMemcpyDeviceToHost);


	double scalar = -((pDistribution_sample->sigma.size() * log(2 * EIGEN_PI) + pDistribution_sample->logdetSigma) / 2);
	dcolwise_dot_all_labels(numLabels, dim, d_z, d_c, scalar, d_r, weight, stream);
	runCuda(hipFreeAsync(d_z, stream));
	runCuda(hipFreeAsync(d_c, stream));

	data.resize(numLabels,1);
	hipMemcpy(data.data(), d_r, sizeof(double) * data.size(), hipMemcpyDeviceToHost);

}

void cudaKernel_gaussian::divide_points_by_mu_all(int dim, const mv_gaussian* dist, double* d_z, hipStream_t& stream, int deviceId)
{
	double* d_mu;

	runCuda(hipMallocAsync((void**)&d_mu, sizeof(double) * dist->mu.size(), stream));
	runCuda(hipMemcpyAsync(d_mu, dist->mu.data(), sizeof(double) * dist->mu.size(), hipMemcpyHostToDevice, stream));
	divide_points_by_mu_all_kernel << <blocks, threads, 0, stream >> > (gpuCapabilities[deviceId].d_points, dim, numLabels, d_mu, d_z);
	runCuda(hipPeekAtLastError());
	runCuda(hipFreeAsync(d_mu, stream));
}

void cudaKernel_gaussian::do_create_sufficient_statistics(
	double* d_pts,
	int rows,
	int cols,
	const std::shared_ptr<hyperparams>& hyperParams,
	const std::shared_ptr<hyperparams>& posterior,
	hipStream_t& stream,
	std::shared_ptr<sufficient_statistics>& ss)
{
	if (cols == 0)
	{
		niw_hyperparams* pNiw_hyperparams = dynamic_cast<niw_hyperparams*>(hyperParams.get());
		ss = std::make_shared<niw_sufficient_statistics>(cols, VectorXd::Zero(pNiw_hyperparams->m.size()), MatrixXd::Zero(pNiw_hyperparams->m.size(), pNiw_hyperparams->m.size()));
	}
	else
	{
		CHECK_TIME("cudaKernel_gaussian::do_create_sufficient_statistics");
		const int mem_size = rows * cols * sizeof(double);

		double* d_c;
		runCuda(hipMallocAsync(&d_c, rows * rows * sizeof(double), stream));
		runCuda(hipStreamSynchronize(stream));

		multiplie_matrix_by_transpose(d_pts, d_c, rows, cols);

		runCuda(hipStreamSynchronize(stream));

		ss = std::make_shared<niw_sufficient_statistics>();
		niw_sufficient_statistics* niw_ss = dynamic_cast<niw_sufficient_statistics*>(ss.get());
		niw_ss->S.resize(rows, rows);
		niw_ss->N = cols;
		runCuda(hipMemcpyAsync(niw_ss->S.data(), d_c, sizeof(double) * rows * rows, hipMemcpyDeviceToHost, stream));
		runCuda(hipFreeAsync(d_c, stream));

		double* d_sum_rowwise;
		runCuda(hipMallocAsync(&d_sum_rowwise, rows * sizeof(double), stream));

		sum_rowwise(d_pts, d_sum_rowwise, rows, cols, stream);
		niw_ss->points_sum.resize(rows);
		runCuda(hipMemcpyAsync(niw_ss->points_sum.data(), d_sum_rowwise, sizeof(double) * rows, hipMemcpyDeviceToHost, stream));

		runCuda(hipFreeAsync(d_sum_rowwise, stream));
	}
}

//Working for Square Matrix only
void cudaKernel_gaussian::mul_scalar_sum_A_AT(double* d_A, double* d_B, int n, double scalar, hipStream_t& stream)
{
	const int BlockSize = 16;

	unsigned int grid_rows = (n + BlockSize - 1) / BlockSize;
	unsigned int grid_cols = (n + BlockSize - 1) / BlockSize;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BlockSize, BlockSize);

	mul_scalar_sum_A_AT_kernel << <dimGrid, dimBlock, 0, stream >> > (d_A, d_B, n, scalar);
	runCuda(hipPeekAtLastError());
}

void cudaKernel_gaussian::mul_scalar_sum_A_AT(double* d_A, double* d_B, int n, double scalar)
{
	const int BlockSize = 16;

	unsigned int grid_rows = (n + BlockSize - 1) / BlockSize;
	unsigned int grid_cols = (n + BlockSize - 1) / BlockSize;
	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(BlockSize, BlockSize);

	mul_scalar_sum_A_AT_kernel << <dimGrid, dimBlock >> > (d_A, d_B, n, scalar);
	runCuda(hipPeekAtLastError());
}

#endif //CudaKernel_gaussian_CU